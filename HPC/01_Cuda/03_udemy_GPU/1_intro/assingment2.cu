
#include <hip/hip_runtime.h>
#include <ctime>
#include <stdio.h>

__global__ void print_3d(int *vector) {

  int threads_per_block = blockDim.x * blockDim.y * blockDim.z;

  int index = threadIdx.x + (threadIdx.y * (blockDim.z * blockDim.x)) +
              (threadIdx.z * blockDim.z) + (blockIdx.x * threads_per_block) +
              (blockIdx.z * gridDim.x * threads_per_block) +
              (blockIdx.y * gridDim.z * gridDim.x * threads_per_block);

  printf("index: %d value: %d\n", index, vector[index]);
}

int main() {

  printf(" starts ... \n");

  int size = 64;
  int byte_size = size * sizeof(int);
  int *h_input;

  h_input = (int *)malloc(byte_size);

  for (int i = 0; i < size; i++) {
    h_input[i] = rand() % 1000;
  }

  int *d_input;

  hipMalloc((void **)&d_input, byte_size);
  hipMemcpy(d_input, h_input, byte_size, hipMemcpyHostToDevice);

  int nx, ny, nz;
  nx = 4;
  ny = 4;
  nz = 4;

  dim3 block(2, 2, 2);
  dim3 grid(nx / block.x, ny / block.y, nz / block.z);
  print_3d<<<grid, block>>>(d_input);

  hipDeviceSynchronize();

  hipDeviceReset();

  printf(" finished. \n");
  return 0;
}