
// Babak Poursartip
// 09/14/2020

// Udemy Cuda
// unique index calculation

#include <hip/hip_runtime.h>
#include <cstdio>

// ===========================================
__global__ void unique_idx_calc_threadIdx(int *input) {
  int tid = threadIdx.x;
  printf(" my threadIdx: %d,value: %d \n", tid, input[tid]);
}

// ===========================================
// 1d grid, 1d block
__global__ void unique_gid_calculation(int *input) {
  int tid = threadIdx.x;
  int offset = blockIdx.x * blockDim.x;
  int gid = tid + offset;
  printf(" blockIdx.x: %d, threadIdx.x: %d, gid: %d, value: %d \n", blockIdx.x,
         tid, gid, input[gid]);
}

// ===========================================
int main() {
  printf(" starts ...");

  int array_size = 8;
  int array_byte_size = sizeof(int) * array_size;
  int h_data[] = {1, 2, 3, 4, 5, 6, 7, 8}; // array on the host

  printf(" data on the host: \n");
  for (int i = 0; i < array_size; ++i)
    printf(" %d", h_data[i]);
  printf("\n\n");

  int *d_data; // array on the device

  hipMalloc((void **)&d_data, array_byte_size);
  hipMemcpy(d_data, h_data, array_byte_size, hipMemcpyHostToDevice);

  /*
  dim3 block(8);
  dim3 grid(1);
  printf(" data on the device: \n");
  unique_idx_calc_threadIdx<<<grid, block>>>(d_data);
  */

  dim3 block(4);
  dim3 grid(2);
  printf(" data on the device: \n");
  unique_gid_calculation<<<grid, block>>>(d_data);

  hipDeviceSynchronize();

  hipDeviceReset();
  printf(" finished.\n");
  return 0;
}
