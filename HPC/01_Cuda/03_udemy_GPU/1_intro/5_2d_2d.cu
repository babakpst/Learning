
// Babak Poursartip
// 09/14/2020

// Udemy Cuda
// unique index calculation

#include <hip/hip_runtime.h>
#include <cstdio>

// ===========================================
// 2d grid, 2d block
__global__ void unique_gid_calculation_2d_2d(int *input) {

  int tid = blockDim.x * threadIdx.y + threadIdx.x;

  int num_threads_in_a_block = blockDim.x * blockDim.y;
  int block_offset = blockIdx.x * num_threads_in_a_block;

  int num_threads_in_a_row = num_threads_in_a_block * gridDim.x;
  int row_offset = num_threads_in_a_row * blockIdx.y;

  int gid = tid + block_offset + row_offset;

  printf("blockIdx.x : %d, blockIdx.y : %d, threadIdx.x : %d, threadIdx.y : "
         "%d, gid : %d - data: %d \n",
         blockIdx.x, blockIdx.y, threadIdx.x, threadIdx.y, gid, input[gid]);
}

// ===========================================
int main() {
  printf(" starts ...");

  int array_size = 16;
  int array_byte_size = sizeof(int) * array_size;
  int h_data[] = {1, 2,  3,  4,  5,  6,  7,  8,
                  9, 10, 11, 12, 13, 14, 15, 16}; // array on the host

  printf(" data on the host: \n");
  for (int i = 0; i < array_size; ++i)
    printf(" %d", h_data[i]);
  printf("\n\n");

  int *d_data; // array on the device

  hipMalloc((void **)&d_data, array_byte_size);
  hipMemcpy(d_data, h_data, array_byte_size, hipMemcpyHostToDevice);

  dim3 block(2, 2);
  dim3 grid(2, 2);
  printf(" data on the device: \n");
  unique_gid_calculation_2d_2d<<<grid, block>>>(d_data);

  hipDeviceSynchronize();

  hipDeviceReset();
  printf(" finished.");
  return 0;
}