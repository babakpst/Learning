

// Babak Poursartip
// 09/15/2020

// udemy CUDA
// memory management in cude
// start thread in a multiple of 32


#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>
#include <time.h>

// =================================
__global__ void mem_trs_test(int *input) {
  // 1d grid, 1d block
  int gid = blockDim.x * blockIdx.x + threadIdx.x;
  printf(" tid: %3d, gid: %3d, value: %4d \n", threadIdx.x, gid, input[gid]);
}

// =================================
__global__ void mem_trs_test2(int *input, int size) {
  // 1d grid, 1d block
  int gid = blockDim.x * blockIdx.x + threadIdx.x;
  if (gid < size)
    printf(" tid: %3d, gid: %3d, value: %4d \n", threadIdx.x, gid, input[gid]);
}

// =================================
int main() {
  printf(" starts ... \n");

  //  int size = 128;
  int size = 150;
  int byte_size = sizeof(int) * size;

  int *h_input;
  h_input = (int *)malloc(byte_size);

  time_t t;
  srand((unsigned)time(&t));

  for (int i = 0; i < size; ++i) {
    h_input[i] = (int)(rand() & 0xff);
  }

  int *d_input;

  // (void*) double pointer
  hipMalloc((void **)&d_input, byte_size);
  hipMemcpy(d_input, h_input, byte_size, hipMemcpyHostToDevice);

  dim3 block(64);
  dim3 grid(2);

  mem_trs_test<<<grid, block>>>(d_input);
  hipDeviceSynchronize();

  printf("\n second part ========\n \n");

  block.x = 32;
  grid.x = 5;

  mem_trs_test2<<<grid, block>>>(d_input, size);
  hipDeviceSynchronize();

  hipFree(d_input);
  free(h_input);

  hipDeviceReset();
  printf(" finished. \n");
  return 0;
}
