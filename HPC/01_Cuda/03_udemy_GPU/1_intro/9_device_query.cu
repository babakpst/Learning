#include "hip/hip_runtime.h"

#include <stdio.h>

void query_device() {
  int deviceCount = 0;              // how many cuda devices are installed.
  hipGetDeviceCount(&deviceCount); // for mutiple cuda device

  if (deviceCount == 0) {
    printf("No CUDA support device found");
  } else {
    printf("More than one CUDA support device found. Modify the code for "
           "others.\n");
  }

  int devNo = 0; // assuming only one cuda device.
  hipDeviceProp_t iProp;
  hipGetDeviceProperties(&iProp, devNo);

  printf("Device %d: %s\n", devNo, iProp.name);
  printf("  Number of multiprocessors:                     %d\n",
         iProp.multiProcessorCount);
  printf("  clock rate :                     %d\n", iProp.clockRate);
  printf("  Compute capability       :                     %d.%d\n",
         iProp.major, iProp.minor);
  printf("  Total amount of global memory:                 %4.2f KB\n",
         iProp.totalGlobalMem / 1024.0);
  printf("  Total amount of constant memory:               %4.2f KB\n",
         iProp.totalConstMem / 1024.0);
  printf("  Total amount of shared memory per block:       %4.2f KB\n",
         iProp.sharedMemPerBlock / 1024.0);
  printf("  Total amount of shared memory per MP:          %4.2f KB\n",
         iProp.sharedMemPerMultiprocessor / 1024.0);
  printf("  Total number of registers available per block: %d\n",
         iProp.regsPerBlock);
  printf("  Warp size:                                     %d\n",
         iProp.warpSize);
  printf("  Maximum number of threads per block:           %d\n",
         iProp.maxThreadsPerBlock);
  printf("  Maximum number of threads per multiprocessor:  %d\n",
         iProp.maxThreadsPerMultiProcessor);
  printf("  Maximum number of warps per multiprocessor:    %d\n",
         iProp.maxThreadsPerMultiProcessor / 32);
  printf("  Maximum Grid size                         :    (%d,%d,%d)\n",
         iProp.maxGridSize[0], iProp.maxGridSize[1], iProp.maxGridSize[2]);
  printf("  Maximum block dimension                   :    (%d,%d,%d)\n",
         iProp.maxThreadsDim[0], iProp.maxThreadsDim[1],
         iProp.maxThreadsDim[2]);
}

int main() { query_device(); }
