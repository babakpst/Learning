
/*
Babak Poursartip
05/14/2020

based on Intro to parallel programming cuda

remarks:

- data on the host, starts with h (h_in)
- data on the device, starts with d (d_in)


*/


#include <hip/hip_runtime.h>
#include <stdio.h>

// kernel
// __global__: declaration specifier (dec spec)
__global__ void square(float*d_out, float * d_in){
int idx = threadIdx.x;
float f = d_in[idx];
d_out[idx] = f*f;
}


int main(){

printf(" code starts ... \n");

const int ARRAY_SIZE = 64;
const int ARRAY_BYTES = ARRAY_SIZE * sizeof(float);

// generate the input array on the host
float h_in[ARRAY_SIZE];
for (int ii = 0; ii < ARRAY_SIZE; ii++){
h_in[ii] = float(ii);
}

float h_out[ARRAY_SIZE];

// declare gpu memory pointers
float *d_in;
float *d_out;

// allocate gpu memory
hipMalloc((void **) &d_in, ARRAY_BYTES);
hipMalloc((void **) &d_out, ARRAY_BYTES);

// transfer the array to the GPU
hipMemcpy(d_in, h_in, ARRAY_BYTES, hipMemcpyHostToDevice);


// launch the kernel
square<<<1, ARRAY_SIZE>>>(d_out, d_in);

// copy back the results in the CPU
hipMemcpy(h_out,d_out, ARRAY_BYTES,hipMemcpyDeviceToHost);


// print out the results
for (int ii = 0; ii < ARRAY_SIZE; ++ii){
printf(" %f", h_out[ii]);
printf( ( (ii%4) !=3 ) ? "\t":"\n" );
}

// free gpu
hipFree(d_in);
hipFree(d_out);

return 0;
}

