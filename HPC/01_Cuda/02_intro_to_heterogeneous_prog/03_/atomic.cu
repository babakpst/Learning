#include "hip/hip_runtime.h"
#include <stdio.h>

#define NUM_THREADS 1000000
#define ARRAY_SIZE 10
#define BLOCK_WIDTH 1000

__global__ void increment_naive(int *g){

int i =blockIdx.x*blockDim.x +threadIdx.x;

i = i%ARRAY_SIZE;
g[i] = g[i]+1;

}

__global__ void increment_atomic(int *g){
int i =blockIdx.x*blockDim.x +threadIdx.x;

i = i%ARRAY_SIZE;
atomicAdd(&g[i],1); //guarantees that only one thread can do read/modify/write operation.

}



int main(){

printf(" %d of threads in %d blocks writing in %d-element array. \n", NUM_THREADS, NUM_THREADS/BLOCK_WIDTH, ARRAY_SIZE);

float h_arr[ARRAY_SIZE];
const int ARRAY_BYTE = ARRAY_SIZE * sizeof(int);

float *d_arr;
hipMalloc((void **) &d_arr, ARRAY_BYTE);
hipMemset((void *) d_arr, 0, ARRAY_BYTE);

increment_naive<<<NUM_THREADS/BLOCK_WIDTH,BLOCK_WIDTH>>>(d_arr);

hipMemcpy(h_arr, d_arr, ARRAY_BYTE, hipMemcpyDeviceToHost);

hipFree(d_arr);

for (int i = 0; i < ARRAY_SIZE; ++i){
printf(" %d ", h_arr[i] );
}
printf("\n");


float *d_arr;
hipMalloc((void **) &d_arr, ARRAY_BYTE);
hipMemset((void *) d_arr, 0, ARRAY_BYTE);

increment_atomic<<<NUM_THREADS/BLOCK_WIDTH,BLOCK_WIDTH>>>(d_arr);

hipMemcpy(h_arr, d_arr, ARRAY_BYTE, hipMemcpyDeviceToHost);

hipFree(d_arr);

for (int i = 0; i < ARRAY_SIZE; ++i){
printf(" %d ", h_arr[i] );
}
printf("\n");

return 0;
}