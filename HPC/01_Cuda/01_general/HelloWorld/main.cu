// Babak Poursartip
// 30 Oct 2019


#include <hip/hip_runtime.h>
#include <sstream>

__global__ void mykernel(void) {
	printf("Hello, World!\n");
}

int main(void){
	mykernel<<<5,1>>>();
	hipDeviceSynchronize();
	return 0;
}
