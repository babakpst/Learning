#include "hip/hip_runtime.h"
//Udacity HW 4
//Radix Sorting

#include "utils.h"
#include ""
#include <thrust/host_vector.h>

const int BLOCK_SIZE = 1024;

/* Red Eye Removal
   ===============
   
   For this assignment we are implementing red eye removal.  This is
   accomplished by first creating a score for every pixel that tells us how
   likely it is to be a red eye pixel.  We have already done this for you - you
   are receiving the scores and need to sort them in ascending order so that we
   know which pixels to alter to remove the red eye.

   Note: ascending order == smallest to largest

   Each score is associated with a position, when you sort the scores, you must
   also move the positions accordingly.

   Implementing Parallel Radix Sort with CUDA
   ==========================================

   The basic idea is to construct a histogram on each pass of how many of each
   "digit" there are.   Then we scan this histogram so that we know where to put
   the output of each digit.  For example, the first 1 must come after all the
   0s so we have to know how many 0s there are to be able to start moving 1s
   into the correct position.

   1) Histogram of the number of occurrences of each digit
   2) Exclusive Prefix Sum of Histogram
   3) Determine relative offset of each digit
        For example [0 0 1 1 0 0 1]
                ->  [0 1 0 1 2 3 2]
   4) Combine the results of steps 2 & 3 to determine the final
      output location for each element and move it there

   LSB Radix sort is an out-of-place sort and you will need to ping-pong values
   between the input and output buffers we have provided.  Make sure the final
   sorted results end up in the output buffer!  Hint: You may need to do a copy
   at the end.

 */



__global__ void predicate(unsigned int* predicate, const unsigned int* d_in, size_t numElems,int bit) {
	int tid = threadIdx.x;
	int global_id = tid + blockDim.x*blockIdx.x;
	if (global_id >= numElems) return;
	unsigned int bin = ((d_in[global_id] >> bit) & 1u);
	predicate[global_id] =bin;
}


__global__ void bielloch_scan(unsigned int* d_out, const unsigned int* d_in, size_t input_size, unsigned int* blockSums) {
	extern __shared__ unsigned int data[];
	
	int tid = threadIdx.x;
	int offset = 1;
	int abs_start = 2*blockDim.x*blockIdx.x;
	
	data[2 * tid] =(abs_start+2*tid)<input_size? d_in[abs_start+2 * tid]:0;
	data[2 * tid+1] = (abs_start + 2 * tid+1)<input_size ? d_in[abs_start+2 * tid+1]:0;

	for (int d = (2 * blockDim.x) >>1; d>0; d>>=1) {
		__syncthreads();
		
		if (tid < d) {
			int ai = offset*(2 * tid + 1) - 1;
			int bi = offset*(2 * tid + 2) - 1;
			
			data[bi] += data[ai];
		}
		offset <<= 1;
	}
	if (tid == 0)data[2*blockDim.x - 1] = 0;

	for (int d = 1; d < 2 * blockDim.x; d<<=1) {
		offset >>= 1;
		__syncthreads();
		if (tid < d) {
			int ai = offset*(2 * tid + 1) - 1;
			int bi = offset*(2 * tid + 2) - 1;
			unsigned int t = data[ai];
			data[ai] = data[bi];
			data[bi] += t;
		}
	}

	__syncthreads();
	
	if (abs_start + 2 * tid < input_size) {
		d_out[abs_start + 2 * tid] = data[2 * tid];
	}
	if (abs_start + 2 * tid+1 < input_size) {
		d_out[abs_start + 2 * tid+1] = data[2 * tid+1];
	}

	if (tid == 0) {
		blockSums[blockIdx.x] = data[blockDim.x * 2 - 1];
		if(abs_start + blockDim.x * 2 - 1<input_size)blockSums[blockIdx.x]+=d_in[abs_start + blockDim.x * 2 - 1];
	}
}

__global__ void adjustIncrement(unsigned int* d, unsigned int* incr, size_t input_size){
	int pos = blockIdx.x * blockDim.x*2 + threadIdx.x * 2 + 1;
	if (pos< input_size)
	{
		d[pos] += incr[blockIdx.x];
		d[pos-1] += incr[blockIdx.x];
	}
	else if (pos-1 < input_size)
	{
		d[pos-1] += incr[blockIdx.x];
	}
}

__global__ void negatePredicate(unsigned int* predicate, size_t input_size) {
	int tid = threadIdx.x;
	int pos = blockDim.x*blockIdx.x + tid;
	if (pos >= input_size)return;
	predicate[pos] = predicate[pos] ? 0 : 1;
}

__global__ void moveElements(unsigned int* d_out, const unsigned int* d_in, const unsigned int* d_histo, 
								const unsigned int* d_predicate,const unsigned int* d_scan_true, const unsigned int* d_scan_false, size_t input_size) {
	int tid = threadIdx.x;
	int pos = blockDim.x*blockIdx.x + tid;
	if (pos >= input_size)return;
	//calculate new index of element at position pos
	int newindex;	
	if (d_predicate[pos])newindex = d_histo[0] + d_scan_false[pos];
	else newindex = d_histo[1] + d_scan_true[pos];
	if (newindex >= input_size) return; //IMP
	d_out[newindex] = d_in[pos];
}



unsigned int biellochScan(unsigned int* d_scan, unsigned int* d_pred, size_t numElems) {
	
	int num_double_blocks = ceil(1.0f*numElems / (2*BLOCK_SIZE));
	unsigned int* d_blocksums;
	checkCudaErrors(hipMalloc(&d_blocksums, num_double_blocks * sizeof(unsigned int)));
	bielloch_scan << <num_double_blocks, BLOCK_SIZE, 2 * BLOCK_SIZE*sizeof(unsigned int) >> > (d_scan, d_pred, numElems, d_blocksums);
	hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

	unsigned int finalSum;
	//Scan of the blocksums array
	if (num_double_blocks > 1) {
		unsigned int* d_scan_temp;
		checkCudaErrors(hipMalloc(&d_scan_temp, num_double_blocks * sizeof(unsigned int)));
		finalSum=biellochScan(d_scan_temp, d_blocksums, num_double_blocks);
		adjustIncrement << <num_double_blocks, BLOCK_SIZE >> > (d_scan, d_scan_temp, numElems);
		hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
		checkCudaErrors(hipFree(d_scan_temp));
	}
	else {
		
		checkCudaErrors(hipMemcpy(&finalSum, d_blocksums, sizeof(unsigned int), hipMemcpyDeviceToHost));
		checkCudaErrors(hipFree(d_blocksums));
	}
	
	return finalSum;

}

void your_sort(unsigned int* const d_inputVals,
               unsigned int* const d_inputPos,
               unsigned int* const d_outputVals,
               unsigned int* const d_outputPos,
               size_t numElems)
{ 
  //PUT YOUR SORT HERE
	int num_blocks = ceil(1.0f*numElems / BLOCK_SIZE);
	
	unsigned int h_histo[2];
	h_histo[0] = 0;

	unsigned int* d_histo;
	unsigned int* d_pred;
	unsigned int* d_scan_true;
	unsigned int* d_scan_false;
	
	checkCudaErrors(hipMalloc(&d_histo, 2 * sizeof(unsigned int)));
	checkCudaErrors(hipMalloc(&d_pred, numElems*sizeof(unsigned int)));
	checkCudaErrors(hipMalloc(&d_scan_true, numElems * sizeof(unsigned int)));
	checkCudaErrors(hipMalloc(&d_scan_false, numElems * sizeof(unsigned int)));
	//for each of the 32 bits
	for (size_t i = 0; i < 32; i++) {

		//compute predicate
		if (i % 2 == 0)predicate << <num_blocks, BLOCK_SIZE >> > (d_pred, d_inputVals, numElems, i);
		else predicate << <num_blocks, BLOCK_SIZE >> > (d_pred, d_outputVals, numElems, i);
		hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

		
	
		//Exclusive Prefix Sum of 2-bins histogram is: [0 numFalse].
		//You can obtain it buy sum-reduce on predicate: equivalent to last sumBlock of BiellochScan
		
		//Compute offset of positives
		//Bielloch scan
		unsigned int number_trues=biellochScan(d_scan_true, d_pred, numElems);

		//Flip bits
		negatePredicate << <num_blocks, BLOCK_SIZE >> > (d_pred, numElems);
		hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

		//Compute offset of negatives
		unsigned int number_falses=biellochScan(d_scan_false, d_pred, numElems);

		h_histo[1] = number_falses;
		checkCudaErrors(hipMemcpy(d_histo, h_histo, 2 * sizeof(unsigned int), hipMemcpyHostToDevice));

		//Moving elements and indices
		if (i % 2 == 0) {
			moveElements << <num_blocks, BLOCK_SIZE >> > (d_outputVals, d_inputVals, d_histo, d_pred, d_scan_true, d_scan_false, numElems);
			hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
			moveElements << <num_blocks, BLOCK_SIZE >> > (d_outputPos, d_inputPos, d_histo, d_pred, d_scan_true, d_scan_false, numElems);

		}
		else {
			moveElements << <num_blocks, BLOCK_SIZE >> > (d_inputVals, d_outputVals, d_histo, d_pred, d_scan_true, d_scan_false, numElems);
			hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
			moveElements << <num_blocks, BLOCK_SIZE >> > (d_inputPos, d_outputPos, d_histo, d_pred, d_scan_true, d_scan_false, numElems);

		}
			hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
			
	}

	//Copy result into d_outputVals
	checkCudaErrors(hipMemcpy(d_outputVals, d_inputVals, numElems * sizeof(unsigned int), hipMemcpyDeviceToDevice));
	checkCudaErrors(hipMemcpy(d_outputPos, d_inputPos, numElems * sizeof(unsigned int), hipMemcpyDeviceToDevice));

	
	checkCudaErrors(hipFree(d_histo));
	checkCudaErrors(hipFree(d_pred));
	checkCudaErrors(hipFree(d_scan_true));
	checkCudaErrors(hipFree(d_scan_false));

}
