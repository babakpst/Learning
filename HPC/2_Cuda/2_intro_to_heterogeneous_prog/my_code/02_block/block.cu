

/*

Babak Poursartip
05/14/2020

*/


#include <hip/hip_runtime.h>
#include <stdio.h>

__global__
void hello(){

printf("I am: %d %d \n",threadIdx.x, blockIdx.x);

}


int main(){

printf(" code start ... \n");

dim3 blocks(16,1,1);
dim3 threads(1,1,1);

hello<<<blocks,threads>>>();
hipDeviceSynchronize();
printf(" end here. \n");

return 0;
}
