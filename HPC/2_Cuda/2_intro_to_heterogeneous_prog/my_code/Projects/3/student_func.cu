#include "hip/hip_runtime.h"

/* Udacity Homework 3
   HDR Tone-mapping

  Background HDR
  ==============

  A High Dynamic Range (HDR) image contains a wider variation of intensity
  and color than is allowed by the RGB format with 1 byte per channel that we
  have used in the previous assignment.  

  To store this extra information we use single precision floating point for
  each channel.  This allows for an extremely wide range of intensity values.

  In the image for this assignment, the inside of church with light coming in
  through stained glass windows, the raw input floating point values for the
  channels range from 0 to 275.  But the mean is .41 and 98% of the values are
  less than 3!  This means that certain areas (the windows) are extremely bright
  compared to everywhere else.  If we linearly map this [0-275] range into the
  [0-255] range that we have been using then most values will be mapped to zero!
  The only thing we will be able to see are the very brightest areas - the
  windows - everything else will appear pitch black.

  The problem is that although we have cameras capable of recording the wide
  range of intensity that exists in the real world our monitors are not capable
  of displaying them.  Our eyes are also quite capable of observing a much wider
  range of intensities than our image formats / monitors are capable of
  displaying.

  Tone-mapping is a process that transforms the intensities in the image so that
  the brightest values aren't nearly so far away from the mean.  That way when
  we transform the values into [0-255] we can actually see the entire image.
  There are many ways to perform this process and it is as much an art as a
  science - there is no single "right" answer.  In this homework we will
  implement one possible technique.

  Background Chrominance-Luminance
  ================================

  The RGB space that we have been using to represent images can be thought of as
  one possible set of axes spanning a three dimensional space of color.  We
  sometimes choose other axes to represent this space because they make certain
  operations more convenient.

  Another possible way of representing a color image is to separate the color
  information (chromaticity) from the brightness information.  There are
  multiple different methods for doing this - a common one during the analog
  television days was known as Chrominance-Luminance or YUV.

  We choose to represent the image in this way so that we can remap only the
  intensity channel and then recombine the new intensity values with the color
  information to form the final image.

  Old TV signals used to be transmitted in this way so that black & white
  televisions could display the luminance channel while color televisions would
  display all three of the channels.
  

  Tone-mapping
  ============

  In this assignment we are going to transform the luminance channel (actually
  the log of the luminance, but this is unimportant for the parts of the
  algorithm that you will be implementing) by compressing its range to [0, 1].
  To do this we need the cumulative distribution of the luminance values.

  Example
  -------

  input : [2 4 3 3 1 7 4 5 7 0 9 4 3 2]
  min / max / range: 0 / 9 / 9

  histo with 3 bins: [4 7 3]

  cdf : [4 11 14]


  Your task is to calculate this cumulative distribution by following these
  steps.

*/

#include "utils.h"
#include <stdlib.h>

__global__ void reduce_kernel(const float* const d_in, float* d_out, int type)
{

   // sdata is allocated in the kernel call: 3rd arg to <<<b, t, shmem>>>
   extern __shared__ float sdata[];

   int myId = threadIdx.x + blockDim.x * blockIdx.x;
   int tid  = threadIdx.x;

   // load shared mem from global mem
   sdata[tid] = d_in[myId];
   __syncthreads();            // make sure entire block is loaded!

   // do reduction in shared mem
   for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1)
   {
      if (tid < s)
      {
        if (type == 0)
          sdata[tid] = min(sdata[tid], sdata[tid + s]);
        else if (type == 1)
          sdata[tid] = max(sdata[tid], sdata[tid + s]);
      }
      __syncthreads();        // make sure all adds at one stage are done!
   }

   // only thread 0 writes result for this block back to global mem
   if (tid == 0)
   {
      d_out[blockIdx.x] = sdata[0];
   }  

}


__global__ void simple_histo(int *d_bins, const float* const d_in, const int numBins, 
   const float lumMin, const float lumRange)
{
      int myId = threadIdx.x + blockDim.x * blockIdx.x;
      float myItem = d_in[myId];
      int bin = (d_in[myId] - lumMin) / lumRange * numBins;
      int myBin  = min(numBins-1, bin);
      //printf(" MYID: %d- mybin: %d- myItem: %f, lumMin: %f, lumRange: %f \n", myId, myBin, myItem, lumMin, lumRange);
      atomicAdd(&(d_bins[myBin]), 1);
}

// Step efficient scan
__global__ void hillis_steele_algo(unsigned int* d_out, const int* d_in, int size) 
{
	extern __shared__ unsigned int temp[];
   
   int tid = threadIdx.x;
   
   int pout = 0, pin=1;
   
   temp[tid] = tid>0? d_in[tid-1]:0; //exclusive scan
	__syncthreads();

	//double buffered
	for (int offset = 1; offset < size; offset <<= 1) {
		pout = 1 - pout;
		pin = 1 - pout;
      if (tid >= offset) 
        temp[size*pout + tid] = temp[size*pin + tid]+temp[size*pin + tid - offset];
      else 
        temp[size*pout + tid] = temp[size*pin + tid];
		__syncthreads();
	}
	d_out[tid] = temp[pout*size + tid];
}


void your_histogram_and_prefixsum(const float* const d_logLuminance,
                                  unsigned int* const d_cdf,
                                  float &min_logLum,
                                  float &max_logLum,
                                  const size_t numRows,
                                  const size_t numCols,
                                  const size_t numBins)
{
  //TODO
  /*Here are the steps you need to implement
    1) find the minimum and maximum value in the input logLuminance channel
       store in min_logLum and max_logLum
    2) subtract them to find the range
    3) generate a histogram of all the values in the logLuminance channel using
       the formula: bin = (lum[i] - lumMin) / lumRange * numBins
    4) Perform an exclusive scan (prefix sum) on the histogram to get
       the cumulative distribution of luminance values (this should go in the
       incoming d_cdf pointer which already has been allocated for you)       */

  
  const int maxThreadPerBlock = 1024;
  int threads = maxThreadPerBlock;
  int size = numCols*numRows;
  int size_byte = size * sizeof(float);
  float* d_intermediate, *d_min, *d_max;
  
  checkCudaErrors(hipMalloc(&d_intermediate, size_byte));
  checkCudaErrors(hipMalloc(&d_min, sizeof(float)));
  checkCudaErrors(hipMalloc(&d_max, sizeof(float)));

  dim3 block(threads, 1, 1);
  dim3 grid(size / threads, 1, 1);
  
  // 1 =============================================================================================
  reduce_kernel <<<grid, block, threads*sizeof(float)>>> (d_logLuminance, d_intermediate, 0);
  reduce_kernel <<<grid, block, threads*sizeof(float)>>> (d_intermediate, d_min, 0);
  reduce_kernel <<<grid, block, threads*sizeof(float)>>> (d_logLuminance, d_intermediate, 1);
  reduce_kernel <<<grid, block, threads*sizeof(float)>>> (d_intermediate, d_max, 1);
  hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
  checkCudaErrors(hipMemcpy(&min_logLum, d_min, sizeof(float), hipMemcpyDeviceToHost));
  checkCudaErrors(hipMemcpy(&max_logLum, d_max, sizeof(float), hipMemcpyDeviceToHost));
  printf(" cuda min: %f, max: %f \n", min_logLum, max_logLum); 
 
  checkCudaErrors(hipFree(d_intermediate));

  // 2 =============================================================================================
  float lumRange =  max_logLum - min_logLum;

  // 3 =============================================================================================
  int* d_histogram;
  size_byte = numBins * sizeof(int);
  checkCudaErrors(hipMalloc(&d_histogram, size_byte));
  checkCudaErrors(hipMemset(d_histogram, 0, numBins * sizeof(int)));
  simple_histo<<<grid, block>>>(d_histogram, d_logLuminance, numBins, min_logLum, lumRange);
  hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
  
  int *h_histogram=(int *) malloc(sizeof( int)*numBins);
  checkCudaErrors(hipMemcpy(h_histogram, d_histogram, sizeof( int)*numBins, hipMemcpyDeviceToHost));
  
  printf("histogram \n");
  for (size_t i = 0; i < numBins; ++i) {
   printf(" %4ld: %d-",  i, h_histogram[i]);
   if (i%10==0) printf(" \n");
 }
 printf(" \n");

  // 4 =============================================================================================
  // scan
  hillis_steele_algo <<<1, numBins, 2*numBins*sizeof(int) >>> (d_cdf, d_histogram, numBins);
  
  unsigned int *h_cdf=(unsigned int *) malloc(sizeof(unsigned int)*numBins);
  checkCudaErrors(hipMemcpy(h_cdf, d_cdf, sizeof(unsigned int)*numBins, hipMemcpyDeviceToHost));
  
  printf("bin \n");
  for (size_t i = 1; i < numBins; ++i) {
   printf(" %4ld: %d-",  i, h_cdf[i]);
   if (i%10==0) printf(" \n");
 }
 printf(" \n");

}
