#include "hip/hip_runtime.h"
//Udacity HW 4
//Radix Sorting

#include "utils.h"
#include <thrust/host_vector.h>
#include <cstddef>

/* Red Eye Removal
   ===============
   
   For this assignment we are implementing red eye removal.  This is
   accomplished by first creating a score for every pixel that tells us how
   likely it is to be a red eye pixel.  We have already done this for you - you
   are receiving the scores and need to sort them in ascending order so that we
   know which pixels to alter to remove the red eye.

   Note: ascending order == smallest to largest

   Each score is associated with a position, when you sort the scores, you must
   also move the positions accordingly.

   Implementing Parallel Radix Sort with CUDA
   ==========================================

   The basic idea is to construct a histogram on each pass of how many of each
   "digit" there are.   
   
   Then we scan this histogram so that we know where to put
   the output of each digit.  For example, the first 1 must come after all the
   0s so we have to know how many 0s there are to be able to start moving 1s
   into the correct position.

   1) Histogram of the number of occurrences of each digit
   2) Exclusive Prefix Sum of Histogram
   3) Determine relative offset of each digit
        For example [0 0 1 1 0 0 1]
                ->  [0 1 0 1 2 3 2]
   4) Combine the results of steps 2 & 3 to determine the final
      output location for each element and move it there

   LSB Radix sort is an out-of-place sort and you will need to ping-pong values
   between the input and output buffers we have provided.  Make sure the final
   sorted results end up in the output buffer!  Hint: You may need to do a copy
   at the end.

 */

// =================================================================================================
__global__ void generatePredicate(const int *d_in, int *d_predicate, 
                                  const int size, const int digit)
{
  int myId = threadIdx.x + blockDim.x * blockIdx.x;
  int myItem = d_in[myId];
  int val = myItem>>digit;
  
  d_predicate[myId] = (val & 1);

}

// =================================================================================================
// Step efficient scan
__global__ void hillis_steele_algo(unsigned int* d_out, const int* d_in, size_t size) 
{
	extern __shared__ unsigned int temp[];
   
   int tid = threadIdx.x;
   
   int pout = 0, pin=1;
   
   temp[tid] = tid>0? d_in[tid-1]:0; //exclusive scan
	__syncthreads();

	//double buffered
	for (int offset = 1; offset < size; offset <<= 1) {
		pout = 1 - pout;
		pin = 1 - pout;
      if (tid >= offset) 
        temp[size*pout + tid] = temp[size*pin + tid]+temp[size*pin + tid - offset];
      else 
        temp[size*pout + tid] = temp[size*pin + tid];
		__syncthreads();
	}
	d_out[tid] = temp[pout*size + tid];
}


int hillis_steele_scan(unsigned int *d_scan, unsigned int *d_predicate, size_t numElems, const int BLOCK_SIZE)
{
  int threads = BLOCK_SIZE;
  int size = numElems;
  
  dim3 block(threads, 1, 1);
  dim3 grid(size/threads, 1, 1);  
  
  hillis_steele_algo <<<grid, block, 2 * BLOCK_SIZE*sizeof(unsigned int)>>>
                    (d_scan, d_predicate, numElems);
	hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
  
  int *h_sum;
  hipMemcpy(*h_sum, *d_scan[numElems-1], sizeof(unsigned int), hipMemcpyDeviceToHost);



  return *h_sum;
}




// =================================================================================================
void your_sort(unsigned int* const d_inputVals,
               unsigned int* const d_inputPos,
               unsigned int* const d_outputVals,
               unsigned int* const d_outputPos,
               const size_t numElems)
{ 
  //TODO
  //PUT YOUR SORT HERE
  
  const int ARRAY_SIZE = numElems;
  
  const int maxThreadPerBlock = 1024;
  const int threads = maxThreadPerBlock;
  const int size = numElems;
  const int SIZE_BYTES = size * sizeof(unsigned int);
  
  const int BIN_COUNT = 2;
  const int BIN_BYTES = BIN_COUNT * sizeof(unsigned int);


    
  unsigned int *h_histogram[2];
  h_histogram[0]=0,h_histogram[1]=0;

  unsigned int *d_histogram;
  unsigned int *d_predicate;
  unsigned int *d_scan_0;
  unsigned int *d_scan_1;

  checkCudaErrors(hipMalloc(&d_histogram, BIN_BYTES));
  checkCudaErrors(hipMalloc(&d_predicate, SIZE_BYTES));
  checkCudaErrors(hipMalloc(&d_scan_0, SIZE_BYTES));
  checkCudaErrors(hipMalloc(&d_scan_1, SIZE_BYTES));
  
  
  dim3 block(threads, 1, 1);
  dim3 grid(size/threads, 1, 1);  
  
  for (size_t d = 0; d< sizeof(unsigned int); ++d)
  {
    
    
    // fill the predicate for COMPACT operation.
    // we alternate btw input and output
    if (d%2==0)
      generatePredicate<<<grid,block>>>(d_inputVals, d_predicate, size, d);
    else if (d%2==1)
      generatePredicate<<<grid,block>>>(d_outputVals, d_predicate, size, d);      
    hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());    

    // histogram to find number of 0s.
    
    // scan to find the position of 0s and 1s.
    unsigned int nOnes = hillis_steele_scan(unsigned int* d_out, const int* d_in, int size);




    // Create the histogram
    simple_histo<<<grid, block>>>(d_inputVals, val, nZeros);
       
    for (size_t i = 0; i< numElems; ++i)
    {
      Radix_sort<<<grid, block>>>(d_inputVals, d_inputPos, val, nZeros);
    }


  }


}
