
/*
babak poursartip
adding two vectors

05/14/2020
*/


#include <hip/hip_runtime.h>
#include <stdio.h>

__global__
void add_vec( float *d_vec1, float *d_vec2, float *d_out, int vec_size)
{
  int tidx = threadIdx.x;
  int tidy = threadIdx.y;
  int tidz = threadIdx.z;

  int bidx = blockIdx.x;
  int bidy = blockIdx.y;
  int bidz = blockIdx.z;

  int gridx = gridDim.x;
  int gridy = gridDim.y;
  int gridz = gridDim.z;

  int blx   = blockDim.x;
  int bly   = blockDim.y;
  int blz   = blockDim.z;

  printf(" I am: %d  %d  %d  %d %d  %d %d  %d %d  %d %d  %d\n", tidx, tidy, tidz, bidx, bidy, bidz, gridx, gridy, gridz, blx, bly, blz);

  int chunk = vec_size/gridx/blx;

  int id_st = bidx*chunk*blx+tidx*chunk;
  int id_en = bidx*chunk*blx+(tidx+1)*chunk;

  printf("here: %d %d %d %d  \n", tidx, bidx, id_st, id_en  );

    for (int ii =id_st; ii < id_en; ++ii){
      d_out[ii]  = d_vec1[ii] + d_vec2[ii];
    }
  //printf(" yes %f  %f  %f \n",d_out[tidx],d_vec1[tidx],d_vec2[tidx]);

}

int main(){

printf(" code starts ... \n");

float *h_vec1, *h_vec2,*h_out;

int vec_size = 1024;
int vec_size_byte = vec_size * sizeof(float);

h_vec1 = (float *) calloc( vec_size,sizeof(float));
h_vec2 = (float *) calloc( vec_size,sizeof(float));
h_out  = (float *) calloc( vec_size,sizeof(float));

  for (int ii = 0; ii<vec_size; ++ii){
    h_vec1[ii]  = 0.1f*(float)ii;
    h_vec2[ii]  = 0.2f*(float)ii;
    h_out[ii]  = 0.222f;
  }  

// define pointers in the gpu
float *d_vec1, *d_vec2,*d_out;

// alloc memory on the gpu
hipMalloc((void **) &d_vec1, vec_size_byte);
hipMalloc((void **) &d_vec2, vec_size_byte);
hipMalloc((void **) &d_out, vec_size_byte);

// copy arrays to the gpu memory
hipMemcpy(d_vec1,h_vec1,vec_size_byte,hipMemcpyHostToDevice);
hipMemcpy(d_vec2,h_vec2,vec_size_byte,hipMemcpyHostToDevice);

// lauch kerner
int blc = 4;
dim3 blocks(blc,1,1);
int thrd = 16;
dim3 threads(thrd,1,1);

add_vec<<<blocks,threads>>>(d_vec1, d_vec2, d_out, vec_size);

// transfer the output to host
hipMemcpy(h_out,d_out,vec_size_byte,hipMemcpyDeviceToHost);

// output
printf("\n");
  for (int ii = 0; ii<vec_size; ++ii){
    printf(" %d %f + %f  = %f \n",ii,h_vec1[ii],h_vec2[ii], h_out[ii]);
  }  

printf(" code ends. \n");

}
