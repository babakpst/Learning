
/*
Babak Poursartip
05/14/2020

based on Intro to parallel programming cuda

remarks:

- data on the host, starts with h (h_in)
- data on the device, starts with d (d_in)


*/


#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void cube(float*d_out, float * d_in)
{
  int idx = threadIdx.x;
  float f = d_in[idx];
  d_out[idx] = f*f*f;
}


int main(){

printf(" code starts ... \n");

const int ARRAY_SIZE = 96;
const int ARRAY_BYTES = ARRAY_SIZE * sizeof(float);

// fill the input array on the host
float h_in[ARRAY_SIZE];
for (int ii = 0; ii < ARRAY_SIZE; ii++)
  h_in[ii] = float(ii);

float h_out[ARRAY_SIZE];

// declare gpu memory pointers
float *d_in;
float *d_out;

// allocate gpu memory
hipMalloc((void **) &d_in, ARRAY_BYTES);
hipMalloc((void **) &d_out, ARRAY_BYTES);

// transfer the array to the GPU
hipMemcpy(d_in, h_in, ARRAY_BYTES, hipMemcpyHostToDevice);

// launch the kernel
cube<<<1, ARRAY_SIZE>>>(d_out, d_in);

// copy back the results in the CPU
hipMemcpy(h_out,d_out, ARRAY_BYTES,hipMemcpyDeviceToHost);

// print out the results
for (int ii = 0; ii < ARRAY_SIZE; ++ii)
{
  printf(" %f", h_out[ii]);
  printf( ( (ii%5) !=4 ) ? "\t":"\n" );
}

// free gpu
hipFree(d_in);
hipFree(d_out);

return 0;
}

