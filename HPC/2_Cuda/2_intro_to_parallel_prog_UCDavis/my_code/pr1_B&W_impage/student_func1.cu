#include "hip/hip_runtime.h"
// Homework 1
// Color to Greyscale Conversion

//A common way to represent color images is known as RGBA - the color
//is specified by how much Red, Grean and Blue is in it.
//The 'A' stands for Alpha and is used for transparency, it will be
//ignored in this homework.

//Each channel Red, Blue, Green and Alpha is represented by one byte.
//Since we are using one byte for each color there are 256 different
//possible values for each color.  This means we use 4 bytes per pixel.

//Greyscale images are represented by a single intensity value per pixel
//which is one byte in size.

//To convert an image from color to grayscale one simple method is to
//set the intensity to the average of the RGB channels.  But we will
//use a more sophisticated method that takes into account how the eye 
//perceives color and weights the channels unequally.

//The eye responds most strongly to green followed by red and then blue.
//The NTSC (National Television System Committee) recommends the following
//formula for color to greyscale conversion:

//I = .299f * R + .587f * G + .114f * B

//Notice the trailing f's on the numbers which indicate that they are 
//single precision floating point constants and not double precision
//constants.

//You should fill in the kernel as well as set the block and grid sizes
//so that the entire image is processed.

#include "utils.h"

__global__
void rgba_to_greyscale(const uchar4* const rgbaImage,
                       unsigned char* const greyImage,
                       int numRows, int numCols)
{
  //TODO
  //Fill in the kernel to convert from color to greyscale
  //the mapping from components of a uchar4 to RGBA is:
  // .x -> R ; .y -> G ; .z -> B ; .w -> A
  //
  //The output (greyImage) at each pixel should be the result of
  //applying the formula: output = .299f * R + .587f * G + .114f * B;
  //Note: We will be ignoring the alpha channel for this conversion

  //First create a mapping from the 2D block and grid locations
  //to an absolute 2D location in the image, then use that to
  //calculate a 1D offset

  
  size_t threadsPerBlock       = blockDim.x * blockDim.y * blockDim.z;
  size_t threadPositionInBlock = threadIdx.x +
                              blockDim.x * threadIdx.y +
                              blockDim.x * blockDim.y * threadIdx.z;

  size_t blockPositionInGrid   = blockIdx.x +
                              gridDim.x * blockIdx.y +
                              gridDim.x * gridDim.y * blockIdx.z;

  size_t tid = blockPositionInGrid * threadsPerBlock + threadPositionInBlock;
  
  /*
  int tid = threadIdx.x;
  int offset = blockIdx.x * blockDim.x;
  int gid = tid + offset;
  tid = gid;
  */
  /*  
  printf("gridDim.x: %d, gridDim.y: %d, gridDim.z: %d, blockDim.x: %d, blockDim.y: %d, blockDim.z: %d, blockIdx.x: %d, blockIdx.y: %d, blockIdx.z: %d, threadIdx.x: %d, threadIdx.y: %d, threadIdx.z: %d, tid: %d \n",
          gridDim.x, gridDim.y, gridDim.z, 
          blockDim.x, blockDim.y, blockDim.z, 
          blockIdx.x, blockIdx.y, blockIdx.z, 
          threadIdx.x, threadIdx.y, threadIdx.z, 
          tid);
  */
  if (tid<numCols*numRows){
  uchar4 rgba = rgbaImage[tid];
  float channelSum = .299f * rgba.x + .587f * rgba.y + .114f * rgba.z;
  greyImage[tid] = channelSum;
  }
  
}

void your_rgba_to_greyscale(const uchar4 * const h_rgbaImage, uchar4 * const d_rgbaImage,
                            unsigned char* const d_greyImage, size_t numRows, size_t numCols)
{
  //You must fill in the correct sizes for the blockSize and gridSize
  //currently only one block with one thread is being launched
  const dim3 blockSize(128, 1, 1);  //TODO
  const dim3 gridSize(numRows*numCols/128+1, 1, 1);  //TODO
  //const dim3 blockSize(numCols, 1, 1);  //TODO
  //const dim3 gridSize(numRows, 1, 1);  //TODO
  rgba_to_greyscale<<<gridSize, blockSize>>>(d_rgbaImage, d_greyImage, numRows, numCols);
  
  hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

}
