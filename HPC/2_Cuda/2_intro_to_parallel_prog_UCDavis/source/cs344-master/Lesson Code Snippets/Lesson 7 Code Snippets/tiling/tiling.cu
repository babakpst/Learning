#include "hip/hip_runtime.h"
#include <stdio.h>
#include "gputimer.h"
#include "utils.h"

const int BLOCKSIZE	= 128;
const int NUMBLOCKS = 1000;					// set this to 1 or 2 for debugging
const int N 		= BLOCKSIZE*NUMBLOCKS;

/* 
 * TODO: modify the foo and bar kernels to use tiling: 
 * 		 - copy the input data to shared memory
 *		 - perform the computation there
 *	     - copy the result back to global memory
 *		 - assume thread blocks of 128 threads
 *		 - handle intra-block boundaries correctly
 * You can ignore boundary conditions (we ignore the first 2 and last 2 elements)
 */
__global__ void foo(float out[], float A[], float B[], float C[], float D[], float E[]){

	int i = threadIdx.x + blockIdx.x*blockDim.x; 
	
	out[i] = (A[i] + B[i] + C[i] + D[i] + E[i]) / 5.0f;
}

__global__ void bar(float out[], float in[]) 
{
	int i = threadIdx.x + blockIdx.x*blockDim.x; 

	out[i] = (in[i-2] + in[i-1] + in[i] + in[i+1] + in[i+2]) / 5.0f;
}

void cpuFoo(float out[], float A[], float B[], float C[], float D[], float E[])
{
	for (int i=0; i<N; i++)
	{
		out[i] = (A[i] + B[i] + C[i] + D[i] + E[i]) / 5.0f;
	}
}

void cpuBar(float out[], float in[])
{
	// ignore the boundaries
	for (int i=2; i<N-2; i++)
	{
		out[i] = (in[i-2] + in[i-1] + in[i] + in[i+1] + in[i+2]) / 5.0f;
	}
}

int main(int argc, char **argv)
{
	// declare and fill input arrays for foo() and bar()
	float fooA[N], fooB[N], fooC[N], fooD[N], fooE[N], barIn[N];
	for (int i=0; i<N; i++) 
	{
		fooA[i] = i; 
		fooB[i] = i+1;
		fooC[i] = i+2;
		fooD[i] = i+3;
		fooE[i] = i+4;
		barIn[i] = 2*i; 
	}
	// device arrays
	int numBytes = N * sizeof(float);
	float *d_fooA;	 	hipMalloc(&d_fooA, numBytes);
	float *d_fooB; 		hipMalloc(&d_fooB, numBytes);
	float *d_fooC;	 	hipMalloc(&d_fooC, numBytes);
	float *d_fooD; 		hipMalloc(&d_fooD, numBytes);
	float *d_fooE; 		hipMalloc(&d_fooE, numBytes);
	float *d_barIn; 	hipMalloc(&d_barIn, numBytes);
	hipMemcpy(d_fooA, fooA, numBytes, hipMemcpyHostToDevice);
	hipMemcpy(d_fooB, fooB, numBytes, hipMemcpyHostToDevice);
	hipMemcpy(d_fooC, fooC, numBytes, hipMemcpyHostToDevice);
	hipMemcpy(d_fooD, fooD, numBytes, hipMemcpyHostToDevice);
	hipMemcpy(d_fooE, fooE, numBytes, hipMemcpyHostToDevice);
	hipMemcpy(d_barIn, barIn, numBytes, hipMemcpyHostToDevice);	

	// output arrays for host and device
	float fooOut[N], barOut[N], *d_fooOut, *d_barOut;
	hipMalloc(&d_fooOut, numBytes);
	hipMalloc(&d_barOut, numBytes);

	// declare and compute reference solutions
	float ref_fooOut[N], ref_barOut[N]; 
	cpuFoo(ref_fooOut, fooA, fooB, fooC, fooD, fooE);
	cpuBar(ref_barOut, barIn);

	// launch and time foo and bar
	GpuTimer fooTimer, barTimer;
	fooTimer.Start();
	foo<<<N/BLOCKSIZE, BLOCKSIZE>>>(d_fooOut, d_fooA, d_fooB, d_fooC, d_fooD, d_fooE);
	fooTimer.Stop();
	
	barTimer.Start();
	bar<<<N/BLOCKSIZE, BLOCKSIZE>>>(d_barOut, d_barIn);
	barTimer.Stop();

	hipMemcpy(fooOut, d_fooOut, numBytes, hipMemcpyDeviceToHost);
	hipMemcpy(barOut, d_barOut, numBytes, hipMemcpyDeviceToHost);
	printf("foo<<<>>>(): %g ms elapsed. Verifying solution...", fooTimer.Elapsed());
	compareArrays(ref_fooOut, fooOut, N);
	printf("bar<<<>>>(): %g ms elapsed. Verifying solution...", barTimer.Elapsed());
	compareArrays(ref_barOut, barOut, N);
}
