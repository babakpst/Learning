#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "compare.h"
#include "gputimer.h"

// Subpart b:
// Compute capability 2.0+ GPUs have support for 3 per-warp instructions.
// Namely, these instructions are:
//
// int __popc(int x) Population Count: Returns the number of bits that are set
// to 1 in the 32-bit integer x.
//
// int __clz(int x) Count Leading Zeros: Returns the number of consecutive zero
// bits beginning at the most significant bit of the 32-bit integer x.
//
// int __ballot(int p) Returns a 32-bit integer in which bit k is set if and only
// if the predicate p provided by the thread in lane k of the warp is non-zero.

__device__ unsigned int warp_reduce(unsigned int p, volatile unsigned int * s) {
    // Assumes values in 'p' are either 1 or 0
    // Should not use 's'
    // Sums p across warp, returning the result.
    // You can do this without using the character '+' in your code at all
    //
    // TODO: Fill in the rest of this function
    //
}

__global__ void reduce(unsigned int * d_out_warp, 
                       const unsigned int * d_in)
{
    extern __shared__ unsigned int s[];
    int t = threadIdx.x;
    int p = d_in[t];

    unsigned int wr = warp_reduce(p, s);
    if (t == 0)
    {
        *d_out_warp = wr;
    }
}

int main(int argc, char **argv)
{
    const int ARRAY_SIZE = 32;
    const int ARRAY_BYTES = ARRAY_SIZE * sizeof(unsigned int);

    // generate the input array on the host
    unsigned int h_in[ARRAY_SIZE];
    unsigned int sum = 0;
    for(int i = 0; i < ARRAY_SIZE; i++) {
        // generate random float in [0, 1]
        h_in[i] = (float)random()/(float)RAND_MAX > 0.5f ? 1 : 0;
        sum += h_in[i];
    }

    // declare GPU memory pointers
    unsigned int * d_in, * d_out_warp;

    // allocate GPU memory
    hipMalloc((void **) &d_in, ARRAY_BYTES);
    hipMalloc((void **) &d_out_warp, sizeof(unsigned int));

    // transfer the input array to the GPU
    hipMemcpy(d_in, h_in, ARRAY_BYTES, hipMemcpyHostToDevice); 

    GpuTimer timer;
    timer.Start();
    // launch the kernel
    reduce<<<1, ARRAY_SIZE, ARRAY_SIZE * sizeof(unsigned int)>>>
        (d_out_warp, d_in);
    timer.Stop();

    printf("Your code executed in %g ms\n", timer.Elapsed());  

    unsigned int h_out_warp;
    // copy back the sum from GPU
    hipMemcpy(&h_out_warp, d_out_warp, sizeof(unsigned int), 
               hipMemcpyDeviceToHost);

    // compare your result against the expected reduce sum
    compare(h_out_warp, sum);

    // free GPU memory allocation
    hipFree(d_in);
    hipFree(d_out_warp);
        
}

