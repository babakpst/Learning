#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "compare.h"
#include "gputimer.h"

// Subpart A:
// Write step 1 as a kernel that operates on threads 0--31.
// Assume that the input flags are 0 for false and 1 for true and are stored
// in a local per-thread register called p (for predicate).
//
// You have access to 31 words of shared memory s[0:31], with s[0]
// corresponding to thread 0 and s[31] corresponding to thread 31.
// You may change the values of s[0:31]. Put the return sum in s[0].
// Your code should execute no more than 5 warp-wide addition operations.

__device__ unsigned int shared_reduce(unsigned int p, volatile unsigned int * s) {
    // Assumes values in 'p' are either 1 or 0
    // Assumes s[0:31] are allocated
    // Sums p across warp, returning the result. Suggest you put
    // result in s[0] and return it
    // You may change any value in s
    // You should execute no more than 5 + operations (if you're doing
    // 31, you're doing it wrong)
    //
    // TODO: Fill in the rest of this function

    return s[0];
}

__global__ void reduce(unsigned int * d_out_shared,
                       const unsigned int * d_in)
{
    extern __shared__ unsigned int s[];
    int t = threadIdx.x;
    int p = d_in[t];
    unsigned int sr = shared_reduce(p, s);
    if (t == 0)
    {
        *d_out_shared = sr;
    }
}

int main(int argc, char **argv)
{
    const int ARRAY_SIZE = 32;
    const int ARRAY_BYTES = ARRAY_SIZE * sizeof(unsigned int);

    // generate the input array on the host
    unsigned int h_in[ARRAY_SIZE];
    unsigned int sum = 0;
    for(int i = 0; i < ARRAY_SIZE; i++) {
        // generate random float in [0, 1]
        h_in[i] = (float)random()/(float)RAND_MAX > 0.5f ? 1 : 0;
        sum += h_in[i];
    }

    // declare GPU memory pointers
    unsigned int * d_in, * d_out_shared;

    // allocate GPU memory
    hipMalloc((void **) &d_in, ARRAY_BYTES);
    hipMalloc((void **) &d_out_shared, sizeof(unsigned int));

    // transfer the input array to the GPU
    hipMemcpy(d_in, h_in, ARRAY_BYTES, hipMemcpyHostToDevice); 

    GpuTimer timer;
    timer.Start();
    // launch the kernel
    reduce<<<1, ARRAY_SIZE, ARRAY_SIZE * sizeof(unsigned int)>>>
        (d_out_shared, d_in);
    timer.Stop();

    printf("Your code executed in %g ms\n", timer.Elapsed());

    unsigned int h_out_shared;
    // copy back the sum from GPU
    hipMemcpy(&h_out_shared, d_out_shared, sizeof(unsigned int), 
               hipMemcpyDeviceToHost);
    
    compare(h_out_shared, sum);

    // free GPU memory allocation
    hipFree(d_in);
    hipFree(d_out_shared);
}

