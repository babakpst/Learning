
#include <hip/hip_runtime.h>
// ====================
__global__
void deviceKernel(int *a, int N)
{
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  int stride = blockDim.x * gridDim.x;

  for (int i = idx; i < N; i += stride)
  {
    a[i] = 1;
  }
}

// ====================
void hostFunction(int *a, int N)
{
  for (int i = 0; i < N; ++i)
  {
    a[i] = 1;
  }
}

// ====================
int main()
{

  int N = 2<<24;
  size_t size = N * sizeof(int);
  int *a;
  hipMallocManaged(&a, size);

    /*
   * Conduct experiments to learn more about the behavior of
   * `cudaMallocManaged`.
   *
   *1- What happens when unified memory is accessed only by the GPU?
   *2- What happens when unified memory is accessed only by the CPU?
   *3- What happens when unified memory is accessed first by the GPU then the CPU?
   *4- What happens when unified memory is accessed first by the CPU then the GPU?
   *
   * Hypothesize about UM behavior, page faulting specificially, before each
   * experiment, and then verify by running `nsys`.
   */

  
  //1  
  //deviceKernel<<<120,128>>>(a,N);
  //cudaDeviceSynchronize();
  
  //2
  //hostFunction(a,N);

  //3
  hostFunction(a,N);
  deviceKernel<<<120,128>>>(a,N);
  hipDeviceSynchronize();
  
  //4
  deviceKernel<<<12,512>>>(a,N);
  hipDeviceSynchronize();
  hostFunction(a,N);

  hipFree(a);
}

