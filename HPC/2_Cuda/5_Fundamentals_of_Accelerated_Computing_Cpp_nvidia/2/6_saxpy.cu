#include "hip/hip_runtime.h"
#include <stdio.h>

#define N 2048 * 2048 // Number of elements in each vector

/*
 * Optimize this already-accelerated codebase. Work iteratively,
 * and use nsys to support your work.
 *
 * Aim to profile `saxpy` (without modifying `N`) running under
 * 20us.
 *
 * Some bugs have been placed in this codebase for your edification.
 */

// ============================
__global__ void saxpy(int * a, int * b, int * c)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x *gridDim.x;

    for (int i= tid; i< N; i+=stride)
        c[i] = 2 * a[i] + b[i];
}
// ============================
__global__ void init(int * a, int val)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x *gridDim.x;

    for (int i= tid; i< N; i+=stride)
        a[i] = val;
}


int main()
{
    int *a, *b, *c;

    int size = N * sizeof (int); // The total number of bytes per vector

    hipMallocManaged(&a, size);
    hipMallocManaged(&b, size);
    hipMallocManaged(&c, size);

    int deviceId;
    hipGetDevice(&deviceId);
    int numberOfSMs;
    hipDeviceGetAttribute(&numberOfSMs, hipDeviceAttributeMultiprocessorCount, deviceId);

    int threads_per_block = 256;
    //int number_of_blocks = (N / threads_per_block) + 1;
    int number_of_blocks = 32*numberOfSMs;


    hipMemPrefetchAsync(a, size, deviceId);
    hipMemPrefetchAsync(b, size, deviceId);
    hipMemPrefetchAsync(c, size, deviceId);

    init<<<number_of_blocks, threads_per_block>>>( a, 2);
    init<<<number_of_blocks, threads_per_block>>>( b, 1);
    init<<<number_of_blocks, threads_per_block>>>( c, 0);
    hipDeviceSynchronize();
    saxpy <<< number_of_blocks, threads_per_block >>> ( a, b, c );
    hipDeviceSynchronize();
    
    // Print out the first and last 5 values of c for a quality check
    for( int i = 0; i < 5; ++i )
        printf("c[%d] = %d, ", i, c[i]);
    printf ("\n");
    for( int i = N-5; i < N; ++i )
        printf("c[%d] = %d, ", i, c[i]);
    printf ("\n");

    hipFree( a ); hipFree( b ); hipFree( c );
}

