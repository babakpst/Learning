
#include <hip/hip_runtime.h>
#include <stdio.h>


// ==========================================
void init(int *a, int N)
{
  int i;
  for (i = 0; i < N; ++i)
  {
    a[i] = i;
  }
}


// ==========================================
__global__
void doubleElements(int *a, int N)
{

  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = gridDim.x * blockDim.x;

  for (int i = idx; i < N + stride; i += stride)
  {
    a[i] *= 2;
  }
}

// ==========================================
bool checkElementsAreDoubled(int *a, int N)
{
  int i;
  for (i = 0; i < N; ++i)
  {
    if (a[i] != i*2) return false;
  }
  return true;
}


// ==========================================
int main()
{
  /*
   * Add error handling to this source code to learn what errors
   * exist, and then correct them. Googling error messages may be
   * of service if actions for resolving them are not clear to you.
   */

  int N = 10000;
  int *a;

  size_t size = N * sizeof(int);
  hipError_t err;
  err = hipMallocManaged(&a, size);

  if (err != hipSuccess)
    printf("Error allocating array: %s\n", hipGetErrorString(err));
    
  init(a, N);

  //size_t threads_per_block = 2048; // error, max is 1024
  size_t threads_per_block = 1024; 
  size_t number_of_blocks = 32;

  doubleElements<<<number_of_blocks, threads_per_block>>>(a, N);
  err = hipGetLastError();
  if (err != hipSuccess)
    printf("Error doubling: %s\n", hipGetErrorString(err));
  
  err = hipDeviceSynchronize();
  if (err != hipSuccess)
    printf("Error synchronize: %s\n", hipGetErrorString(err));


  bool areDoubled = checkElementsAreDoubled(a, N);
  printf("All elements were doubled? %s\n", areDoubled ? "TRUE" : "FALSE");

  hipFree(a);
}

