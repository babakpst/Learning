
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N  64

// ====================================================
__global__ void matrixMulGPU( int * a, int * b, int * c )
{
  /*
   * Build out this kernel.
   */
  int row = blockIdx.x * blockDim.x + threadIdx.x;
  int col = blockIdx.y * blockDim.y + threadIdx.y;
  int val = 0;
  if (row < N && col < N){
      val = 0;
      for ( int k = 0; k < N; ++k )
        val += a[row * N + k] * b[k * N + col];
      c[row * N + col] = val;
  }

}


// ====================================================
/*
 * This CPU function already works, and will run to create a solution matrix
 * against which to verify your work building out the matrixMulGPU kernel.
 */

void matrixMulCPU( int * a, int * b, int * c )
{
  int val = 0;

  for( int row = 0; row < N; ++row )
    for( int col = 0; col < N; ++col )
    {
      val = 0;
      for ( int k = 0; k < N; ++k )
        val += a[row * N + k] * b[k * N + col];
      c[row * N + col] = val;
    }
}


// ====================================================
int main()
{
  int *a, *b, *c_cpu, *c_gpu; // Allocate a solution matrix for both the CPU and the GPU operations

  int size = N * N * sizeof (int); // Number of bytes of an N x N matrix

  // Allocate memory
  hipMallocManaged (&a, size);
  hipMallocManaged (&b, size);
  hipMallocManaged (&c_cpu, size);
  hipMallocManaged (&c_gpu, size);

  // Initialize memory; create 2D matrices
  for( int row = 0; row < N; ++row )
    for( int col = 0; col < N; ++col )
    {
      a[row*N + col] = row;
      b[row*N + col] = col+2;
      c_cpu[row*N + col] = 0;
      c_gpu[row*N + col] = 0;
    }

  /*
   * Assign `threads_per_block` and `number_of_blocks` 2D values
   * that can be used in matrixMulGPU above.
   */
  int threads = 8;
  dim3 threads_per_block(threads,threads);
  dim3 number_of_blocks(8,8);
  //dim3 number_of_blocks((N+threads-1)/threads,(N+threads-1)/threads);

  matrixMulGPU <<< number_of_blocks, threads_per_block >>> ( a, b, c_gpu );

  hipDeviceSynchronize();

  // Call the CPU version to check our work
  matrixMulCPU( a, b, c_cpu );

  // Compare the two answers to make sure they are equal
  bool error = false;
  for( int row = 0; row < N && !error; ++row )
    for( int col = 0; col < N && !error; ++col )
      if (c_cpu[row * N + col] != c_gpu[row * N + col])
      {
        printf("FOUND ERROR at c[%d][%d]\n", row, col);
        error = true;
        break;
      }
  if (!error)
    printf("Success!\n");

  // Free all our allocated memory
  hipFree(a); hipFree(b);
  hipFree( c_cpu ); hipFree( c_gpu );
}

