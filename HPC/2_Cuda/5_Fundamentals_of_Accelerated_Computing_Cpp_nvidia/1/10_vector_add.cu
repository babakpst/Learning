

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <assert.h>

// ======================================================
inline hipError_t checkCuda(hipError_t result)
{
  if (result != hipSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
    assert(result == hipSuccess);
  }
  return result;
}

// ======================================================
void initWith(float num, float *a, int N)
{
  for(int i = 0; i < N; ++i)
  {
    a[i] = num;
  }
}

// ======================================================
__global__
void addVectorsInto(float *result, float *a, float *b, int N)
{
  int ind = threadIdx.x + blockDim.x * blockIdx.x; // stride
  int gridSize = blockDim.x * gridDim.x;
  
  for(int i = ind; i < N; i+=gridSize)
  {
    result[i] = a[i] + b[i];
  }
}

// ======================================================
void checkElementsAre(float target, float *array, int N)
{
  for(int i = 0; i < N; i++)
  {
    if(array[i] != target)
    {
      printf("FAIL: array[%d] - %0.0f does not equal %0.0f\n", i, array[i], target);
      exit(1);
    }
  }
  printf("SUCCESS! All values added correctly.\n");
}

// ======================================================
int main()
{
  const int N = 2<<20;
  size_t size = N * sizeof(float);

  float *a;
  float *b;
  float *c;

  //a = (float *)malloc(size);
  //b = (float *)malloc(size);
  //c = (float *)malloc(size);
  
  checkCuda(hipMallocManaged(&a, size)); 
  checkCuda(hipMallocManaged(&b, size)); 
  checkCuda(hipMallocManaged(&c, size)); 
  

  initWith(3, a, N);
  initWith(4, b, N);
  initWith(0, c, N);

  int threadsPerBlock = 1024;
  dim3 grid((N+threadsPerBlock-1)/threadsPerBlock);
  dim3 block(threadsPerBlock);
  addVectorsInto<<<grid, block>>>(c, a, b, N);
  checkCuda(hipGetLastError());
  checkCuda(hipDeviceSynchronize());

  checkElementsAre(7, c, N);

  hipFree(a);
  hipFree(b);
  hipFree(c);
}

