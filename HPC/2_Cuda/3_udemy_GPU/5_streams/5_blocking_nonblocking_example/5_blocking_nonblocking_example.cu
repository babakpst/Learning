#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include "hip/hip_runtime.h"
#include ""
#include "common.h"
#include "cuda_common.cuh"

__global__ void blocking_nonblocking_test1()
{
	int gid = blockDim.x * blockIdx.x + threadIdx.x;
	if (gid == 0)
	{
		for (size_t i = 0; i < 10000; i++)
		{
			printf("kernel 1 \n");
		}
	}
}

int main(int argc, char ** argv)
{
	int size = 1 << 15;
	
	dim3 block(128);
	dim3 grid(size / block.x);

	//  1 --
	hipStream_t stm1,stm2,stm3;
	
	// gpuErrchk(hipStreamCreate(&stm1));  // blocking stream
	// gpuErrchk(hipStreamCreate(&stm2));  // blocking stream
	// gpuErrchk(hipStreamCreate(&stm3));  // blocking stream
	
	
	// // even though streams are blocking, but there is no null stream, so all three kernels start in parallel.
	// blocking_nonblocking_test1 <<<grid, block, 0 , stm1 >>> ();
	// blocking_nonblocking_test1 <<<grid, block, 0 , stm2  >>> ();
	// // blocking_nonblocking_test1 <<<grid, block >>> ();
	// blocking_nonblocking_test1 <<<grid, block, 0, stm3 >>> ();

	// //  2 -- blocking
	// gpuErrchk(hipStreamCreate(&stm1));
	// gpuErrchk(hipStreamCreate(&stm2));
	// gpuErrchk(hipStreamCreate(&stm3));
		
	// // there is a null stream. CUDA waits on all operations previously issued to all blocking streams before starting 
	// // that operation. So, here, the three kernels run in serial. 
	// blocking_nonblocking_test1 <<<grid, block, 0 , stm1 >>> ();
	// // blocking_nonblocking_test1 <<<grid, block, 0 , stm2  >>> ();
	// blocking_nonblocking_test1 <<<grid, block >>> ();
	// blocking_nonblocking_test1 <<<grid, block, 0, stm3 >>> ();
	
	// gpuErrchk(hipStreamDestroy(stm1));
	// gpuErrchk(hipStreamDestroy(stm2));
	// gpuErrchk(hipStreamDestroy(stm3));

	
	// //  3 -- to void blocking nature of null stream, we can create non-blocking streams with Flags.
	// gpuErrchk(hipStreamCreateWithFlags(&stm1, hipStreamNonBlocking));
	// gpuErrchk(hipStreamCreate(&stm2));
	// gpuErrchk(hipStreamCreateWithFlags(&stm3,hipStreamNonBlocking));

	// blocking_nonblocking_test1 <<<grid, block, 0 , stm1 >>> ();
	// // blocking_nonblocking_test1 <<<grid, block, 0 , stm2  >>> ();
	// blocking_nonblocking_test1 <<<grid, block >>> ();
	// blocking_nonblocking_test1 <<<grid, block, 0, stm3 >>> ();

	// gpuErrchk(hipStreamDestroy(stm1));
	// gpuErrchk(hipStreamDestroy(stm2));
	// gpuErrchk(hipStreamDestroy(stm3));

	
	// //  4 -- blockin 
	gpuErrchk(hipStreamCreateWithFlags(&stm1, hipStreamDefault));
	gpuErrchk(hipStreamCreate(&stm2));
	gpuErrchk(hipStreamCreateWithFlags(&stm3, hipStreamDefault));

	blocking_nonblocking_test1 <<<grid, block, 0 , stm1 >>> ();
	// blocking_nonblocking_test1 <<<grid, block, 0 , stm2  >>> ();
	blocking_nonblocking_test1 <<<grid, block >>> ();
	blocking_nonblocking_test1 <<<grid, block, 0, stm3 >>> ();


	gpuErrchk(hipStreamDestroy(stm1));
	gpuErrchk(hipStreamDestroy(stm2));
	gpuErrchk(hipStreamDestroy(stm3));
	gpuErrchk(hipDeviceSynchronize());

	gpuErrchk(hipDeviceReset());
	return 0;
}