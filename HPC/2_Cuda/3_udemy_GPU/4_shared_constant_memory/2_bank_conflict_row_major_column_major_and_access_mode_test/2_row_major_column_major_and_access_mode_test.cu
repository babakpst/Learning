#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h> // why do we need this?
#include <time.h>   
#include "hip/hip_runtime.h"
#include ""
#include "common.h"


// profile
// sudo nvprof --metrics shared_load_transactions_per_request,shared_store_transactions_per_request ./bin/main

// we have to use these directive to allocate static shared memory
#define NX 32
#define NY 32

// =============================================================
// kernel to store in shared memore in row major format (no bank conflict), 
// but read in the col major format (bank conflict).
__global__ void storeRow_ReadCol(int *out)
{
  __shared__ int tile[NX][NY];  // row major format

  int idx = threadIdx.y * blockDim.x + threadIdx.x;

  // store/fill the array on shared memory
  // NOTICE the order of threads in the matrix
  tile[threadIdx.y][threadIdx.x] = idx; // store operation-row major
  // printf("number:  %s\n",tile[threadIdx.y][threadIdx.x]);
  
  __syncthreads();
  // printf("number:  %s\n", idx);
  
  // load from the shared memory and write in the global memory
  // bank conflict: all threads from a warp are loading from one banch.
  out[idx] = tile[threadIdx.x][threadIdx.y];
  // printf("number:  %s\n", out[idx]);
  // __syncthreads;
}

// =============================================================
// wirte in the shared memory in the col major order, 
// and read from the memory in the row major order.
__global__ void storeCol_ReadRow(int *out)
{
  __shared__ int tile[NX][NY];  // row major format

  int idx = threadIdx.y * blockDim.x + threadIdx.x;

  // store/fill the array on shared memory
  // NOTICE the order of threads in the matrix
  // all threads from warp are storing from one bank
  tile[threadIdx.x][threadIdx.y] = idx; // store operation-row major

  __syncthreads();

  // load from the shared memory and write in the global memory
  out[idx] = tile[threadIdx.y][threadIdx.x];
}


// =============================================================
// wirte in the shared memory in the row major order, 
// and read from the memory in the row major order.
__global__ void storeRow_ReadRow(int *out)
{
  __shared__ int tile[NX][NY];  // row major format

  int idx = threadIdx.y * blockDim.x + threadIdx.x;

  // store/fill the array on shared memory
  // NOTICE the order of threads in the matrix
  tile[threadIdx.y][threadIdx.x] = idx; // store operation-row major

  __syncthreads();

  // load from the shared memory and write in the global memory
  out[idx] = tile[threadIdx.y][threadIdx.x];
}


int main(int argc, char* argv[])
{
  int mem_config = 0; // 0: 32 bit bank width, 1: 64 bank width
  
  if (argc>1) mem_config = atoi(argv[1]);

  if (mem_config==1) // 64 bit bank width
    hipDeviceSetSharedMemConfig(hipSharedMemBankSizeEightByte);
  else if (mem_config == 0)  // 32 bit bank width
    hipDeviceSetSharedMemConfig(hipSharedMemBankSizeFourByte);

  hipSharedMemConfig pConfig;
  hipDeviceGetSharedMemConfig(&pConfig);
  printf("GPU bank setting %s\n", pConfig==1?"4-Byte width":"8-Byte width");

  int nx = 32;
  int ny = 32;

  bool iprintf = 0;

  if (argc > 2) iprintf = atoi(argv[2]);


  int byteSize = nx * ny * sizeof(int);

  dim3 block(nx,ny);
  dim3 grid(1,1);

  printf(" grid block: (%d, %d), thread block: (%d, %d)\n", block.x, block.y, grid.x, grid.y);

  int *ref = (int *)malloc(byteSize);
  
  int *d_C; 
  hipMalloc((int**)&d_C, byteSize);
  
  clock_t start, end;

  hipMemset(&d_C, 0, byteSize);
  start = clock();
  storeRow_ReadCol<<<grid, block>>>(d_C);
  hipDeviceSynchronize();
  end = clock();
  hipMemcpy(ref, d_C, byteSize, hipMemcpyDeviceToHost);
  if (iprintf) printArray(" store row - read col: ", ref, nx*ny, nx);
  printf(" execution time: %4.6f \n", (double)((double)(end - start) / CLOCKS_PER_SEC));

  hipMemset(&d_C, 0, byteSize);
  start = clock();
  storeCol_ReadRow<<<grid, block>>>(d_C);
  hipDeviceSynchronize();
  end = clock();
  hipMemcpy(ref, d_C, byteSize, hipMemcpyDeviceToHost);
  if (iprintf) printArray(" store col - read row: ", ref, nx*ny, nx);
  printf(" execution time: %4.6f \n", (double)((double)(end - start) / CLOCKS_PER_SEC));


  hipMemset(&d_C, 0, byteSize);
  start = clock();
  storeRow_ReadRow<<<grid, block>>>(d_C);
  hipDeviceSynchronize();
  end = clock();
  hipMemcpy(ref, d_C, byteSize, hipMemcpyDeviceToHost);
  if (iprintf) printArray(" store row - read row: ", ref, nx*ny, nx);
  printf(" execution time: %4.6f \n", (double)((double)(end - start) / CLOCKS_PER_SEC));


  hipFree(d_C);
  free(ref);

  hipDeviceReset();
  return 0;
}

