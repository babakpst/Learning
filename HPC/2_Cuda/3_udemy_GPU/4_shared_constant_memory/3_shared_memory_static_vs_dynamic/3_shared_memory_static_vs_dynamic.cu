#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h> // why do we need this?
#include <time.h>   
#include "hip/hip_runtime.h"
#include ""
#include "common.h"


// profile
// sudo nvprof --metrics shared_load_transactions_per_request,shared_store_transactions_per_request ./bin/main

// we have to use these directive to allocate static shared memory
#define NX 32
#define NY 32

// =============================================================
// kernel to store in shared memore in row major format (no bank conflict), 
// but read in the col major format (bank conflict).
__global__ void storeRow_ReadColDYN(int *out)
{
  extern __shared__ int tile[];  // row major format
  
  int row_idx = threadIdx.y * blockDim.x + threadIdx.x;
  int col_idx = threadIdx.x * blockDim.y + threadIdx.y;

  // store/fill the array on shared memory
  // NOTICE the order of threads in the matrix
  tile[row_idx] = row_idx; // store operation-row major
  // printf("number:  %s\n",tile[threadIdx.y][threadIdx.x]);
  
  __syncthreads();
  // printf("number:  %s\n", idx);
  
  // load from the shared memory and write in the global memory
  // bank conflict: all threads from a warp are loading from one banch.
  out[row_idx] = tile[col_idx];
  // printf("number:  %s\n", out[idx]);
  // __syncthreads;
}

// =============================================================
// wirte in the shared memory in the col major order, 
// and read from the memory in the row major order.
__global__ void storeCol_ReadRowDYN(int *out)
{
  extern __shared__ int tile[];  // row major format
  
  int row_idx = threadIdx.y * blockDim.x + threadIdx.x;
  int col_idx = threadIdx.x * blockDim.y + threadIdx.y;

  // store/fill the array on shared memory
  // NOTICE the order of threads in the matrix
  // all threads from warp are storing from one bank
  tile[col_idx] = row_idx; // store operation-row major

  __syncthreads();

  // load from the shared memory and write in the global memory
  out[row_idx] = tile[row_idx];
}


// =============================================================
// wirte in the shared memory in the row major order, 
// and read from the memory in the row major order.
__global__ void storeRow_ReadRowDYN(int *out)
{
  extern __shared__ int tile[];  // row major format
  
  int row_idx = threadIdx.y * blockDim.x + threadIdx.x;
  int col_idx = threadIdx.x * blockDim.y + threadIdx.y;
  // store/fill the array on shared memory
  // NOTICE the order of threads in the matrix
  tile[row_idx] = row_idx; // store operation-row major

  __syncthreads();

  // load from the shared memory and write in the global memory
  out[row_idx] = tile[row_idx];
}


int main(int argc, char* argv[])
{
  int mem_config = 0; // 0: 32 bit bank width, 1: 64 bank width
  
  if (argc>1) mem_config = atoi(argv[1]);

  if (mem_config==1) // 64 bit bank width
    hipDeviceSetSharedMemConfig(hipSharedMemBankSizeEightByte);
  else if (mem_config == 0)  // 32 bit bank width
    hipDeviceSetSharedMemConfig(hipSharedMemBankSizeFourByte);

  hipSharedMemConfig pConfig;
  hipDeviceGetSharedMemConfig(&pConfig);
  printf("GPU bank setting %s\n", pConfig==1?"4-Byte width":"8-Byte width");

  int nx = 32;
  int ny = 32;

  bool iprintf = 0;

  if (argc > 2) iprintf = atoi(argv[2]);

  int byteSize = nx * ny * sizeof(int);

  dim3 block(nx,ny);
  dim3 grid(1,1);

  printf(" grid block: (%d, %d), thread block: (%d, %d)\n", block.x, block.y, grid.x, grid.y);

  int *ref = (int *)malloc(byteSize);
  
  int *d_C; 
  hipMalloc((int**)&d_C, byteSize);
  
  clock_t start, end;

  hipMemset(&d_C, 0, byteSize);
  start = clock();
  storeRow_ReadColDYN<<<grid, block, byteSize>>>(d_C);
  hipDeviceSynchronize();
  end = clock();
  hipMemcpy(ref, d_C, byteSize, hipMemcpyDeviceToHost);
  if (iprintf) printArray(" store row - read col: ", ref, nx*ny, nx);
  printf(" execution time: %4.6f \n", (double)((double)(end - start) / CLOCKS_PER_SEC));

  hipMemset(&d_C, 0, byteSize);
  start = clock();
  storeCol_ReadRowDYN<<<grid, block, byteSize>>>(d_C);
  hipDeviceSynchronize();
  end = clock();
  hipMemcpy(ref, d_C, byteSize, hipMemcpyDeviceToHost);
  if (iprintf) printArray(" store col - read row: ", ref, nx*ny, nx);
  printf(" execution time: %4.6f \n", (double)((double)(end - start) / CLOCKS_PER_SEC));

  hipMemset(&d_C, 0, byteSize);
  start = clock();
  storeRow_ReadRowDYN<<<grid, block, byteSize>>>(d_C);
  hipDeviceSynchronize();
  end = clock();
  hipMemcpy(ref, d_C, byteSize, hipMemcpyDeviceToHost);
  if (iprintf) printArray(" store row - read row: ", ref, nx*ny, nx);
  printf(" execution time: %4.6f \n", (double)((double)(end - start) / CLOCKS_PER_SEC));

  hipFree(d_C);
  free(ref);

  hipDeviceReset();
  return 0;
}

