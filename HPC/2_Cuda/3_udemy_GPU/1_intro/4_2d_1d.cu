
// Babak Poursartip
// 09/14/2020

// Udemy Cuda
// unique index calculation

#include <hip/hip_runtime.h>
#include <cstdio>

// ===========================================
// 2d grid, 1d block
__global__ void unique_gid_calculation_2d(int *input) {

  int tid = threadIdx.x;

  int block_offset = blockIdx.x * blockDim.x;
  int row_offset = blockIdx.y * (blockDim.x * gridDim.x);

  int gid = row_offset + block_offset + tid;
  printf(" blockIdx.x: %3d,  blockIdx.y: %3d, threadIdx.x: %3d, gid: %3d, "
         "value: %4d \n",
         blockIdx.x, blockIdx.y, tid, gid, input[gid]);
}

// ===========================================
int main() {
  printf(" starts ...");

  int array_size = 16;
  int array_byte_size = sizeof(int) * array_size;
  int h_data[] = {1, 2,  3,  4,  5,  6,  7,  8,
                  9, 10, 11, 12, 13, 14, 15, 16}; // array on the host

  printf(" data on the host: \n");
  for (int i = 0; i < array_size; ++i)
    printf(" %d", h_data[i]);
  printf("\n\n");

  int *d_data; // array on the device

  hipMalloc((void **)&d_data, array_byte_size);
  hipMemcpy(d_data, h_data, array_byte_size, hipMemcpyHostToDevice);

  dim3 block(4);
  dim3 grid(2, 2);
  printf(" data on the device: \n");
  unique_gid_calculation_2d<<<grid, block>>>(d_data);

  hipDeviceSynchronize();

  hipDeviceReset();
  printf(" finished.");
  return 0;
}