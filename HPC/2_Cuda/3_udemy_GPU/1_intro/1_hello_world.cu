

// Babak Poursartip
// 09/14/2020

// Udemy Cuda
// launch hello world


#include <hip/hip_runtime.h>
#include <iostream>

// The kernel needs a modifier before the function type, in this case __global__
// other options: __host__, __device__, __shared__
// The return type is always void !?
__global__ void hello_cuda() { printf(" Hello, CUDA is here!\n"); }

int main() {
  std::cout << " starts ...\n";

  // launch the kernal: this is an asyn function call, means that the host does
  // not need to wait until the kernel finishes its job.

  // max no. of threads/block x<=1024, y<=1024m, z<=64, and x*y*z<=1024
  // max no. grids: up to (2^31 - 1) blocks in the x, and at most 65535 blocks
  // in the y and z dimensions

  // dim3 block(4); // equivalent to block(4,1,1)
  // dim3 grid(8);  // equivalent to grid(4,1,1)

  // It is more convenient to specify the number of threads in each direction,
  // separately.
  int nx, ny;
  nx = 16;
  ny = 4;

  dim3 block(8, 2);                      // equivalent to block(4,1,1)
  dim3 grid(nx / block.x, ny / block.y); // equivalent to grid(4,1,1)

  hello_cuda<<<grid, block>>>();

  // If we want the host to wait until the kernel finishes its job, we need to
  // explicity ask the host. Host will wait here, until all previous launch
  // kernels finish their executions.
  hipDeviceSynchronize();

  hipDeviceReset();

  std::cout << " Finished.\n";

  return 0;
}
