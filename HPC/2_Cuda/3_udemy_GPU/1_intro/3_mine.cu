#include "hip/hip_runtime.h"

// Babak Poursartip
// 09/14/2020

// Udemy Cuda
// unique index calculation

#include <cstdio>
#include <cuda_device_runtime_api.h>
#include <hip/hip_runtime_api.h>
#include <hip/driver_types.h>
#include <vector>

__global__ void unique_idx_calc_threadIdx(std::vector<int> *input) {

  int tid = threadIdx.x;
  // printf(" my threadIdx: %d,value: %d \n", tid, (*input)[tid]);
  input->set(0, 100);
  // printf(" my threadIdx: %d,value: %d \n", tid, input->at(tid));
}

int main() {
  printf(" starts ...");

  // int array_size = 8;
  // int array_byte_size = sizeof(int) * array_size;
  // int h_data[] = {1, 2, 3, 4, 5, 6, 7, 8}; // array on the host
  // std::vector<int> h_data(array_size);
  std::vector<int> h_data{1, 2, 3, 4, 5, 6, 7, 8};
  // h_data = {1, 2, 3, 4, 5, 6, 7, 8};

  printf(" data on the host: \n");
  for (auto c : h_data)
    printf(" %d", c);
  printf("\n\n");

  std::vector<int> *d_data; // array on the device

  hipMalloc((void **)&d_data, h_data.size() * sizeof(std::vector<int>));

  hipMemcpy(d_data, &h_data[0], h_data.size() * sizeof(std::vector<int>),
             hipMemcpyHostToDevice);

  dim3 block(8);
  dim3 grid(1);

  printf(" data on the device: \n");
  unique_idx_calc_threadIdx<<<grid, block>>>(d_data);
  hipDeviceSynchronize();

  hipDeviceReset();

  printf(" finished.\n");
  return 0;
}