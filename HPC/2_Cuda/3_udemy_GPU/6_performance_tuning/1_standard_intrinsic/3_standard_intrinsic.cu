#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#include "hip/hip_runtime.h"
#include ""


// to see the difference between the standard power function  (powf) and the intrinsic function (__powf),
// we need to generate the assembly kernel corresponding to the case.
// command to generate PTX instruction (should be assembly).
// nvcc --ptx -o ptx_instruction.ptx 3_standard_intrinsic.cu

__global__ void standard(float *ptr)
{
	*ptr = powf(*ptr, 2.0f);
}

__global__ void intrinsic(float *ptr)
{ 
	*ptr = __powf(*ptr, 2.0f);
}

int main()
{
	float value = 23;	
	int SIZE = sizeof(float);

	float *d_val;
	hipMalloc((void**)&d_val, SIZE);
	hipMemcpy(d_val, &value, SIZE, hipMemcpyHostToDevice);
	standard << <1, 1 >> > (d_val);
	intrinsic << <1, 1 >> > (d_val);
	hipDeviceSynchronize();

	hipDeviceReset();
	return 0;
}