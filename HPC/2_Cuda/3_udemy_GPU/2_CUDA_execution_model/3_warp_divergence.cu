
// Babak Poursartip
// 09/28/2020
// section 2: video 21
// warp divergence
// to profile the code
// nvcc ./2
// nvcc --metrics branch_efficiency ./2 // this is not working


#include <hip/hip_runtime.h>
#include <iostream>

__global__ void code_without_divergence() {
  int gid = blockDim.x * blockIdx.x + threadIdx.x;

  float a = 0, b = 0;

  int warp_id = gid / 32;

  if (warp_id % 2 == 0) {
    a = 100.0;
    b = 50.0;
  } else {
    a = 200.0;
    b = 72.0;
  }
}

__global__ void code_with_divergence() {
  int gid = blockDim.x * blockIdx.x + threadIdx.x;

  float a = 0, b = 0;

  if (gid % 2 == 0) {
    a = 100.0;
    b = 50.0;
  } else {
    a = 200.0;
    b = 72.0;
  }
}

int main() {
  printf(" starts ...\n");

  int size = 1 << 22;
  printf("size: %d \n", size);

  dim3 block(128);
  dim3 grid((size + block.x - 1) / block.x);

  code_without_divergence<<<grid, block>>>();
  hipDeviceSynchronize();

  code_with_divergence<<<grid, block>>>();
  hipDeviceSynchronize();

  hipDeviceSynchronize();
  printf(" finished.\n");

  return 0;
}