
// Babak Poursartip
// 09/28/2020

// warp divergence

//>>>>>>>>>>>>>>. to check the branch_efficiency:
// nvcc -G 2_warp_divergence.cu  -o 2_warp_divergence.out
// sudo nvprof --metrics branch_efficiency  ./2_warp_divergence.out
// here is the outpu:

/*
starts ...
size: 4194304 - block.x: 128 - grid.x: 32768
==32934== NVPROF is profiling process 32934, command: ./2_warp_divergence.out
execution time without warp divergence: 0.192828 
execution time with warp divergence: 0.008774 
done.
==32934== Profiling application: ./2_warp_divergence.out
==32934== Profiling result:
==32934== Metric result:
Invocations                               Metric Name                        Metric Description         Min         Max         Avg
Device "NVIDIA GeForce GT 1030 (0)"
   Kernel: code_without_divergence(void)
         1                         branch_efficiency                         Branch Efficiency     100.00%     100.00%     100.00%
   Kernel: code_with_divergence(void)
         1                         branch_efficiency                         Branch Efficiency      83.33%      83.33%      83.33%

*/


#include <hip/hip_runtime.h>
#include <iostream>
#include <time.h>

__global__ void code_without_divergence() 
{
  int gid = blockDim.x * blockIdx.x + threadIdx.x;

  float a = 0, b = 0;

  int warp_id = gid / 64;

  if (warp_id % 2 == 0) {
    a = 100.0;
    b = 50.0;
  } else {
    a = 200.0;
    b = 72.0;
  }
}

__global__ void code_with_divergence() 
{
  int gid = blockDim.x * blockIdx.x + threadIdx.x;

  float a = 0, b = 0;

  if (gid % 2 == 0) {
    a = 100.0;
    b = 50.0;
  } else {
    a = 200.0;
    b = 72.0;
  }
}

int main() {
  printf(" starts ...\n");

  int size = 1 << 22;
  
  dim3 block(128);
  dim3 grid((size + block.x - 1) / block.x);
  printf(" size: %d - block.x: %d - grid.x: %d\n", size, block.x, grid.x);
  
  clock_t start, end;

  start = clock();
  code_without_divergence<<<grid, block>>>();
  hipDeviceSynchronize();
  end = clock();
  printf(" execution time without warp divergence: %4.6f \n", (double)((double)(end - start) / CLOCKS_PER_SEC));

  start = clock();
  code_with_divergence<<<grid, block>>>();
  hipDeviceSynchronize();
  end = clock();
  printf(" execution time with warp divergence: %4.6f \n", (double)((double)(end - start) / CLOCKS_PER_SEC));

  hipDeviceSynchronize();
  printf(" done.\n");

  return 0;
}