
// Babak Poursartip
// 09/28/2020

// warp divergence


#include <hip/hip_runtime.h>
#include <iostream>
#include <time.h>

__global__ void code_without_divergence() 
{
  int gid = blockDim.x * blockIdx.x + threadIdx.x;

  float a = 0, b = 0;

  int warp_id = gid / 64;

  if (warp_id % 2 == 0) {
    a = 100.0;
    b = 50.0;
  } else {
    a = 200.0;
    b = 72.0;
  }
}

__global__ void code_with_divergence() 
{
  int gid = blockDim.x * blockIdx.x + threadIdx.x;

  float a = 0, b = 0;

  if (gid % 2 == 0) {
    a = 100.0;
    b = 50.0;
  } else {
    a = 200.0;
    b = 72.0;
  }
}

int main() {
  printf(" starts ...\n");

  int size = 1 << 22;
  
  dim3 block(128);
  dim3 grid((size + block.x - 1) / block.x);
  printf(" size: %d - block.x: %d - grid.x: %d\n", size, block.x, grid.x);
  
  clock_t start, end;

  start = clock();
  code_without_divergence<<<grid, block>>>();
  hipDeviceSynchronize();
  end = clock();
  printf(" execution time without warp divergence: %4.6f \n", (double)((double)(end - start) / CLOCKS_PER_SEC));

  start = clock();
  code_with_divergence<<<grid, block>>>();
  hipDeviceSynchronize();
  end = clock();
  printf(" execution time with warp divergence: %4.6f \n", (double)((double)(end - start) / CLOCKS_PER_SEC));

  hipDeviceSynchronize();
  printf(" done.\n");

  return 0;
}