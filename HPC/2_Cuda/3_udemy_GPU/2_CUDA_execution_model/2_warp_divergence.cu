
// Babak Poursartip
// 09/28/2020

// warp divergence


#include <hip/hip_runtime.h>
#include <iostream>

__global__ void code_without_divergence() {
  int gid = blockDim.x * blockIdx.x + threadIdx.x;

  float a = 0, b = 0;

  int warp_id = gid / 32;

  if (warp_id % 2 == 0) {
    a = 100.0;
    b = 50.0;
  } else {
    a = 200.0;
    b = 72.0;
  }
}

__global__ void code_with_divergence() {
  int gid = blockDim.x * blockIdx.x + threadIdx.x;

  float a = 0, b = 0;

  if (gid % 2 == 0) {
    a = 100.0;
    b = 50.0;
  } else {
    a = 200.0;
    b = 72.0;
  }
}

int main() {
  printf(" starts ...\n");

  int size = 1 << 22;
  printf("size: %d \n", size);

  dim3 block(128);
  dim3 grid((size + block.x - 1) / block.x);

  code_without_divergence<<<grid, block>>>();
  hipDeviceSynchronize();

  code_with_divergence<<<grid, block>>>();
  hipDeviceSynchronize();

  hipDeviceSynchronize();
  printf(" finished.\n");

  return 0;
}