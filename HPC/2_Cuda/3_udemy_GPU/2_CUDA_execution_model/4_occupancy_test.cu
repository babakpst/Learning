
// Babak Poursartip
// 09/29/2020

// section 2: video 24

/*
- If each warp is not fully occupant, that would be a wast of resources.
- We need to calculate the occupancy of SM which is equal to:
   occupancy = active warps/max warps
  * max warps can be obtained from the device manual.
  * active warps needs to be calculated: it is equal to the min of warps
obtained based on the size of register and shared memory. Refer to the slides
2-6

- To get the value of the shared memory and register, use the following command:
nvcc --ptxas-options=-v -o 4_occupancy_test.out 4_occupancy_test.cu


- Warp allocation granularity means that warps are allocated in multiples of a
number. For example, multiples of 4.

- Use the excel sheet to calculate the CUDA occupancy.

*/

#include <hip/hip_runtime.h>
#include <iostream>

__global__ void occupancy_test(int *results) {
  int gid = blockDim.x * blockIdx.x + threadIdx.x;

  int x1 = 1;
  int x2 = 2;
  int x3 = 3;
  int x4 = 4;
  int x5 = 5;
  int x6 = 6;
  int x7 = 7;
  int x8 = 8;
  results[gid] = x1 + x2 + x3 + x4 + x5 + x6 + x7 + x8;
}

int main() {
  int size = 1 << 16;
  printf(" size: %d \n", size);

  int byte_size = size * sizeof(int);

  int *d_results;
  hipMalloc((void **)&d_results, byte_size);
  hipMemset(d_results, 0, byte_size);

  dim3 block(128);
  dim3 grid((size + block.x - 1) / block.x);
  printf(" grids: %d, block: %d \n", grid.x, block.x);
  occupancy_test<<<grid, block>>>(d_results);
  hipDeviceSynchronize();
  return 0;
}