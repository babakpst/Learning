
// Babak Poursartip
// 10/01/2020

// profile/profiling with nvprof
/*

nvprof modes:
1- summary mode
2- GPU and API trace mode
3- event metrics summary mode
4- event, metrics trace mode

- To run nvprof, first create the executable (nvcc file.cu -o file.out). Then,
profile using: nvprof ./file.out (This would be the summary mode)

metrics:
- sm_efficiency
- achieved_occupancy
- branch_efficiency
- gld_efficiency
- gld_throughput
- dram_read_throughput
- inst_per_warp
- stall_sync


arguments :
1 - kernel (0:1D or 1:2D),
2 - input size (2 pow (x))
3 - for 2D kernel nx,
4 - block.x
5 - block.y

- To run with metrics: 

$ nvcc 5_sum_array_profiling.cu -o 5
$ sudo nvprof # ./5 # this runs in the summary mode (default operation)
$ sudo nvprof --metrics gld_efficiency,sm_efficiency,achieved_occupancy ./5 0 22 
$ sudo nvprof --metrics gld_efficiency,sm_efficiency,achieved_occupancy ./5 0 25 0 7
$ sudo nvprof --metrics gld_efficiency,sm_efficiency,achieved_occupancy ./5 1 25 20 7 2 
$ sudo nvprof --metrics gld_efficiency,sm_efficiency,achieved_occupancy ./5 1 25 20 8 2 

*/


#include <hip/hip_runtime.h>
#include <cstdio>
#include <iostream>

// =================================
// cuda error check macro
#define gpuErrchk(ans)                                                         \
  { gpuAssert(ans, __FILE__, __LINE__); }

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true) 
{
  if (code != hipSuccess) 
  {
    fprintf(stderr, "GPUassert: %s. File: %s, line: %d. \n", hipGetErrorString(code), file, line);
    if (abort) {
      // printf(" Exists from the gpuErrorCheck func.\n");
      exit(code);
    }
  }
}

// ==============================================
void compare_arrays(const float *a, const float *b, const int size) 
{
  for (int i = 0; i < size; ++i) {
    if (a[i] != b[i]) {
      printf("\n Arrays are not equal!! %d %f %f \n", i, a[i], b[i]);
      return;
    }
  }
  printf("\n Arrays are identical!! \n");
}

// ==============================================
void sum_array_cpu(float *a, float *b, float *c, const int size) 
{
  for (int i = 0; i < size; ++i) {
    c[i] = a[i] + b[i];
  }
}

// ==============================================
// 1D grid, and 1D block. Thus, nx = size
__global__ void sum_array_1Dgrid_1Dblock(float *a, float *b, float *c, int nx) 
{
  int gid = blockDim.x * blockIdx.x + threadIdx.x;
  c[gid] = a[gid] + b[gid];
  // printf("inside %d \n", gid);
}
// ==============================================
// 2D grid, and 2D block. Thus, nx*ny = size.
__global__ void sum_arrays_2Dgrid_2Dblock(float *a, float *b, float *c, int nx, int ny) {
  int gidx = blockDim.x * blockIdx.x + threadIdx.x;
  int gidy = blockDim.y * blockIdx.y + threadIdx.y;
  int gid = gidy * nx + gidx;

  if (gidx < nx && gidy < ny)
    c[gid] = a[gid] + b[gid];
}

// ==============================================
void run_sum_array_1d(int argc, char **argv) {

  printf(" Running 1D grid ");

  int size = 1 << 22; // the default size of the array.
  int block_size = 128;
  // int nx, ny = 0;

  if (argc > 2)
    size = 1 << atoi(argv[2]);

  if (argc > 4)
    block_size = 1 << atoi(argv[4]);

  const int byte_size = size * sizeof(float);

  printf(" size of the array: %d, %d \n", size, byte_size);

  float *h_a, *h_b, *h_out, *h_ref;

  h_a = (float *)malloc(byte_size);
  h_b = (float *)malloc(byte_size);
  h_out = (float *)malloc(byte_size);
  h_ref = (float *)malloc(byte_size);

  if (!h_a)
    printf(" host memory allocation error\n");

  for (int i = 0; i < size; ++i) {
    h_a[i] = i % 10;
    h_b[i] = i % 7;
  }

  sum_array_cpu(h_a, h_b, h_out, size);

  dim3 block(block_size);
  dim3 grid((size + block.x - 1) / block.x);

  printf(" launching the Kernel: grid(%d,%d,%d) - block(%d,%d,%d) \n", grid.x,
         grid.y, grid.z, block.x, block.y, block.z);

  float *d_a, *d_b, *d_c;

  gpuErrchk(hipMalloc((void **)&d_a, byte_size));
  gpuErrchk(hipMalloc((void **)&d_b, byte_size));
  gpuErrchk(hipMalloc((void **)&d_c, byte_size));

  gpuErrchk(hipMemset(d_c, 0, byte_size));

  gpuErrchk(hipMemcpy(d_a, h_a, byte_size, hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(d_b, h_b, byte_size, hipMemcpyHostToDevice));

  sum_array_1Dgrid_1Dblock<<<grid, block>>>(d_a, d_b, d_c, size);
  gpuErrchk(hipDeviceSynchronize());

  gpuErrchk(hipMemcpy(h_ref, d_c, byte_size, hipMemcpyDeviceToHost));

  compare_arrays(h_out, h_ref, size);

  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);
  free(h_ref);
  free(h_out);
  free(h_b);
  free(h_a);
}

// ==============================================
void run_sum_array_2d(int argc, char **argv) {

  printf(" Running 2D grid ");

  int size = 1 << 22; // the default size of the array = 4194304
  int block_x = 128;
  int nx = 1 << 14; // 16384
  int ny = size / nx;
  int block_y = 8;

  if (argc > 2)
    size = 1 << atoi(argv[2]);
  if (argc > 3) {
    nx = 1 << atoi(argv[3]);
    ny = size / nx;
  }

  if (argc > 4) {
    int pow = atoi(argv[4]);
    if (pow < 3 || pow > 10) {
      printf("Block size is invalid, default block size used (%d,%d)\n",
             block_x, block_y);
    } else {
      block_x = 1 << pow;
      block_y = 1024 / block_x;
    }
  }

  unsigned int byte_size = size * sizeof(float);

  printf("Input size : %d, nx : %d, ny : %d, block_x : %d, block_y : %d \n",
         size, nx, ny, block_x, block_y);

  float *h_a, *h_b, *h_out, *h_ref;

  h_a = (float *)malloc(byte_size);
  h_b = (float *)malloc(byte_size);
  h_out = (float *)malloc(byte_size);
  h_ref = (float *)malloc(byte_size);

  if (!h_a)
    printf(" host memory allocation error\n");

  for (int i = 0; i < size; ++i) {
    h_a[i] = i % 10;
    h_b[i] = i % 7;
  }

  sum_array_cpu(h_a, h_b, h_out, size);

  dim3 block(block_x, block_y);
  dim3 grid((nx + block_x - 1) / block_x, (ny + block_y - 1) / block_y);

  printf(" launching the Kernel: grid(%d,%d,%d) - block(%d,%d,%d) \n", grid.x,
         grid.y, grid.z, block.x, block.y, block.z);

  float *d_a, *d_b, *d_c;
  gpuErrchk(hipMalloc((void **)&d_a, byte_size));
  gpuErrchk(hipMalloc((void **)&d_b, byte_size));
  gpuErrchk(hipMalloc((void **)&d_c, byte_size));

  gpuErrchk(hipMemset(d_c, 0, byte_size));

  gpuErrchk(hipMemcpy(d_a, h_a, byte_size, hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(d_b, h_b, byte_size, hipMemcpyHostToDevice));

  sum_arrays_2Dgrid_2Dblock<<<grid, block>>>(d_a, d_b, d_c, nx, ny);

  gpuErrchk(hipDeviceSynchronize());

  gpuErrchk(hipMemcpy(h_ref, d_c, byte_size, hipMemcpyDeviceToHost));

  compare_arrays(h_out, h_ref, size);

  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);
  free(h_ref);
  free(h_out);
  free(h_b);
  free(h_a);
}

// ==============================================
////arguments :
////1 - kernel (0:1D or 1:2D),
////2 - input size (2 pow (x))
////3 - for 2D kernel nx,
////4 - block.x
////5 - block.y
// ==============================================
int main(int argc, char **argv) 
{
  printf("Sum array code for nvprof: \n");

  if (argc > 1) 
  {
    if (atoi(argv[1]) > 0)
      run_sum_array_2d(argc, argv);
    else
      run_sum_array_1d(argc, argv);
  } 
  else
    run_sum_array_1d(argc, argv);

  // query_device();
  return 0;
}



