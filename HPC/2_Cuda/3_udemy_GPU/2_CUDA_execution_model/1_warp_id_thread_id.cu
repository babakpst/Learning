
// Babak Poursartip
// 09/28/2020

// section 2: video 20


#include <hip/hip_runtime.h>
#include <iostream>

__global__ void print_details_of_warps() {

  int gid = blockIdx.y + gridDim.x * blockDim.x + blockIdx.x * blockDim.x +
            threadIdx.x;

  int warp_id = threadIdx.x / 32;
  int gbid = blockIdx.y * gridDim.x + blockIdx.x;
  printf(" tid: %d, bid.x: %2d, bid.y: %d, gid: %3d, warp_id: %d, gbid: %d\n",
         threadIdx.x, blockIdx.x, blockIdx.y, gid, warp_id, gbid);
}

int main() {

  printf(" starts ...\n");

  dim3 block(42);
  dim3 grid(2, 2);

  print_details_of_warps<<<grid, block>>>();
  hipDeviceSynchronize();

  printf(" finished \n");

  hipDeviceReset();
  return EXIT_SUCCESS;
}
