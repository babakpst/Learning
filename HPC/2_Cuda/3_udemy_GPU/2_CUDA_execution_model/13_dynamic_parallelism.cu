
// to build
// nvcc 13_dynamic_parallelism.cu -arch=sm_61 -rdc=true -o 13
// For profiler:  sudo nvvp ./exec 


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

// #include "cuda.h"
// #include "cuda_runtime.h"
// #include "devive_launch_parameters.h"

__global__ void dynamic_parallelism(int size, int depth)
{
  printf("depth: %d - tid: %d \n", depth, threadIdx.x);

  if (size==1) return;

  if (threadIdx.x==0)
  {
    dynamic_parallelism<<<1,size/2>>>(size/2, depth+1);
  }
}


int main(int argc, char* argbv[])
{

  dynamic_parallelism<<<1,16>>>(16,0);
  hipDeviceSynchronize();
  hipDeviceReset();

  printf(" done\n");

  return 0;
}

