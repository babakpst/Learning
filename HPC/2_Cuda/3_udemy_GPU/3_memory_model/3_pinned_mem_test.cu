#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>

// nvcc 3_pinned_mem_test.cu -o 3_pinned_memory
// nvprof ./3_pinned_memory
// nvprof --print-gpu-trace ./3_pinned_memory


int main(int argc, char **argv) 
{   								
	// memory size   128 MBs
	int isize = 1<<28;
	int nbytes = isize * sizeof(float);
											
	// allocate the host memory (pageable)
	// float *h_a = (float *)malloc(nbytes);
	
  // pinned memory
  float *h_a;
	hipHostMalloc((float **)&h_a, nbytes, hipHostMallocDefault);

	// allocate the device memory   
	float *d_a; 
   hipMalloc((float **)&d_a, nbytes);
									
	// initialize the host memory   
	for(int i=0;i<isize;i++) 
		h_a[i] = 7;
									
	// transfer data from the host to the device   
	hipMemcpy(d_a, h_a, nbytes, hipMemcpyHostToDevice);
									
	// transfer data from the device to the host   
	hipMemcpy(h_a, d_a, nbytes, hipMemcpyDeviceToHost);
							
	// free memory   
	hipFree(d_a);
	// free(h_a);
	hipHostFree(h_a);
									
	// reset device    
	hipDeviceReset();   
	return EXIT_SUCCESS;
}