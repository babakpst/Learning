#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#include "hip/hip_runtime.h"


/* 

with L1 cache
To enable L1 cache read, use this flag: -Xptxas -dlcm=ca. So, memory access word length is 128 Bytes.
$ nvcc -Xptxas -dlcm=ca -o 6_misaligned_read 6_misaligned_read.cu
$ sudo nvprof --metrics gld_efficiency,gld_transactions ./6_misaligned_read

>>>> output with offset 0:
Invocations                               Metric Name                        Metric Description         Min         Max         Avg
Device "NVIDIA GeForce GT 1030 (0)"
    Kernel: misaligned_read_test(float*, float*, float*, int, int)
          1                            gld_efficiency             Global Memory Load Efficiency     100.00%     100.00%     100.00%
          1                          gld_transactions                  Global Load Transactions    33554434    33554434    33554434

>>>> output with offset 22:
Invocations                               Metric Name                        Metric Description         Min         Max         Avg
Device "NVIDIA GeForce GT 1030 (0)"
    Kernel: misaligned_read_test(float*, float*, float*, int, int)
          1                            gld_efficiency             Global Memory Load Efficiency      80.00%      80.00%      80.00%
          1                          gld_transactions                  Global Load Transactions    33554418    33554418    33554418

without L1 cache:

/* To disable L1 cache read, use this flag: -Xptxas -dlcm=ca. So, memory access word length is 32 Bytes.
$ nvcc -Xptxas -dlcm=cg -o 6_misaligned_read 6_misaligned_read.cu
$  sudo nvprof --metrics gld_efficiency,gld_transactions ./6_misaligned_read

>>>> output with offset 0:
Invocations                               Metric Name                        Metric Description         Min         Max         Avg
Device "NVIDIA GeForce GT 1030 (0)"
    Kernel: misaligned_read_test(float*, float*, float*, int, int)
          1                            gld_efficiency             Global Memory Load Efficiency     100.00%     100.00%     100.00%
          1                          gld_transactions                  Global Load Transactions    33554434    33554434    33554434

>>>> output with offset 22:
Invocations                               Metric Name                        Metric Description         Min         Max         Avg
Device "NVIDIA GeForce GT 1030 (0)"
    Kernel: misaligned_read_test(float*, float*, float*, int, int)
          1                            gld_efficiency             Global Memory Load Efficiency      80.00%      80.00%      80.00%
          1                          gld_transactions                  Global Load Transactions    33554418    33554418    33554418

*/

__global__ void misaligned_read_test(float* a, float* b, float *c, int size, int offset)
{
  int gid = blockIdx.x * blockDim.x + threadIdx.x;
  int k = gid + offset;

  if (k < size)
    c[gid] = a[k]+ b[k]; // here global read is misalighned by offset

  //c[gid] = a[gid];
}

int main(int argc, char** argv)
{
  printf("Runing 1D grid \n");
  int size = 1 << 25;
  int block_size = 128;
  unsigned int byte_size = size * sizeof(float);
  int offset = 0;

  if (argc > 1)
    offset = atoi(argv[1]);

  printf("Input size : %d \n", size);

  float * h_a, *h_b, *h_ref;
  h_a = (float*)malloc(byte_size);
  h_b = (float*)malloc(byte_size);
  h_ref = (float*)malloc(byte_size);


  if (!h_a)
    printf("host memory allocation error \n");

  for (size_t i = 0; i < size; i++)
  {
    h_a[i] = i % 10;
    h_b[i] = i % 7;
  }

  dim3 block(block_size);
  dim3 grid((size + block.x - 1) / block.x);

  printf("Kernel is lauch with grid(%d,%d,%d) and block(%d,%d,%d) \n",grid.x, grid.y, grid.z, block.x, block.y, block.z);

  float *d_a, *d_b, *d_c;

  hipMalloc((void**)&d_a, byte_size);
  hipMalloc((void**)&d_b, byte_size);
  hipMalloc((void**)&d_c, byte_size);
  hipMemset(d_c, 0, byte_size);

  hipMemcpy(d_a, h_a, byte_size, hipMemcpyHostToDevice);
  hipMemcpy(d_b, h_b, byte_size, hipMemcpyHostToDevice);

  misaligned_read_test <<<grid, block >>> (d_a, d_b, d_c, size, offset);

  hipDeviceSynchronize();
  hipMemcpy(h_ref, d_c, byte_size, hipMemcpyDeviceToHost);

  hipFree(d_c);
  hipFree(d_b);
  hipFree(d_a);
  free(h_ref);
  free(h_b);
  free(h_a);
}