#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include "cuda_common.cuh"
#include "common.h"

//assume grid is 1D and block is 1D then nx = size
__global__ void sum_arrays_1Dgrid_1Dblock(float* a, float* b, float *c, int nx)
{
	int gid = blockIdx.x * blockDim.x + threadIdx.x;
	c[gid] = a[gid] + b[gid];
}

//assume grid is 2D and block is 2D then nx * ny = size
__global__ void sum_arrays_2Dgrid_2Dblock(float* a, float* b, float *c, int nx, int ny)
{
	int gidx = blockIdx.x * blockDim.x + threadIdx.x;
	int gidy = blockIdx.y*blockDim.y + threadIdx.y;
	
	int gid = gidy * nx + gidx;
	
	if(gidx < nx && gidy < ny)
		c[gid] = a[gid] + b[gid];
}

void run_sum_array_1d(int argc, char** argv)
{
	printf("Runing 1D grid \n");
	int size = 1 << 22;
	int block_size = 128;

	int nx, ny = 0;

	if (argc > 2)
		size = 1 << atoi(argv[2]);

	if (argc > 4)
		block_size = 1 << atoi(argv[4]);


	unsigned int byte_size = size * sizeof(float);

	printf("Input size : %d \n",size);

	float * h_a, *h_b, *h_out, *h_ref;
	h_a = (float*)malloc(byte_size);
	h_b = (float*)malloc(byte_size);
	h_out = (float*)malloc(byte_size);
	h_ref = (float*)malloc(byte_size);


	if (!h_a)
		printf("host memory allocation error \n");

	for (size_t i = 0; i < size; i++)
	{
		h_a[i] = i % 10;
		h_b[i] = i % 7;
	}

	sum_array_cpu(h_a, h_b, h_out,size);

	dim3 block( block_size);
	dim3 grid((size+block.x -1)/block.x);

	printf("Kernel is lauch with grid(%d,%d,%d) and block(%d,%d,%d) \n",
		grid.x,grid.y,grid.z,block.x,block.y, block.z);

	float *d_a, *d_b, *d_c;

	gpuErrchk(hipMalloc((void**)&d_a, byte_size));
	gpuErrchk(hipMalloc((void**)&d_b, byte_size));
	gpuErrchk(hipMalloc((void**)&d_c, byte_size));
	gpuErrchk(hipMemset(d_c,0,byte_size));

	gpuErrchk(hipMemcpy(d_a,h_a,byte_size,hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(d_b, h_b, byte_size, hipMemcpyHostToDevice));

	sum_arrays_1Dgrid_1Dblock << <grid, block >> > (d_a, d_b, d_c, size);

	gpuErrchk(hipDeviceSynchronize());
	gpuErrchk(hipMemcpy(h_ref,d_c,byte_size,hipMemcpyDeviceToHost));

	compare_arrays(h_out, h_ref,size);

	hipFree(d_c);
	hipFree(d_b);
	hipFree(d_a);
	free(h_ref);
	free(h_out);
	free(h_b);
	free(h_a);
}

void run_sum_array_2d(int argc, char** argv)
{
	printf("Runing 2D grid \n");

	int size = 1 << 22; // 4194304
	int block_x = 128;

	int nx = 1 << 14; // 16384 
	int ny = size / nx;
	int block_y = 8;

	if (argc > 2)
		size = 1 << atoi(argv[2]);

	if (argc > 3)
		nx = 1 << atoi(argv[3]);

	ny = size / nx;

	if (argc > 4)
	{
		int pow = atoi(argv[4]);
		if (pow < 3 || pow > 10)
		{
			printf("Block size is invalid, default block size used (%d,%d)\n",block_x,block_y);
		}
		else
		{
			block_x = 1 << pow;
			block_y = 1024 / block_x;
		}
	}

	unsigned int byte_size = size * sizeof(float);

	printf("Input size : %d, nx : %d, ny : %d, block_x : %d, block_y : %d \n",
		size,nx,ny,block_x,block_y);

	float * h_a, *h_b, *h_out, *h_ref;
	h_a = (float*)malloc(byte_size);
	h_b = (float*)malloc(byte_size);
	h_out = (float*)malloc(byte_size);
	h_ref = (float*)malloc(byte_size);


	if (!h_a)
		printf("host memory allocation error \n");

	for (size_t i = 0; i < size; i++)
	{
		h_a[i] = i % 10;
		h_b[i] = i % 7;
	}
	
	sum_array_cpu(h_a, h_b, h_out, size);

	dim3 block(block_x,block_y);
	dim3 grid((nx + block_x -1)/ block_x, (ny + block_y -1)/block_y );

	printf("Kernel is lauch with grid(%d,%d,%d) and block(%d,%d,%d) \n",
		grid.x, grid.y, grid.z, block.x, block.y, block.z);

	float *d_a, *d_b, *d_c;

	gpuErrchk(hipMalloc((void**)&d_a, byte_size));
	gpuErrchk(hipMalloc((void**)&d_b, byte_size));
	gpuErrchk(hipMalloc((void**)&d_c, byte_size));
	gpuErrchk(hipMemset(d_c, 0, byte_size));

	gpuErrchk(hipMemcpy(d_a, h_a, byte_size, hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(d_b, h_b, byte_size, hipMemcpyHostToDevice));
	
	sum_arrays_2Dgrid_2Dblock << <grid, block >> > (d_a, d_b, d_c,nx,ny);
	gpuErrchk(hipDeviceSynchronize());

	gpuErrchk(hipMemcpy(h_ref, d_c, byte_size, hipMemcpyDeviceToHost));

	compare_arrays(h_out, h_ref, size);

	hipFree(d_c);
	hipFree(d_b);
	hipFree(d_a);
	free(h_ref);
	free(h_out);
	free(h_b);
	free(h_a);
}

////arguments :
////1 - kernel (0:1D or 1:2D), 
////2 - input size (2 pow (x))
////3 - for 2D kernel nx, 
////4 - block.x 
////5 - block.y  
//int main(int argc, char** argv)
//{
//	printf("\n----------------------- SUM ARRAY EXAMPLE FOR NVPROF ------------------------ \n\n");
//	if (argc > 1)
//	{
//		if (atoi(argv[1]) > 0)
//		{
//			run_sum_array_2d(argc, argv);
//		}
//		else
//		{
//			run_sum_array_1d(argc, argv);
//		}
//	}
//	else
//	{
//		run_sum_array_1d(argc, argv);
//	}
//
//	//query_device();
//	return 0;
//}