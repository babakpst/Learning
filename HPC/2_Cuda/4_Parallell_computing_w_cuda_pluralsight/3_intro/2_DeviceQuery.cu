
//#include <iostream>
#include <cstdio>
#include "hip/hip_runtime.h"


int main(){
printf("\n Let's start ... \n");

int count;
hipGetDeviceCount(&count);

hipDeviceProp_t prop;
for (int i = 0; i < count; ++ i)
{
  hipGetDeviceProperties(&prop,i);
  printf(" Device: %d, %s\n", i,  prop.name);

  printf(" Compute Capability: %d.%d\n", prop.major, prop.minor);  
  

  printf(" Max grid dimensions: (%dx%dx%d)\n", prop.maxGridSize[0],
                                               prop.maxGridSize[1],
                                               prop.maxGridSize[2]);

  printf(" Max block dimensions: (%dx%dx%d)\n", prop.maxThreadsDim[0],
                                                prop.maxThreadsDim[1],
                                                prop.maxThreadsDim[2]);
  /*
  std::cout << " Device: " << i << prop.name << std::endl;
  std::cout << " Compute Capability: " << prop.major << "." << prop.minor << std::endl;  
  std::cout << " Max grid dimensions: (" << prop.maxGridSize[0] << "x" 
                                         << prop.maxGridSize[1] << "x"
                                         << prop.maxGridSize[2] << ")\n";
  std::cout << " Max block dimensions: (" << prop.maxThreadsDim[0] << "x" 
                                          << prop.maxThreadsDim[1] << "x"
                                          << prop.maxThreadsDim[2] << ")\n";
  */
}


printf("\n done \n");
return 0;
}
