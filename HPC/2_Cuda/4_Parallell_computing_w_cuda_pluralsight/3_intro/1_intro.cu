

// Babak Poursartip
// 01/28/2021

// CUDA
//topic:



#include <hip/hip_runtime.h>
#include "cstdio"
#include <algorithm>
#include <functional>


// =================================
__global__ void addCuda( int *a,  int *b, int *c)
{
  int i = threadIdx.x; 
  c[i] = a[i] + b[i];
}

// =================================
void addIndex( int *a,  int *b, int *c, int i)
{
  c[i] = a[i] + b[i];
}
// =================================
void add( int *a,  int *b, int *c, const int &count)
{
  std::transform(a,a+count, b, c , std::plus<>{});
}

// =================================
int main(){
printf("\n starts ... \n");

const int count = 6;
int ha[]={5,2,5,8,7,6};
int hb[]={15,12,15,18,17,16};

// serial adding
int hc[count];
add(ha, hb, hc, count);
printf(" 1 ================\n");
std::for_each(hc,hc+count, [](int&c){printf("%d, ",c);});
printf("\n");

// add index
for (int i = 0; i < count; ++i)
  addIndex(ha, hb, hc, i);
printf(" 2 ================\n");
std::for_each(hc,hc+count, [](int&c){printf("%d, ",c);});
printf("\n");


// cuda adding
int size = count * sizeof(int);
int * da, *db, * dc;
hipMalloc(&da, size);
hipMalloc(&db, size);
hipMalloc(&dc, size);

hipMemcpy(da, ha, size, hipMemcpyHostToDevice);
hipMemcpy(db, hb, size, hipMemcpyHostToDevice);

addCuda<<<1,count>>>(da, db, dc);

hipMemcpy(hc, db, size, hipMemcpyDeviceToHost);
printf(" 3 ================\n");
std::for_each(hc,hc+count, [](int&c){printf("%d, ",c);});
printf("\n");

hipDeviceReset();

printf("\n finished. \n");

return 0;
}
