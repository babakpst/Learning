#include "hip/hip_runtime.h"

#include <pthread.h>

#include <cmath>
#include <iostream>


// ===================
typedef void *(*func)(void*); // bbk what is it?

// ===================
pthread_t startThread(func f, void *data)
{
  pthread_t t;
  pthread_create(&t, NULL, f, data);
  return t;
}

// ===================
void endThread(pthread_t t)
{
  pthread_join(t, NULL);
}

// ===================
struct work
{
  int device;
  int offset;
  int count;
  int *a, *b, *c;
};

// ===================
__global__ void add(const int *a, const int *b, int *c)
{
  int tid = threadIdx.x;
  c[tid] = a[tid] + b[tid];
}
 
// ===================
void* dowork(void* spec)
{
  work *w= reinterpret_cast<work*>(spec); 


  if(w->device != 0)
  {
    hipSetDevice(w->device);
    hipSetDeviceFlags(hipDeviceMapHost);
  }


  int *da, *db, *dc;
  hipHostGetDevicePointer(&da, w->a, 0);
  hipHostGetDevicePointer(&db, w->b, 0);
  hipHostGetDevicePointer(&dc, w->c, 0);

  da += w->offset;
  db += w->offset;
  dc += w->offset;

  add<<<1,w->count/2>>>(da, db, dc);

return NULL;

}

// ===================
int main(){

std::cout << "starts ...\n";

  int dc;
  hipGetDeviceCount(&dc);

  std::cout << "no. of devices: " << dc << std::endl;

  for (int i = 0; i < dc; ++i)
  {
    hipDeviceProp_t props;
    hipGetDeviceProperties(&props, i);

    std::cout << "Device: " << i << " - " << props.name;
    std::cout << "compute capability: " << props.major << "." << props.minor << std::endl;
    std::cout << " map history memory? " << props.canMapHostMemory << std::endl;
  }

  // allocate
  int count = 128;
  int *a, *b, *c;
  hipSetDevice(0);
  hipSetDeviceFlags(hipDeviceMapHost);
  hipHostAlloc(&a, count*sizeof(int), hipHostMallocMapped | hipHostMallocPortable | hipHostMallocWriteCombined);
  hipHostAlloc(&b, count*sizeof(int), hipHostMallocMapped | hipHostMallocPortable | hipHostMallocWriteCombined);
  hipHostAlloc(&c, count*sizeof(int), hipHostMallocMapped | hipHostMallocPortable | hipHostMallocWriteCombined);

//
  for (int i = 0; i < count; ++i)
  {
      a[i] = i;
      b[i] = count-i;
  }

  // 
  work task[2];
  task[0].device = 0;
  task[0].offset = 0;
  task[1].device = 1;
  task[1].offset = count/2;
  task[0].count = task[1].count = count;
  task[0].a = task[1].a = a;
  task[0].b = task[1].b = b;
  task[0].c = task[1].c = c;

  // creating two threads for each task.
  pthread_t t = startThread(dowork, &task[1]);
  dowork(&task[0]);
  endThread(t);


  // output
  for (int i = 0; i<count; ++i)
  {
  std::cout << c[i] << "\t";
  }

  hipHostFree(a);
  hipHostFree(b);
  hipHostFree(c);


std::cout << "\nends \n";
return 0;


}




































