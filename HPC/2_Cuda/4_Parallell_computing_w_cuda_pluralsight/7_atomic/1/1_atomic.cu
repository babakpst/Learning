#include "hip/hip_runtime.h"

// Babak Poursartip
// 02/27/2021

// CUDA
//topic: atomic

#include <iostream>
#include "sm_20_atomic_functions.h"

// ==============================
__device__ int dSum=0;

__global__ void sumArray(int* d)
{
  int tid = threadIdx.x;
  dSum +=d[tid];

}

// ==============================
__global__ void sumArrayAtomic(int* d)
{
  int tid = threadIdx.x;
  atomicAdd(&dSum, d[tid]);

}

// ==============================
int main()
{
printf(" starts \n");

const int count = 256;
const int size = count * sizeof(int);

int h[count];

for (int i = 0; i < count; ++i) h[i] = i+1;

int* d;
hipMalloc(&d, size);
hipMemcpy(d,h, size, hipMemcpyHostToDevice);

int Sum;
sumArray<<<1, count>>>(d);
hipMemcpyFromSymbol(&Sum, HIP_SYMBOL(dSum), sizeof(int));
std::cout << " sum is: " << Sum << std::endl;

sumArrayAtomic<<<1, count>>>(d);
hipMemcpyFromSymbol(&Sum, HIP_SYMBOL(dSum), sizeof(int));
std::cout << " sum is: " << Sum << std::endl;

hipFree(d);

printf(" done \n");
return 0;
} 
