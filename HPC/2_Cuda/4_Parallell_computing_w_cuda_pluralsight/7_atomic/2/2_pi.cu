#include "hip/hip_runtime.h"

/*
Babak Poursartip
02/27/2021

CUDA
topic: Monte Carlo method to calculate Pi.
For a circle of radius r, surrounded by a square:

Area of circle = pi*r^2
Area of square = 4*r^2

Thus, 
pi = 4* area of circle / area of square.

which is equivalent to: 
pi = 4 * (number of pionts in the circle)/ number of points outside the square(=total number).

*/

#include <iostream>
#include "hiprand.h"
#include <iomanip>
#include "sm_20_atomic_functions.h"


// ==============================
__device__ int dCount = 0;

__global__ void countPoints(const float* xs, const float* ys)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  
  float x = xs[idx] - 0.5f;
  float y = ys[idx] - 0.5f;
  int n = sqrtf(x*x + y*y) > 0.5f ?0:1;
  atomicAdd(&dCount, n);
}


// ==============================
int main()
{
printf(" starts \n");

const int count = 512*512;
const int size = count * sizeof(float);

hipError_t cudaStatus;
hiprandStatus_t hiprandStatus;

hiprandGenerator_t gen;

hiprandStatus = hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_MTGP32);
hiprandSetPseudoRandomGeneratorSeed(gen, time(0));

float* x;
float* y;

cudaStatus = hipMalloc(&x, size);
cudaStatus = hipMalloc(&y, size);

hiprandStatus = hiprandGenerateUniform(gen, x, count);
hiprandStatus = hiprandGenerateUniform(gen, y, count);

countPoints<<<512,512>>>(x,y);

int hCount;
hipMemcpyFromSymbol(&hCount, HIP_SYMBOL(dCount), sizeof(int));

hipFree(x);
hipFree(y);

std::cout << std::setprecision(12) << " Pi is: " << (4.0f * (float)hCount/ (float)count) << std::endl;

printf(" done \n");
return 0;
} 
