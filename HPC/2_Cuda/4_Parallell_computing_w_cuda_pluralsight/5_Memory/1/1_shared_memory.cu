

// Babak Poursartip
// 02/21/2021

// CUDA
//topic: shared memory



#include <hip/hip_runtime.h>
#include <cstdio>
#include <ctime>
#include <iostream>

#include <hiprand/hiprand.h>

// ==============================
__global__ void sumSingleBlock(int *d)
{

  extern __shared__ int dcopy[];

  int tid = threadIdx.x;

  // copy d to dcopy
  dcopy[tid*2] = d[tid*2];
  dcopy[tid*2+1] = d[tid*2+1];


  // tc: number of participating threads
  //for (int tc = blockDim.x; tc > 0; tc >>=1) // changes the number of threads by half(tc>>=1)
  for (int tc = blockDim.x, stepSize = 1; tc > 0; tc /=2, stepSize *=2) // changes the number of threads by half(tc>>=1)
  {
    // thread must be allowed to write
    if (tid < tc)
    {
      int pa =  tid * stepSize * 2;
      int pb = pa + stepSize;
      dcopy[pa] += dcopy[pb];

# if __CUDA_ARCH__>=200
    printf("%d, %d, %d, %d, %d \n", tid, tc, stepSize, pa, pb);
#endif
    }    
  }
  if (tid == 0) d[0] = dcopy[0];
}


// ==============================
int main()
{
printf(" starts \n");

const int count  = 32;
const int size = count * sizeof(int);

int h[count];

for (int i = 0; i < count; ++i)
  h[i] = i + 1;
  
int *d;
hipMalloc(&d, size);
hipMemcpy(d, h, size, hipMemcpyHostToDevice);

sumSingleBlock<<<1, count/2, size>>>(d);

int result;
hipMemcpy(&result, d, sizeof(int), hipMemcpyDeviceToHost);
std::cout << " sum: "<< result << std::endl;

hipFree(d);

printf(" done \n");
return 0;
} 