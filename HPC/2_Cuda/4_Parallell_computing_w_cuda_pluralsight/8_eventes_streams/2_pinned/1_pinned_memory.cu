
/*
Babak Poursartip
02/27/2021

CUDA
topic: pinned memory

- Instead of using malloc or new to allocation memory on the CPU(host), we use cudaHostAlloc(). This will allocate a pinned memory on the host.

- To free the memory, we use cudaFreeHost, instead of delete to deallocate. 

- The disadvantage is that you cannot swap the memory to the disk & we need to have enough memory to use this memory. 

*/


#include <hip/hip_runtime.h>
#include <iostream>


// ==============================
float timing(bool pinned, bool toDevice)
{

  std::cout << " p: " << pinned <<" t: " << toDevice << std::endl;
  const int count = 1 << 20;
  const int iterations = 1 << 6;
  const int size = count * sizeof(int);
  
  hipEvent_t start, end;

  int *d, *h;

  float elapsed;
  hipError_t status;
  
  hipEventCreate(&start);
  hipEventCreate(&end);
  
  hipMalloc(&d, size);
  if(pinned) 
    hipHostAlloc(&h, size, hipHostMallocDefault);
  else 
    h = new int[count];
    
  hipEventRecord(start);
  
  for (int i = 0; i < iterations; ++i)
  {
    if (toDevice)
      status = hipMemcpy(d, h, size, hipMemcpyHostToDevice);
    else
      status = hipMemcpy(h, d, size, hipMemcpyDeviceToHost);
  }

  hipEventRecord(end);
  hipEventSynchronize(end);
  
  hipEventElapsedTime(&elapsed, start, end); 
  
  if(pinned)
    hipHostFree(h);
  else
    delete[] h;
    
    
  hipFree(d);
  hipEventDestroy(start);
  hipEventDestroy(end);  
  
  return elapsed;
}

// ==============================
int main()
{
printf(" starts \n");

std::cout << "From device - paged memory: \t" << timing(false, false) << std::endl;
std::cout << "To device -   paged memory: \t"   << timing(false, true) << std::endl;
std::cout << "From device - pinned memory: \t" << timing(true, false) << std::endl;
std::cout << "To device -   pinned memory: \t"   << timing(true, true) << std::endl;

printf(" done \n");
return 0;
} 
