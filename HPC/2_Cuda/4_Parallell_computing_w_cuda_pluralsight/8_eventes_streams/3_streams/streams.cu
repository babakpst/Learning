#include "hip/hip_runtime.h"

/*
Babak Poursartip
02/27/2021

CUDA
topic: stream.

- Instead of using malloc or new to allocation memory on the CPU(host), we use hipHostAlloc(). This will allocate a pinned memory on the host.

- To free the memory, we use hipHostFree, instead of delete to deallocate. 

- The disadvantage is that you cannot swap the memory to the disk & we need to have enough memory to use this memory. 

*/

#include <iostream>
#include <cmath>
#include <ctime>


// ==============================
const int chunkCount = 1 << 20;
const int totalCount = chunkCount << 3;

__global__ void kernel(float *a, float *b, float *c)
{
  int tid = blockDim.x * blockIdx.x+threadIdx.x;
  if (tid < chunkCount)
    c[tid] = erff(a[tid]+b[tid]);
 
}

// ==============================
int main()
{
printf(" starts \n");


hipDeviceProp_t prop;
int device;

hipGetDevice(&device);
hipGetDeviceProperties(&prop, device);

if (!prop.deviceOverlap)
{
  return 0;
}


hipEvent_t start, end;
hipEventCreate(&start);
hipEventCreate(&end);

hipStream_t stream;

hipStreamCreate(&stream);

float *ha, *hb, *hc, *da, *db, *dc;

const int totalSize = totalCount * sizeof(float);
const int chunkSize = chunkCount * sizeof(float);

hipMalloc(&da, chunkSize);
hipMalloc(&db, chunkSize);
hipMalloc(&dc, chunkSize);

hipHostAlloc(&ha, totalSize, hipHostMallocDefault);
hipHostAlloc(&hb, totalSize, hipHostMallocDefault);
hipHostAlloc(&hc, totalSize, hipHostMallocDefault);


srand((unsigned)time(0));

// random numbers for the two input vectors
for (int i = 0; i < totalCount; ++i)
{
  ha[i] = rand() / RAND_MAX;
  hb[i] = rand() / RAND_MAX;
}

hipEventRecord(start, stream);

for (int i = 0; i < totalCount; i += chunkCount)
{
  hipMemcpyAsync(da, ha+i, chunkSize, hipMemcpyHostToDevice, stream);
  hipMemcpyAsync(db, ha+i, chunkSize, hipMemcpyHostToDevice, stream);  
  kernel<<<chunkCount/64,64,0,stream>>>(da, db, dc);
  hipMemcpyAsync(hc+i, dc, chunkSize, hipMemcpyHostToDevice, stream);    
}

hipStreamSynchronize(stream);
hipEventRecord(end, stream);
hipEventSynchronize(end);

float elapsed;
hipEventElapsedTime(&elapsed, start, end);

std::cout << " it took(ms): " << elapsed << std::endl;

hipHostFree(ha);
hipHostFree(hb);
hipHostFree(hc);

hipFree(da);
hipFree(db);
hipFree(dc);

hipStreamDestroy(stream);

printf(" done \n");
return 0;
} 
