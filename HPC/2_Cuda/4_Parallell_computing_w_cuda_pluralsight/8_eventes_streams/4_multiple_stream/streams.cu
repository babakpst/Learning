#include "hip/hip_runtime.h"

/*
Babak Poursartip
02/27/2021

CUDA
topic: stream.

- Instead of using malloc or new to allocation memory on the CPU(host), we use hipHostAlloc(). This will allocate a pinned memory on the host.

- To free the memory, we use hipHostFree, instead of delete to deallocate. 

- The disadvantage is that you cannot swap the memory to the disk & we need to have enough memory to use this memory. 

*/

#include <iostream>
#include <cmath>
#include <ctime>


// ==============================
const int chunkCount = 1 << 20;
const int totalCount = chunkCount << 3;

__global__ void kernel(float *a, float *b, float *c)
{
  int tid = blockDim.x * blockIdx.x+threadIdx.x;
  if (tid < chunkCount)
    c[tid] = erff(a[tid]+b[tid]);
 
}

// ==============================
int main()
{
printf(" starts \n");


hipDeviceProp_t prop;
int device;

hipGetDevice(&device);
hipGetDeviceProperties(&prop, device);

if (!prop.deviceOverlap)
{
  return 0;
}


hipEvent_t start, end;
hipEventCreate(&start);
hipEventCreate(&end);

hipStream_t stream1, stream2;

hipStreamCreate(&stream1);
hipStreamCreate(&stream2);

float *ha, *hb, *hc, *da1, *db1, *dc1, *da2, *db2, *dc2;

const int totalSize = totalCount * sizeof(float);
const int chunkSize = chunkCount * sizeof(float);

hipMalloc(&da1, chunkSize);
hipMalloc(&db1, chunkSize);
hipMalloc(&dc1, chunkSize);
hipMalloc(&da2, chunkSize);
hipMalloc(&db2, chunkSize);
hipMalloc(&dc2, chunkSize);

hipHostAlloc(&ha, totalSize, hipHostMallocDefault);
hipHostAlloc(&hb, totalSize, hipHostMallocDefault);
hipHostAlloc(&hc, totalSize, hipHostMallocDefault);


srand((unsigned)time(0));

// random numbers for the two input vectors
for (int i = 0; i < totalCount; ++i)
{
  ha[i] = rand() / RAND_MAX;
  hb[i] = rand() / RAND_MAX;
}

hipEventRecord(start, stream1);

for (int i = 0; i < totalCount; i += chunkCount*2)
{

  /*
  if ((i%2) == 0)
  {
  hipMemcpyAsync(da1, ha+i, chunkSize, hipMemcpyHostToDevice, stream1);
  hipMemcpyAsync(db1, ha+i, chunkSize, hipMemcpyHostToDevice, stream1);  
  kernel<<<chunkCount/64,64,0,stream1>>>(da1, db1, dc1);
  hipMemcpyAsync(hc+i, dc1, chunkSize, hipMemcpyHostToDevice, stream1);  
  }
  else
  {
  hipMemcpyAsync(da2, ha+i, chunkSize, hipMemcpyHostToDevice, stream2);
  hipMemcpyAsync(db2, ha+i, chunkSize, hipMemcpyHostToDevice, stream2);  
  kernel<<<chunkCount/64,64,0,stream2>>>(da2, db2, dc2);
  hipMemcpyAsync(hc+i, dc2, chunkSize, hipMemcpyHostToDevice, stream2);  
  }
  */
  hipMemcpyAsync(da1, ha+i, chunkSize, hipMemcpyHostToDevice, stream1);
  hipMemcpyAsync(da2, ha+i+chunkCount, chunkSize, hipMemcpyHostToDevice, stream2);
  hipMemcpyAsync(db1, ha+i, chunkSize, hipMemcpyHostToDevice, stream1);  
  hipMemcpyAsync(db2, ha+i+chunkCount, chunkSize, hipMemcpyHostToDevice, stream2);  
  kernel<<<chunkCount/64,64,0,stream1>>>(da1, db1, dc1);
  kernel<<<chunkCount/64,64,0,stream2>>>(da2, db2, dc2);
  hipMemcpyAsync(hc+i, dc1, chunkSize, hipMemcpyHostToDevice, stream1);
  hipMemcpyAsync(hc+i+chunkCount, dc2, chunkSize, hipMemcpyHostToDevice, stream2);

}

hipStreamSynchronize(stream1);
hipStreamSynchronize(stream2);

hipEventRecord(end, stream1);
hipEventSynchronize(end);

float elapsed;
hipEventElapsedTime(&elapsed, start, end);

std::cout << " it took(ms): " << elapsed << std::endl;

hipHostFree(ha);
hipHostFree(hb);
hipHostFree(hc);

hipFree(da1);
hipFree(db1);
hipFree(dc1);

hipFree(da2);
hipFree(db2);
hipFree(dc2);

hipStreamDestroy(stream1);
hipStreamDestroy(stream2);

printf(" done \n");
return 0;
} 
