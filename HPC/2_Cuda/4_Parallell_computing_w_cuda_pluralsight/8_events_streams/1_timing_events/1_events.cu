#include "hip/hip_runtime.h"

// Babak Poursartip
// 02/28/2021

// CUDA
//topic: event- measure the time in CUDA.

#include <iostream>
#include "sm_20_atomic_functions.h"

// ==============================
__device__ int dSum=0;

__global__ void sumArray(int* d)
{
  int tid = threadIdx.x;
  dSum +=d[tid];

}

// ==============================
__global__ void sumArrayAtomic(int* d)
{
  int tid = threadIdx.x;
  atomicAdd(&dSum, d[tid]);

}

// ==============================
int main()
{
printf(" starts \n");

const int count = 256;
const int size = count * sizeof(int);

int h[count];

for (int i = 0; i < count; ++i) h[i] = i+1;

int* d;
hipMalloc(&d, size);
hipMemcpy(d,h, size, hipMemcpyHostToDevice);

int Sum;
sumArray<<<1, count>>>(d);
hipMemcpyFromSymbol(&Sum, HIP_SYMBOL(dSum), sizeof(int));
std::cout << " sum is: " << Sum << std::endl;


// adding the events to this kernel <<<<<<<<<<<<<<<<<<<<<<<<<
hipEvent_t start, end;
hipEventCreate(&start);
hipEventCreate(&end);

hipEventRecord(start);

sumArrayAtomic<<<1, count>>>(d);

hipEventRecord(end);
hipEventSynchronize(end);

float elapsed;
hipEventElapsedTime(&elapsed, start, end); // converts the event into milisecond.

// <<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<

hipMemcpyFromSymbol(&Sum, HIP_SYMBOL(dSum), sizeof(int));
std::cout << " sum is: " << Sum << std::endl;

std::cout << " The operation took: " << elapsed << "ms\n";

hipFree(d);

printf(" done \n");
return 0;
} 
