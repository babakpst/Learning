
// Babak Poursartip
// 02/14/2021

// CUDA
//topic: scan



#include <hip/hip_runtime.h>
#include <cstdio>
#include <ctime>
#include <iostream>



// ==============================
__global__ void sum(int *d)
{
  int tds = blockDim.x;
  int tid = threadIdx.x;
  // tc: total number of threads

  for (int tc = tds, stepSize = 1; tc > 0; tc /=2, stepSize *=2) // changes the number of threads by half(tc>>=1)
  {
    // thread must be allowed to write
    if (tid < tc)
    {
      d[tid+stepSize] += d[tid];

# if __CUDA_ARCH__>=200
      printf("%d, %d, %d, %d \n", tds, tid, stepSize, tc);
#endif
    }    
    tc -=stepSize;
  }
}


// ==============================
int main()
{
printf(" starts \n");

const int count  = 16;
const int size = count * sizeof(int);

int h[count];

for (int i = 0; i < count; ++i)
  h[i] = i + 1;
  
int *d;
hipMalloc(&d, size);
hipMemcpy(d, h, size, hipMemcpyHostToDevice);

sum<<<1, count-1>>>(d);


hipMemcpy(h, d, size, hipMemcpyDeviceToHost);

for (int i = 0; i < count;  ++i)
  std::cout << h[i] << " ";
  std::cout << std::endl;

hipFree(d);

printf(" done \n");
return 0;
} 
