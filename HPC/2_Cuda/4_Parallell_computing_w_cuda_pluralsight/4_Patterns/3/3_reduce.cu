

// Babak Poursartip
// 02/14/2021

// CUDA
//topic: gather



#include <hip/hip_runtime.h>
#include <cstdio>
#include <ctime>
#include <iostream>

#include <hiprand/hiprand.h>

// ==============================
__global__ void sumSingleBlock(int *d)
{
  int tid = threadIdx.x;
  // tc: number of participating threads
  //for (int tc = blockDim.x; tc > 0; tc >>=1) // changes the number of threads by half(tc>>=1)
  for (int tc = blockDim.x, stepSize = 1; tc > 0; tc /=2, stepSize *=2) // changes the number of threads by half(tc>>=1)
  {
    // thread must be allowed to write
    if (tid < tc)
    {
      int pa =  tid * stepSize * 2;
      int pb = pa + stepSize;
      d[pa] += d[pb];

# if __CUDA_ARCH__>=200
    printf("%d, %d, %d, %d, %d \n", tid, tc, stepSize, pa, pb);

#endif
    }    
  }
}


// ==============================
int main()
{
printf(" starts \n");

const int count  = 32;
const int size = count * sizeof(int);

int h[count];

for (int i = 0; i < count; ++i)
  h[i] = i + 1;
  
int *d;
hipMalloc(&d, size);
hipMemcpy(d, h, size, hipMemcpyHostToDevice);

sumSingleBlock<<<1, count/2>>>(d);

int result;
hipMemcpy(&result, d, sizeof(int), hipMemcpyDeviceToHost);
std::cout << " sum: "<< result << std::endl;

hipFree(d);

printf(" done \n");
return 0;
} 
