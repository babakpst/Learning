
// Babak Poursartip
// 02/6/2021

// CUDA
//topic: gather
// y = f(a_m, b_n, ... )

// black scholes (option price) 

#define _USE_MATH_DEFINES

#include <hip/hip_runtime.h>
#include "cstdio"
#include "hiprand/hiprand.h"
#include <math.h> 

// ====================================
__device__ __host__ __inline__ float N(float x)
{
  return 0.5+0.5*erf(x* M_SQRT1_2);
}

// ====================================
__device__ __host__ void price(float k, float s, float t, float r, float v, float *c, float *p)
{
	float srt = v * sqrtf(t);
	float d1 = (logf(s/k)+(r+0.5*v*v)*t) / srt;
	float d2 = d1 - srt;
	float kert = k * expf(-r*t);
	*c = N(d1)*s - N(d2)*kert;
	*p = kert - s + *c;
}


// ====================================
__global__ void price(float *k, float *s, float *t, float *r, float *v, float *c, float *p )
{
  int idx = threadIdx.x;
  price(k[idx], s[idx], t[idx], r[idx], v[idx], &c[idx], &p[idx]);
}


// ====================================
int main(){

printf(" starts ... \n");

  const int count = 512;  //no of elements
  const int size = count * sizeof(float);
  float *args[5];

  hiprandGenerator_t gen;
  hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_MTGP32);

    for (int i = 0; i < 5; ++i)
    {
      hipMalloc(&args[i], size);
      hiprandGenerateUniform(gen, args[i], count);
    }

  float *dc, *dp;
  hipMalloc(&dc, size);
  hipMalloc(&dp, size);

  price<<<1, count>>>(args[0], args[1], args[2], args[3], args[4], dc, dp);


printf(" done. \n");
return 0;
}



