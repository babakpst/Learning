
#include <cstdio>
#include <ctime>
#include <iostream>

#include "hiprand/hiprand.h"
#include "hip/hip_runtime.h"


// ==============================
__global__ void add10(float *d, int count)
{
int threadsPerBlock        = blockDim.x * blockDim.y * blockDim.z;
int threadPositionInBlock = threadIdx.x +
                            blockDim.x * threadIdx.y +
                            blockDim.x * blockDim.y * threadIdx.z;

int blockPositionInGrid   = blockIdx.x +
                            gridDim.x * blockIdx.y +
                            gridDim.x * gridDim.y * blockIdx.z;

int tid = blockPositionInGrid * threadsPerBlock + threadPositionInBlock;
if (tid < count)
  d[tid] += 11;

}

// ==============================
int main()
{
printf(" starts \n");

hiprandGenerator_t gen;
hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_MTGP32);
hiprandSetPseudoRandomGeneratorSeed(gen, time(0));

const int count = 123456;
const int size = count * sizeof(float);
float *d;
float h[count];

hipMalloc(&d, size);
hiprandGenerateUniform(gen, d, count);

dim3 block(8,8,8); 
dim3 grid(16,16);

add10<<<grid,block>>>(d, count);

hipMemcpy(h, d, size, hipMemcpyDeviceToHost);
hipFree(d);

for (int i = 0; i < 100; ++i)
  printf(" %f ", h[i]);

printf(" done \n");
return 0;
}
