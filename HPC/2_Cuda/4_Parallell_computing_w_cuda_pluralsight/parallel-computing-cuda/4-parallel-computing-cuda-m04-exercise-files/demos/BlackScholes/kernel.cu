
#include "hip/hip_runtime.h"


#define _USE_MATH_DEFINES
#include <iostream>
#include <math.h>
#include <hiprand/hiprand.h>
using namespace std;







__device__ __host__ __inline__ float N(float x)
{
	return 0.5 + 0.5*erf(x*M_SQRT1_2);
}


__device__ __host__ void price(float k, float s, float t, float r, float v, float* c, float* p)
{
	float srt = v * sqrtf(t);
	float d1 = (logf(s/k)+(r+0.5*v*v)*t) / srt;
	float d2 = d1 - srt;
	float kert = k * expf(-r*t);
	*c = N(d1)*s - N(d2)*kert;
	*p = kert - s + *c;
}



__global__ void price(float* k, float* s, float* t, float* r, float* v, float* c, float* p)
{
	int idx = threadIdx.x;
	price(k[idx], s[idx], t[idx], r[idx], v[idx], &c[idx], &p[idx]);
}



int main()
{


	const int count = 512;
	hiprandStatus_t hiprandStatus;
	hipError_t hipError_t;
	float* args[5];
	hiprandGenerator_t gen;
	hiprandStatus = hiprandCreateGenerator(&gen, hiprandRngType_t::HIPRAND_RNG_PSEUDO_MTGP32);
	for (int i = 0; i < 5; ++i) 
	{
		hipMalloc(&args[i], sizeof(float)*1024);
		hiprandStatus = hiprandGenerateUniform(gen, args[i], count);
	}

	float *dc, *dp;
	hipError_t = hipMalloc(&dc, count*sizeof(float));
	hipError_t = hipMalloc(&dp, count*sizeof(float));

	price<<<1,count>>>(args[0], args[1], args[2], args[3], args[4], dc, dp);

	float hc[count] = { 0 };
	float hp[count] = { 0 };
	hipMemcpy(hc, dc, sizeof(float)*count, hipMemcpyKind::hipMemcpyDeviceToHost);
	hipMemcpy(hp, dp, sizeof(float)*count, hipMemcpyKind::hipMemcpyDeviceToHost);

	hipFree(dc);
	hipFree(dp);
	for (int i = 0; i < 5; ++i)
		hipFree(&args[i]);
	hipDeviceReset();
  return 0;
}

