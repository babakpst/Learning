#include "hip/hip_runtime.h"

#include "hiprand/hiprand.h"

#include <ctime>
#include <cstdio>
#include <iostream>
using namespace std;

__global__ void addTen(float* d, int count)
{
  int threadsPerBlock = blockDim.x * blockDim.y * blockDim.z;
  int threadPosInBlock = threadIdx.x + blockDim.x * threadIdx.y + blockDim.x * blockDim.y * threadIdx.z;
  int blockPosInGrid = blockIdx.x + gridDim.x * blockIdx.y + gridDim.x * gridDim.y * blockIdx.z;
  int tid = blockPosInGrid * threadsPerBlock + threadPosInBlock;
  if (tid < count)
    d[tid] = d[tid] + 10;
}

int main()
{
  hiprandGenerator_t gen;
  hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_MTGP32);
	hiprandSetPseudoRandomGeneratorSeed(gen, time(0));

	hipError_t status;

  const int count = 123456;
  const int size = count * sizeof(float);
  float* d;
  float h[count];
  hipMalloc(&d, size);
  hiprandGenerateUniform(gen, d, count);

  dim3 block(8,8,8);
  dim3 grid(16,16);
  addTen<<<grid,block>>>(d,count);

  status = hipMemcpy(h,d,size,hipMemcpyDeviceToHost);

  hipFree(d);

  for (int i = 0; i < 100; ++i)
    cout << h[i] << '\t';

  getchar();
}
