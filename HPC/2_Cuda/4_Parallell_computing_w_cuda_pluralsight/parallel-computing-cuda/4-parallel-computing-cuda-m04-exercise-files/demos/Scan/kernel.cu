
#include "hip/hip_runtime.h"


#include <iostream>
using namespace std;

__global__ void runningSum(int* d)
{
  int threads = blockDim.x;
  int tid = threadIdx.x;

  for (int tc = threads, step = 1; tc > 0; step *= 2)
  {
    // check this thread is allowed to do things
    if (tid < tc)
    {
      d[tid+step] += d[tid];
    }
    tc -= step;
  }
}

int main()
{
  const int count = 16;
  const int size = count * sizeof(int);

  int h[count];
  for (int i = 0; i < count; ++i)
    h[i] = i + 1;

  int* d;
  hipMalloc(&d, size);
  hipMemcpy(d,h,size,hipMemcpyHostToDevice);

  runningSum<<<1,count-1>>>(d);

  hipMemcpy(h,d,size,hipMemcpyDeviceToHost);
  

  for (int i = 0; i < count; ++i)
    cout << h[i] << '\t';

  hipFree(d);

  getchar();

  return 0;
}