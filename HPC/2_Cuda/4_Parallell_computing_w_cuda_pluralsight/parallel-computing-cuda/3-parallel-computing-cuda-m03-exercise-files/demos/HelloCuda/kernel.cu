#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>

void check(hipError_t e)
{
	if (e != hipSuccess)
	{
		printf(hipGetErrorString(e));
	}
}

__global__ void addArraysGPU(int* a, int* b, int* c)
{
  int i = threadIdx.x;
  c[i] = a[i] + b[i];
}

void main()
{
  const int count = 5;
  int ha[] = { 1, 2, 3, 4, 5 };
  int hb[] = { 10, 20, 30, 40, 50 };
  int hc[count];

  int *da, *db, *dc;
  int size = sizeof(int)*count;

  hipMalloc(&da, size);
  hipMalloc(&db, size);
  hipMalloc(&dc, size);

  hipMemcpy(da,ha,size,hipMemcpyHostToDevice);
  hipMemcpy(db,hb,size,hipMemcpyHostToDevice);

  addArraysGPU<<<1,count>>>(da,db,dc);

  hipMemcpy(hc,dc,size,hipMemcpyDeviceToHost);

  printf("%d %d %d %d %d",
	  hc[0],
	  hc[1],
	  hc[2],
	  hc[3],
	  hc[4]);

  hipFree(da);
  hipFree(db);
  hipFree(dc);

  //getchar();

}