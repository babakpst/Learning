#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#define _USE_MATH_DEFINES
#include <math.h>
#include <Windows.h>

// assume that the bitmap is rectangular, stride is neglected

__global__ void kernel(unsigned char* src)
{
  __shared__ float temp[16][16];

  int x = threadIdx.x + blockIdx.x * blockDim.x;
  int y = threadIdx.y + blockIdx.y * blockDim.y;
  int offset = x + y * blockDim.x * gridDim.x;
  const float period = 128.0f;
  temp[threadIdx.x][threadIdx.y] = 255 * 
    (sinf(x*2.0f*M_PI/period) + 1.0f) *
    (sinf(y*2.0f*M_PI/period) + 1.0f) / 4.0f;

  // comment out line below to see image break
  __syncthreads();

  src[offset*4] = 0;
  src[offset*4+1] = temp[15-threadIdx.x][15-threadIdx.y];
  src[offset*4+2] = 0;
  src[offset*4+3] = 255;
}

extern "C" __declspec(dllexport) void GenerateBitmap(unsigned char* dst, int dimension)
{
  int size = dimension * dimension * 4; // assume RGBA
  hipError_t status;

  // allocate as much memory
  unsigned char* src;
  status = hipMalloc(&src, size);
  
  dim3 blocks(dimension/16, dimension/16);
  dim3 threads(16,16);
  kernel<<<blocks,threads>>>(src);

  hipMemcpy(dst, src, size, hipMemcpyDeviceToHost);
  hipFree(src);
}

BOOL APIENTRY DllMain( HMODULE hModule,
                       DWORD  ul_reason_for_call,
                       LPVOID lpReserved
					 )
{
	switch (ul_reason_for_call)
	{
	case DLL_PROCESS_ATTACH:
	case DLL_THREAD_ATTACH:
	case DLL_THREAD_DETACH:
	case DLL_PROCESS_DETACH:
		break;
	}
	return TRUE;
}