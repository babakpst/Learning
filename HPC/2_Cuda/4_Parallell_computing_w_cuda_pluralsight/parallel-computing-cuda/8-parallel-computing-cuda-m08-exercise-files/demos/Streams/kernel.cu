#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <cmath>
#include <ctime>
using namespace std;

const int chunkCount = 1 << 20;
const int totalCount = chunkCount << 3;

// does c = erff(a+b)
__global__ void kernel(float* a, float* b, float* c)
{
  int tid = blockDim.x*blockIdx.x + threadIdx.x;
  if (tid < chunkCount)
    c[tid] = erff(a[tid] + b[tid]);
}



// 88 msec
int main1()
{
  hipDeviceProp_t prop;
  int device;
  hipGetDevice(&device);
  hipGetDeviceProperties(&prop, device);
  if (!prop.deviceOverlap)
  {
    cout << "This won't work" << endl;
    return 0;
  }

  hipEvent_t start, end;
  hipEventCreate(&start);
  hipEventCreate(&end);

  hipStream_t stream;
  hipStreamCreate(&stream);

  float *ha, *hb, *hc, *da, *db, *dc;
  const int totalSize = totalCount * sizeof(float);
  const int chunkSize = chunkCount * sizeof(float);

  // allocate memory
  hipMalloc(&da, chunkSize);
  hipMalloc(&db, chunkSize);
  hipMalloc(&dc, chunkSize);
  hipHostAlloc(&ha, totalSize, hipHostMallocDefault);
  hipHostAlloc(&hb, totalSize, hipHostMallocDefault);
  hipHostAlloc(&hc, totalSize, hipHostMallocDefault);

  // fill a and b with noise
  srand((unsigned)time(0));
  for (int i = 0; i < totalCount; i++)
  {
    ha[i] = rand() / RAND_MAX;
    hb[i] = rand() / RAND_MAX;
  }

  hipEventRecord(start, stream);

  for (int i = 0; i < totalCount; i+= chunkCount)
  {
    hipMemcpyAsync(da,ha+i,chunkSize,hipMemcpyHostToDevice,stream);
    hipMemcpyAsync(db,hb+i,chunkSize,hipMemcpyHostToDevice,stream);
    kernel<<<chunkCount/64,64,0,stream>>>(da,db,dc);
    hipMemcpyAsync(hc+i,dc,chunkSize,hipMemcpyDeviceToHost,stream);
  }

  hipStreamSynchronize(stream);

  hipEventRecord(end, 0);
  hipEventSynchronize(end);
  float elapsed;
  hipEventElapsedTime(&elapsed, start, end);

  cout << "This took " << elapsed << " msec" << endl;

  hipHostFree(ha);
  hipHostFree(hb);
  hipHostFree(hc);
  hipFree(da);
  hipFree(db);
  hipFree(dc);
  hipStreamDestroy(stream);

  getchar();
}

int main()
{
  hipDeviceProp_t prop;
  int device;
  hipGetDevice(&device);
  hipGetDeviceProperties(&prop, device);
  if (!prop.deviceOverlap)
  {
    cout << "This won't work" << endl;
    return 0;
  }

  hipEvent_t start, end;
  hipEventCreate(&start);
  hipEventCreate(&end);

  hipStream_t stream1, stream2;
  hipStreamCreate(&stream1);
  hipStreamCreate(&stream2);

  float *ha, *hb, *hc, *d1a, *d1b, *d1c, *d2a, *d2b, *d2c;
  const int totalSize = totalCount * sizeof(float);
  const int chunkSize = chunkCount * sizeof(float);

  // allocate memory
  hipMalloc(&d1a, chunkSize);
  hipMalloc(&d1b, chunkSize);
  hipMalloc(&d1c, chunkSize);
  hipMalloc(&d2a, chunkSize);
  hipMalloc(&d2b, chunkSize);
  hipMalloc(&d2c, chunkSize);
  hipHostAlloc(&ha, totalSize, hipHostMallocDefault);
  hipHostAlloc(&hb, totalSize, hipHostMallocDefault);
  hipHostAlloc(&hc, totalSize, hipHostMallocDefault);

  // fill a and b with noise
  srand((unsigned)time(0));
  for (int i = 0; i < totalCount; i++)
  {
    ha[i] = rand() / RAND_MAX;
    hb[i] = rand() / RAND_MAX;
  }

  hipEventRecord(start, stream1);

  for (int i = 0; i < totalCount; i += chunkCount*2)
  {
    hipMemcpyAsync(d1a,ha+i,chunkSize,hipMemcpyHostToDevice,stream1);
    hipMemcpyAsync(d1b,hb+i,chunkSize,hipMemcpyHostToDevice,stream1);
    hipMemcpyAsync(d2a,ha+i+chunkCount,chunkSize,hipMemcpyHostToDevice,stream2);
    hipMemcpyAsync(d2b,hb+i+chunkCount,chunkSize,hipMemcpyHostToDevice,stream2);
    kernel<<<chunkCount/64,64,0,stream1>>>(d1a,d1b,d1c);
    kernel<<<chunkCount/64,64,0,stream1>>>(d2a,d2b,d2c);
    hipMemcpyAsync(hc+i,d1c,chunkSize,hipMemcpyDeviceToHost,stream1);
    hipMemcpyAsync(hc+i+chunkCount,d2c,chunkSize,hipMemcpyDeviceToHost,stream2);
  }

  hipStreamSynchronize(stream1);
  hipStreamSynchronize(stream2);

  hipEventRecord(end, 0);
  hipEventSynchronize(end);
  float elapsed;
  hipEventElapsedTime(&elapsed, start, end);

  cout << "This took " << elapsed << " msec" << endl;

  hipHostFree(ha);
  hipHostFree(hb);
  hipHostFree(hc);
  hipFree(d1a);
  hipFree(d1b);
  hipFree(d1c);
  hipFree(d2a);
  hipFree(d2b);
  hipFree(d2c);
  hipStreamDestroy(stream1);
  hipStreamDestroy(stream2);

  getchar();
}

// also 88 msec :(
int main2()
{
  hipDeviceProp_t prop;
  int device;
  hipGetDevice(&device);
  hipGetDeviceProperties(&prop, device);
  if (!prop.deviceOverlap)
  {
    cout << "This won't work" << endl;
    return 0;
  }

  hipEvent_t start, end;
  hipEventCreate(&start);
  hipEventCreate(&end);

  hipStream_t stream1, stream2;
  hipStreamCreate(&stream1);
  hipStreamCreate(&stream2);

  float *ha, *hb, *hc, *d1a, *d1b, *d1c, *d2a, *d2b, *d2c;
  const int totalSize = totalCount * sizeof(float);
  const int chunkSize = chunkCount * sizeof(float);

  // allocate memory
  hipMalloc(&d1a, chunkSize);
  hipMalloc(&d1b, chunkSize);
  hipMalloc(&d1c, chunkSize);
  hipMalloc(&d2a, chunkSize);
  hipMalloc(&d2b, chunkSize);
  hipMalloc(&d2c, chunkSize);
  hipHostAlloc(&ha, totalSize, hipHostMallocDefault);
  hipHostAlloc(&hb, totalSize, hipHostMallocDefault);
  hipHostAlloc(&hc, totalSize, hipHostMallocDefault);

  // fill a and b with noise
  srand((unsigned)time(0));
  for (int i = 0; i < totalCount; i++)
  {
    ha[i] = rand() / RAND_MAX;
    hb[i] = rand() / RAND_MAX;
  }

  hipEventRecord(start, stream1);

  for (int i = 0; i < totalCount; i+= chunkCount)
  {
    if ((i % 2) == 0)
    {
      hipMemcpyAsync(d1a,ha+i,chunkSize,hipMemcpyHostToDevice,stream1);
      hipMemcpyAsync(d1b,hb+i,chunkSize,hipMemcpyHostToDevice,stream1);
      kernel<<<chunkCount/64,64,0,stream1>>>(d1a,d1b,d1c);
      hipMemcpyAsync(hc+i,d1c,chunkSize,hipMemcpyDeviceToHost,stream1);
    } 
    else 
    {
      hipMemcpyAsync(d2a,ha+i,chunkSize,hipMemcpyHostToDevice,stream2);
      hipMemcpyAsync(d2b,hb+i,chunkSize,hipMemcpyHostToDevice,stream2);
      kernel<<<chunkCount/64,64,0,stream2>>>(d2a,d2b,d2c);
      hipMemcpyAsync(hc+i,d2c,chunkSize,hipMemcpyDeviceToHost,stream2);
    }
  }

  hipStreamSynchronize(stream1);
  hipStreamSynchronize(stream2);

  hipEventRecord(end, 0);
  hipEventSynchronize(end);
  float elapsed;
  hipEventElapsedTime(&elapsed, start, end);

  cout << "This took " << elapsed << " msec" << endl;

  hipHostFree(ha);
  hipHostFree(hb);
  hipHostFree(hc);
  hipFree(d1a);
  hipFree(d1b);
  hipFree(d1c);
  hipFree(d2a);
  hipFree(d2b);
  hipFree(d2c);
  hipStreamDestroy(stream1);
  hipStreamDestroy(stream2);

  getchar();
}