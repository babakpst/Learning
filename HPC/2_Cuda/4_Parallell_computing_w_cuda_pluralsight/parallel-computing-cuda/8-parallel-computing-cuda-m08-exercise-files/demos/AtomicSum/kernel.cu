#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include "sm_20_atomic_functions.h"

#include <iostream>
using namespace std;

__device__ int dSum = 0;

__global__ void sum(int* d)
{
  int tid = threadIdx.x;
  //dSum += d[tid];
  atomicAdd(&dSum, d[tid]);
}

int main()
{
  const int count = 256;
  const int size = count * sizeof(int);

  int h[count];
  for (int i = 0; i < count; ++i)
    h[i] = i+1;

  int* d;
  hipMalloc(&d, size);
  hipMemcpy(d,h,size,hipMemcpyHostToDevice);

  hipEvent_t start, end;
  hipEventCreate(&start);
  hipEventCreate(&end);

  hipEventRecord(start);

  sum<<<1,count>>>(d);

  hipEventRecord(end);
  hipEventSynchronize(end);

  float elapsed;
  hipEventElapsedTime(&elapsed, start, end);

  int hSum;
  hipMemcpyFromSymbol(&hSum, HIP_SYMBOL(dSum), sizeof(int));
  cout << "The sum of numbers from 1 to " << count 
    << " is " << hSum << " and it took " << elapsed << " msec" << endl;
  getchar();

  hipFree(d);

  return 0;
}