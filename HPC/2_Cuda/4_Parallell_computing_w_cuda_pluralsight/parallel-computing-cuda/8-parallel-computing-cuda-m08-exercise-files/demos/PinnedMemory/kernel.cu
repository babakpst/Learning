#include "hip/hip_runtime.h"

#include <iostream>
using namespace std;

float timeMemory(bool pinned, bool toDevice)
{
  const int count = 1 << 20;
  const int iterations = 1 << 6;
  const int size = count * sizeof(int);
  hipEvent_t start, end;
  int *h, *d;
  float elapsed;
  hipError_t status;

  hipEventCreate(&start);
  hipEventCreate(&end);

  hipMalloc(&d, size);
  if (pinned)
    hipHostAlloc(&h, size, hipHostMallocDefault);
  else
    h = new int[count];

  hipEventRecord(start);

  for (int i = 0; i < iterations; i++)
  {
    if (toDevice)
      status = hipMemcpy(d, h, size, hipMemcpyHostToDevice);
    else
      status = hipMemcpy(h, d, size, hipMemcpyDeviceToHost);
  }

  hipEventRecord(end);
  hipEventSynchronize(end);
  hipEventElapsedTime(&elapsed, start, end);

  if (pinned)
    hipHostFree(h);
  else
    delete [] h;

  hipFree(d);
  hipEventDestroy(start);
  hipEventDestroy(end);

  return elapsed;
}

int main()
{
  cout << "From device, paged memory:\t" << timeMemory(false, false) << endl;
  cout << "To device,   paged memory:\t" << timeMemory(false, true) << endl;
  cout << "From device, pinned memory:\t" << timeMemory(true, false) << endl;
  cout << "To device,   pinned memory:\t" << timeMemory(true, true) << endl;

  getchar();
  return 0;
}