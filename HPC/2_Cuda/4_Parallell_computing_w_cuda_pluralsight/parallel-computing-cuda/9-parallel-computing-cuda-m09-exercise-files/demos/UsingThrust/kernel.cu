#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/sort.h>
using namespace thrust;

#include <ctime>
using namespace std;

int myrand()
{
  return rand() % 10;
}

void main()
{
  int count = 1024;
  host_vector<int> h(count);
  generate(begin(h), end(h), myrand);
  device_vector<int> d = h;
  sort(begin(d), end(d));
  h = d;
  for (int i = 0; i < count; i++)
  {
    cout << h[i] << "\t";
  }

  getchar();
}