#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <cmath>
using namespace std;

__device__ __inline__ float trim(unsigned char value)
{
  return fminf((unsigned char)255, fmaxf(value, (unsigned char)0));
}

__device__ __inline__ float poly(float x, float a, float b, float c)
{
  return a*x*x*x+b*x*x+c*x;
}

__global__ void kernel(unsigned char* img, const float* a)
{
  int ix = blockIdx.x;
  int iy = threadIdx.x;
  int tid = iy*blockDim.x + ix;

  float x = (float)ix / blockDim.x;
  float y = (float)iy / gridDim.x;

  //placeholder

  img[tid*4+0] = trim(poly(z,a[0],a[1],a[2]) * 255.0f);
  img[tid*4+1] = trim(poly(z,a[3],a[4],a[5]) * 255.0f);
  img[tid*4+2] = trim(poly(z,a[6],a[7],a[8]) * 255.0f);
  img[tid*4+3] = 255;
}