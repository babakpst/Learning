#include "hip/hip_runtime.h"

#include <cmath>
using namespace std;

__device__ __inline__ float trim(unsigned char value)
{
  return fminf((unsigned char)255, fmaxf(value, (unsigned char)0));
}

__device__ __inline__ float poly(float x, float a, float b, float c)
{
  return a*x*x*x+b*x*x+c*x;
}

__global__ void kernel(unsigned char* img, const float* a)
{
  int ix = blockIdx.x;
  int iy = threadIdx.x;
  int tid = iy*blockDim.x + ix;

  float x = (float)ix / blockDim.x;
  float y = (float)iy / gridDim.x;

  double z = powf(x,powf(sinhf(((((0.546098f < x ? 0.546098f : x) < sinf(x) ? (0.546098f < x ? 0.546098f : x) : sinf(x))-(sinf(y)+(0.205267f > 0.0f ? log(0.205267f) : -log(-0.205267f)))) < (sinhf((y*x)) == 0.0f ? x : 1.0f/sinhf((y*x))) ? (((0.546098f < x ? 0.546098f : x) < sinf(x) ? (0.546098f < x ? 0.546098f : x) : sinf(x))-(sinf(y)+(0.205267f > 0.0f ? log(0.205267f) : -log(-0.205267f)))) : (sinhf((y*x)) == 0.0f ? x : 1.0f/sinhf((y*x))))),sinhf((powf((powf(x,0.285562f) > 0.0f ? sqrt(powf(x,0.285562f)) : -sqrt(-powf(x,0.285562f))),((y-x)+(y == 0.f ? x : x/y)))+(((y > 0.0f ? log(y) : -log(-y)) == 0.f ? coshf(x) : coshf(x)/(y > 0.0f ? log(y) : -log(-y))) < ((y == 0.f ? x : x/y) == 0.f ? (x*y) : (x*y)/(y == 0.f ? x : x/y)) ? ((y > 0.0f ? log(y) : -log(-y)) == 0.f ? coshf(x) : coshf(x)/(y > 0.0f ? log(y) : -log(-y))) : ((y == 0.f ? x : x/y) == 0.f ? (x*y) : (x*y)/(y == 0.f ? x : x/y)))))));

  img[tid*4+0] = trim(poly(z,a[0],a[1],a[2]) * 255.0f);
  img[tid*4+1] = trim(poly(z,a[3],a[4],a[5]) * 255.0f);
  img[tid*4+2] = trim(poly(z,a[6],a[7],a[8]) * 255.0f);
  img[tid*4+3] = 255;
}