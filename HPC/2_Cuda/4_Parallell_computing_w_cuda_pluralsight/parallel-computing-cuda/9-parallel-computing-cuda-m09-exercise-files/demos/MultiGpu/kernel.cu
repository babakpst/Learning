#include "hip/hip_runtime.h"

#include <pthread.h>
#include <iostream>
using namespace std;

typedef void *(*func)(void*);

pthread_t startThread(func f, void* data)
{
  pthread_t t;
  pthread_create(&t, NULL, f, data);
  return t;
}

void endThread(pthread_t t)
{
  pthread_join(t, NULL);
}

struct work
{
  int device;
  int offset;
  int count;
  int *a;
  int *b;
  int *c;
};

__global__ void add(const int* a, const int* b, int* c)
{
  int tid = threadIdx.x;
  c[tid] = a[tid] + b[tid];
}

void* doWork(void* spec)
{
  work* w = reinterpret_cast<work*>(spec);

  // set device if not primary
  if (w->device != 0)
  {
    hipSetDevice(w->device);
    hipSetDeviceFlags(hipDeviceMapHost);
  }

  // get device pointers for a..c
  int *da, *db, *dc;
  hipHostGetDevicePointer(&da, w->a, 0);
  hipHostGetDevicePointer(&db, w->b, 0);
  hipHostGetDevicePointer(&dc, w->c, 0);

  da += w->offset;
  db += w->offset;
  dc += w->offset;

  add<<<1,w->count/2>>>(da,db,dc);

  return NULL;
}

int main()
{
  int dc;
  hipGetDeviceCount(&dc);

  cout << "We have " << dc << " devices" << endl;

  for (int i = 0; i < dc; i++)
  {
    hipDeviceProp_t props;
    hipGetDeviceProperties(&props, i);

    cout << "Device " << i << ": " << props.name << endl;
    cout << "Compute capability " << props.major << "." << props.minor << endl;
    cout << "Can map host memory? " << props.canMapHostMemory << endl;
  }

  // allocate a, b and c as zero-copy
  int count = 128;
  int *a, *b, *c;
  hipSetDevice(0);
  hipSetDeviceFlags(hipDeviceMapHost);
  hipHostAlloc(&a, count * sizeof(int),
    hipHostMallocMapped |
    hipHostMallocPortable |
    hipHostMallocWriteCombined);
  hipHostAlloc(&b, count * sizeof(int),
    hipHostMallocMapped |
    hipHostMallocPortable |
    hipHostMallocWriteCombined);
  hipHostAlloc(&c, count * sizeof(int),
    hipHostMallocMapped |
    hipHostMallocPortable |
    hipHostMallocWriteCombined);

  // write some values to a and b
  for (int i = 0; i < count; i++)
  {
    a[i] = i;
    b[i] = count-i;
  }

  // specify what needs to be done and where
  work tasks[2];
  tasks[0].device = 0;
  tasks[0].offset = 0;
  tasks[1].device = 1;
  tasks[1].offset = count/2;
  tasks[0].count = tasks[1].count = count;
  tasks[0].a = tasks[1].a = a;
  tasks[0].b = tasks[1].b = b;
  tasks[0].c = tasks[1].c = c;

  // start work on this + other thread
  pthread_t t = startThread(doWork, &tasks[1]);
  doWork(&tasks[0]);
  endThread(t);

  // print the results
  for (int i = 0; i < count; i++)
  {
    cout << c[i] << "\t";
  }

  // free up memory
  hipHostFree(a);
  hipHostFree(b);
  hipHostFree(c);

  //getchar();

  return 0;
}
