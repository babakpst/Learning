#include "hip/hip_runtime.h"


#include <iostream>
#include <numeric>
using namespace std;

__global__ void sumSingleBlock(int* d)
{
  int tid = threadIdx.x;

  // number of participating threads halves on each iteration
  for (int tc = blockDim.x, stepSize = 1; tc > 0; tc >>= 1, stepSize <<= 1)
  {
    // thread must be allowed to write
    if (tid < tc)
    {
      int pa = tid * stepSize * 2;
      int pb = pa + stepSize;
      d[pa] += d[pb];
    }
  }
}

__global__ void sumSingleBlock2(int* d)
{
  extern __shared__ int dcopy[];
  int tid = threadIdx.x;
  dcopy[tid*2] = d[tid*2];
  dcopy[tid*2+1] = d[tid*2+1];

  // number of participating threads halves on each iteration
  for (int tc = blockDim.x, stepSize = 1; tc > 0; tc >>= 1, stepSize <<= 1)
  {
    // thread must be allowed to write
    if (tid < tc)
    {
      int pa = tid * stepSize * 2;
      int pb = pa + stepSize;
      dcopy[pa] += dcopy[pb];
    }
  }

  if (tid == 0)
  {
    d[0] = dcopy[0];
  }
}

int main()
{
  hipError_t status;

  const int count = 256;
  const int size = count * sizeof(int);
  int* h = new int[count];
  for (int i = 0; i < count; ++i)
    h[i] = i+1;

  int* d;
  status = hipMalloc(&d, size);

  status = hipMemcpy(d,h,size, hipMemcpyHostToDevice);
    
  sumSingleBlock2<<<1,count/2,size>>>(d);

  int result;
  status = hipMemcpy(&result,d,sizeof(int),hipMemcpyDeviceToHost);

  cout << "Sum is " << result << endl;

  getchar();

  hipFree(d);
  delete [] h;
  
  return 0;
}