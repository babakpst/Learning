#include "hip/hip_runtime.h"

#include "hiprand/hiprand.h"

#include <iostream>
#include <iomanip>
using namespace std;

__device__ int dCount = 0;

__global__ void countPoints(const float* xs, const float* ys)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	float x = xs[idx] - 0.5f;
	float y = ys[idx] - 0.5f;
	int n = sqrtf(x*x + y*y) > 0.5f ? 0 : 1;
	atomicAdd(&dCount, n);
}

int main(int argc, char* argv[])
{ 
	const int count = 512*512;
	const int size = count * sizeof(float);
	hipError_t cudaStatus;
	hiprandStatus_t hiprandStatus;
	hiprandGenerator_t gen;
	
	hiprandStatus = hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_MTGP32);
	hiprandSetPseudoRandomGeneratorSeed(gen, time(0));

	float *x;
	float *y;
	cudaStatus = hipMalloc((void**)&x, size);
	cudaStatus = hipMalloc((void**)&y, size);

	hiprandStatus = hiprandGenerateUniform(gen, x, count);
	hiprandStatus = hiprandGenerateUniform(gen, y, count);

	countPoints<<<512,512>>>(x, y);

	hipDeviceSynchronize();
  
	int hCount;
	hipMemcpyFromSymbol(&hCount, HIP_SYMBOL(dCount), sizeof(int));


	hipFree(x);
	hipFree(y);

	hipDeviceReset();

	cout << setprecision(12)
		  << "Pi is approximately " 
		  << (4.0f * (float)hCount / (float)count)
		  << endl;

  getchar();

	return 0;
}