#include "hip/hip_runtime.h"

// nvcc main.cu -o main
//#define DEBUG 0

#include <stdio.h>
#include <limits.h>

bool checkResults (float *gold, float *d_data, int dimx, int dimy, float rel_tol) {
  for (int iy = 0; iy < dimy; ++iy) {
    for (int ix = 0; ix < dimx; ++ix) {
      int idx = iy * dimx + ix;

      float gdata = gold[idx];
      float ddata = d_data[idx];

      if (isnan(gdata) || isnan(ddata)) {
        printf("Nan detected: gold %f, device %f\n", gdata, ddata);
        return false;
      }

      float rdiff;
      if (fabs(gdata) == 0.f)
        rdiff = fabs(ddata);
      else
        rdiff = fabs(gdata - ddata) / fabs(gdata);

      if (rdiff > rel_tol) {
        printf("Error solutions don't match at iy=%d, ix=%d.\n", iy, ix);
        printf("gold: %f, device: %f\n", gdata, ddata);
        printf("rdiff: %f\n", rdiff);
        return false;
      }
    }
  }
  return true;
}

void computeCpuResults(float *g_data, int dimx, int dimy, int niterations, int nreps) {
  for (int r = 0; r < nreps; r++) {
    printf("Rep: %d\n", r);
#pragma omp parallel for
    for (int iy = 0; iy < dimy; ++iy) {
      for (int ix = 0; ix < dimx; ++ix) {
        int idx = iy * dimx + ix; // row-major

        float value = g_data[idx];
        // if (DEBUG) printf("x: %d, y: %d, idx: %d, val: %f \n", ix, iy, idx, value);
        for (int i = 0; i < niterations; i++) {
          if (ix % 4 == 0) {
            value += sqrtf(logf(value) + 1.f);
          } else if (ix % 4 == 1) {
            value += sqrtf(cosf(value) + 1.f);
          } else if (ix % 4 == 2) {
            value += sqrtf(sinf(value) + 1.f);
          } else if (ix % 4 == 3) {
            value += sqrtf(tanf(value) + 1.f);
          }
        }
        g_data[idx] = value;
      }
    }
  }
}

// change this-version 2
__global__ void kernel_A(float *g_data, int dimx, int dimy, int niterations) {
  // for (int iy = blockIdx.y * blockDim.y + threadIdx.y; iy < dimy; iy += blockDim.y * gridDim.y) {
    // for (int ix = blockIdx.x * blockDim.x + threadIdx.x; ix < dimx; ix += blockDim.x * gridDim.x) {
      
      int ix = blockIdx.x * blockDim.x + threadIdx.x;
      int iy = blockIdx.y * blockDim.y + threadIdx.y;
      int idx = iy * dimx + ix;
      int sidx = threadIdx.y * blockDim.x + threadIdx.x;

      extern __shared__ float s_data[];
      
      s_data[sidx] = g_data[idx];
      __syncthreads();

      int col_idx = threadIdx.x * blockDim.y + threadIdx.y;
      float value = s_data[col_idx];
      // float value = s_data[sidx];
      // float value = g_data[idx];

      // printf("gpu- x: %d, y: %d, idx: %d, sidx: %d, col_idx: %d, val: %f \n", ix, iy, idx, sidx, col_idx, value);
      // printf("gpu- bIdx.x: %d, bIdx.y: %d, tIdx.x: %d, tIdx.y: %d, x: %d, y: %d, idx: %d, sidx: %d, col_idx: %d, ix mod 4: %d \n", blockIdx.x, blockIdx.y, threadIdx.x, threadIdx.y, ix, iy, idx, sidx, col_idx, ix%4 );
      
      for (int i = 0; i < niterations/4; i++) 
      {
        if (iy % 4 == 0) {
          value += __fsqrt_rn(__logf(value) + 1.f);
          value += __fsqrt_rn(__logf(value) + 1.f);
          value += __fsqrt_rn(__logf(value) + 1.f);
          value += __fsqrt_rn(__logf(value) + 1.f);
        } else if (iy % 4 == 1) {
          value += __fsqrt_rn(__cosf(value) + 1.f);
          value += __fsqrt_rn(__cosf(value) + 1.f);
          value += __fsqrt_rn(__cosf(value) + 1.f);
          value += __fsqrt_rn(__cosf(value) + 1.f);
        } else if (iy % 4 == 2) {
          value += __fsqrt_rn(__sinf(value) + 1.f);
          value += __fsqrt_rn(__sinf(value) + 1.f);
          value += __fsqrt_rn(__sinf(value) + 1.f);
          value += __fsqrt_rn(__sinf(value) + 1.f);
        } else if (iy % 4 == 3) {
          value += __fsqrt_rn(__tanf(value) + 1.f);
          value += __fsqrt_rn(__tanf(value) + 1.f);
          value += __fsqrt_rn(__tanf(value) + 1.f);
          value += __fsqrt_rn(__tanf(value) + 1.f);
        }
      }
      for (int i = 0; i < niterations%4; i++) 
      {  
        if (iy % 4 == 0) {
          value += __fsqrt_rn(__logf(value) + 1.f);
        } else if (iy % 4 == 1) {
          value += __fsqrt_rn(__cosf(value) + 1.f);
        } else if (iy % 4 == 2) {
          value += __fsqrt_rn(__sinf(value) + 1.f);
        } else if (iy % 4 == 3) {
          value += __fsqrt_rn(__tanf(value) + 1.f);
        }
      }

      s_data[col_idx] = value;
      __syncthreads();
      // s_data[sidx] = value;
      // __syncthreads();
      // g_data[idx] = value;
      g_data[idx] = s_data[sidx];
    // }
  // }
}


// // change this-version 1
// __global__ void kernel_A(float *g_data, int dimx, int dimy, int niterations) {
//   // for (int iy = blockIdx.y * blockDim.y + threadIdx.y; iy < dimy; iy += blockDim.y * gridDim.y) {
//     // for (int ix = blockIdx.x * blockDim.x + threadIdx.x; ix < dimx; ix += blockDim.x * gridDim.x) {
      
//       int ix = blockIdx.x * blockDim.x + threadIdx.x;
//       int iy = blockIdx.y * blockDim.y + threadIdx.y;
//       int idx = iy * dimx + ix;

//       float value = g_data[idx];
//       // printf("gpu- x: %d, y: %d, idx: %d, val: %f \n", ix, iy, idx, value);
//       for (int i = 0; i < niterations; i++) 
//       {

//         // value += (ix % 4 == 0 ? sqrtf(logf(value) + 1.f) :  (ix % 4 == 1 ? sqrtf(cosf(value) + 1.f) : (ix % 4 == 2 ? sqrtf(sinf(value) + 1.f) : (ix % 4 == 3 ? sqrtf(tanf(value) + 1.f) : 0))));

//         // value += (ix % 4 == 0 ? sqrtf(logf(value) + 1.f) : 0);
//         // value += (ix % 4 == 1 ? sqrtf(cosf(value) + 1.f) : 0);
//         // value += (ix % 4 == 2 ? sqrtf(sinf(value) + 1.f) : 0);
//         // value += (ix % 4 == 3 ? sqrtf(tanf(value) + 1.f) : 0);

//         if (ix % 4 == 0) {
//           value += sqrtf(logf(value) + 1.f);
//         } else if (ix % 4 == 1) {
//           value += sqrtf(cosf(value) + 1.f);
//         } else if (ix % 4 == 2) {
//           value += sqrtf(sinf(value) + 1.f);
//         } else if (ix % 4 == 3) {
//           value += sqrtf(tanf(value) + 1.f);
//         }
//       }
//       g_data[idx] = value;
//     // }
//   // }
// }


// change this
void launchKernel(float * d_data, int dimx, int dimy, int niterations) {
  // Only change the contents of this function and the kernel(s). You may
  // change the kernel's function signature as you see fit. 

  //query number of SMs
  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, 0);
  // int num_sms = prop.multiProcessorCount;
  

  // printf(" SM counts: %d \n", num_sms);
  // dim3 block(1, 32);
  // dim3 grid(1, num_sms);
  
  // dim3 block(128, 128);
  // dim3 grid(64, 64);

  // dim3 block(16, 16);
  // dim3 grid(512, 512);

  // dim3 block(1, 1024);
  // dim3 grid(8192, 8);

  // dim3 block(1024, 1);
  // dim3 grid(8, 8192);

  // dim3 block(2048, 1);
  // dim3 grid(4, 8192);

  dim3 block(32, 32);
  dim3 grid(256, 256);

  // dim3 block(64, 64);
  // dim3 grid(128, 128);
  
  // kernel_A<<<grid, block>>>(d_data, dimx, dimy, niterations);
  kernel_A<<<grid, block,  block.x * block.y * sizeof(float)>>>(d_data, dimx, dimy, niterations);
  hipDeviceSynchronize();
}


float timing_experiment(float *d_data, int dimx, int dimy, int niterations, int nreps) {
  float elapsed_time_ms = 0.0f;
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipEventRecord(start, 0);
  for (int i = 0; i < nreps; i++) {
    launchKernel(d_data, dimx, dimy, niterations);
  }
  hipEventRecord(stop, 0);
  hipDeviceSynchronize();
  hipEventElapsedTime(&elapsed_time_ms, start, stop);
  elapsed_time_ms /= nreps;

  hipEventDestroy(start);
  hipEventDestroy(stop);

  return elapsed_time_ms;
}

int main() {
  int dimx = 8 * 1024;
  int dimy = 8 * 1024;

  // int dimx = 1 * 1024;
  // int dimy = 1 * 1024;

  int nreps = 10;
  int niterations = 5;

  int nbytes = dimx * dimy * sizeof(float);

  float *d_data = 0, *h_data = 0, *h_gold = 0;

  hipMalloc((void **)&d_data, nbytes);

  if (0 == d_data) {
    printf("couldn't allocate GPU memory\n");
    return -1;
  }

  printf("allocated %.2f MB on GPU\n", nbytes / (1024.f * 1024.f));

  h_data = (float *)malloc(nbytes);
  h_gold = (float *)malloc(nbytes);

  if (0 == h_data || 0 == h_gold) {
    printf("couldn't allocate CPU memory\n");
    return -2;
  }

  printf("allocated %.2f MB on CPU\n", 2.0f * nbytes / (1024.f * 1024.f));

  for (int i = 0; i < dimx * dimy; i++) h_gold[i] = 1.0f + 0.01*(float)rand()/(float)RAND_MAX;
  hipMemcpy(d_data, h_gold, nbytes, hipMemcpyHostToDevice);

  
  timing_experiment(d_data, dimx, dimy, niterations, 1);
  printf("Verifying solution\n");

  hipMemcpy(h_data, d_data, nbytes, hipMemcpyDeviceToHost);

  float rel_tol = .001;
  printf("\n Computing CPU results\n");
  computeCpuResults(h_gold, dimx, dimy, niterations, 1);
  bool pass = checkResults(h_gold, h_data, dimx, dimy, rel_tol);

  if (pass) {
    printf("Results are correct\n");
  } else {
    printf("FAIL:  results are incorrect\n");
  }  

  float elapsed_time_ms = 0.0f;
 
  elapsed_time_ms = timing_experiment(d_data, dimx, dimy, niterations, nreps);
  printf("A:  %8.2f ms\n", elapsed_time_ms);

  printf("CUDA: %s\n", hipGetErrorString(hipGetLastError()));

  if (d_data) hipFree(d_data);
  if (h_data) free(h_data);

  hipDeviceReset();

  return 0;
}

