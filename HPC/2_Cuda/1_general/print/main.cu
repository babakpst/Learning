// #include "cuda_runtime.h"
// #include "device_launch_parameters.h"

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
	if (code != hipSuccess)
	{
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

__global__ void scan_efficient_1G(int * input, int* auxiliry_array, int input_size);
__global__ void scan_summation(int * input, int * auxiliry_array, int input_size);

__global__ void testprint(float *d_input, int size)
{
  int tid = threadIdx.x;
  int gid = blockDim.x*blockIdx.x +tid;
  printf(" gid: %d, tid: %d, value: %f\n",gid, tid, d_input[gid]);
}


int main()
{

  int size = 64;
  float *h_input=new float[size];
  for (int i = 0; i< size; ++i)
  {
    h_input[i] = i*1.1;
    printf(" -- %f", h_input[i]);
  }
  printf("\n");

  float *d_input;
  int byte_size = size*sizeof(float);
  gpuErrchk(hipMalloc(&d_input, byte_size));  
  gpuErrchk(hipMemcpy(d_input, h_input, byte_size,	hipMemcpyHostToDevice));


  dim3 block(32,1,1);
  dim3 grid(2,1,1);
  
  testprint<<<grid,block>>>(d_input, size);
  
  gpuErrchk(hipDeviceSynchronize());

  gpuErrchk(hipFree(d_input));
	free(h_input);
	gpuErrchk(hipDeviceReset());

  printf(" done \n");  
  return 0;
}