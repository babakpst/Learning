
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
// Kernel function to add the elements of two arrays
__global__
void add(int n, float *x, float *y)
{
  for (int i = 0; i < n; i++)
    y[i] = x[i] + y[i];
}

int main(void)
{

  std::cout << " code begins \n";

  int N = 1<<20;
  float *x, *y;

  std::cout << " size of the vecotr: " << N << "\n";

  // Allocate Unified Memory – accessible from CPU or GPU
  hipMallocManaged(&x, N*sizeof(float));
  hipMallocManaged(&y, N*sizeof(float));

  std::cout << " memory allocated. \n";

  // initialize x and y arrays on the host
  for (int i = 0; i < N; i++) {
    std::cout << " Counter: " << i << "\n";
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

  std::cout << " vector filled. \n";

  // Run kernel on 1M elements on the GPU
  add<<<1, 1>>>(N, x, y);
  
  std::cout << " done with the summation. \n";

  // Wait for GPU to finish before accessing on host
  hipDeviceSynchronize();

  std::cout << " synchronization \n";

  // Check for errors (all values should be 3.0f)
  float maxError = 0.0f;
  for (int i = 0; i < N; i++)
    maxError = fmax(maxError, fabs(y[i]-3.0f));
  std::cout << "Max error: " << maxError << std::endl;

  std::cout << " error calculated \n";

  // Free memory
  hipFree(x);
  hipFree(y);
 
  std::cout << " End successfully. \n";
 
  return 0;
}
