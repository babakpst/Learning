
// Babak Poursartip
// 09/14/2020

// Udemy Cuda
// unique index calculation

#include <hip/hip_runtime.h>
#include <cstdio>


// ===========================================
// 2d grid, 2d block
__global__ void unique_gid_calculation_3d_3d(int *input) {


  int threadsPerBlock       = blockDim.x * blockDim.y * blockDim.z;
  int threadPositionInBlock = threadIdx.x +
                              blockDim.x * threadIdx.y +
                              blockDim.x * blockDim.y * threadIdx.z;

  int blockPositionInGrid   = blockIdx.x +
                              gridDim.x * blockIdx.y +
                              gridDim.x * gridDim.y * blockIdx.z;

  int tid = blockPositionInGrid * threadsPerBlock + threadPositionInBlock;

  printf("gridDim.x: %d, gridDim.y: %d, gridDim.z: %d, blockDim.x: %d, blockDim.y: %d, blockDim.z: %d, blockIdx.x: %d, blockIdx.y: %d, blockIdx.z: %d, threadIdx.x: %d, threadIdx.y: %d, threadIdx.z: %d, tid: %d \n",
          gridDim.x, gridDim.y, gridDim.z, 
          blockDim.x, blockDim.y, blockDim.z, 
          blockIdx.x, blockIdx.y, blockIdx.z, 
          threadIdx.x, threadIdx.y, threadIdx.z, 
          tid);


}

// ===========================================
// 2d grid, 2d block
__global__ void unique_gid_calculation_2d_2d(int *input) {

  int tid = blockDim.x * threadIdx.y + threadIdx.x;

  int num_threads_in_a_block = blockDim.x * blockDim.y;
  int block_offset = blockIdx.x * num_threads_in_a_block;

  int num_threads_in_a_row = num_threads_in_a_block * gridDim.x;
  int row_offset = num_threads_in_a_row * blockIdx.y;

  int gid = tid + block_offset + row_offset;

  printf("gridDim.x: %d,  gridDim.y: %d,  gridDim.z: %d, blockDim.x: %d, blockDim.y: %d, blockDim.z: %d, blockIdx.x: %d, blockIdx.y: %d, blockIdx.z: %d, threadIdx.x: %d, threadIdx.y: %d, threadIdx.z: %d, gid: %d \n",
          gridDim.x, gridDim.y, gridDim.z, 
          blockDim.x, blockDim.y, blockDim.z, 
          blockIdx.x, blockIdx.y, blockIdx.z, 
          threadIdx.x, threadIdx.y, threadIdx.z, 
          gid);
}

// ===========================================
int main() {
  printf(" starts ...");

  int nCol = 6;
  int nRow = 8;
  int array_size = nCol*nRow;
  int array_byte_size = sizeof(int) * array_size;
  int h_data[array_size]; 
  
  for (int i = 0; i< array_size; ++i)
  {
    h_data[i] = i;
  }
  
  

  printf(" data on the host: \n");
  for (int i = 0; i < array_size; ++i)
  {
    printf(" %d", h_data[i]);
    if((i%8)==0) printf("\n");
  }  
  printf("\n\n");

  int *d_data; // array on the device

  hipMalloc((void **)&d_data, array_byte_size);
  hipMemcpy(d_data, h_data, array_byte_size, hipMemcpyHostToDevice);

  dim3 grid(nCol, 1, 1);
  dim3 block(nRow, 1, 1);


  printf(" data on the device: \n");
  //unique_gid_calculation_2d_2d<<<grid, block>>>(d_data);
  unique_gid_calculation_3d_3d<<<grid, block>>>(d_data);

  hipDeviceSynchronize();

  hipDeviceReset();
  printf(" finished.");
  return 0;
}
