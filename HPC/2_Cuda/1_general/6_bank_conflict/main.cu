
#include <hip/hip_runtime.h>
#include <time.h> 
#include <cstdio>

struct padding
{
  // float x,y,z,w;
  float x[128];
  float padding; // this avoids bank conclict
};

__global__ void test_bank_conflict()
{
  int idx = threadIdx.y*blockDim.x + threadIdx.x;
  
  extern __shared__ padding smem[];
  smem[idx].x[0]++; 

  // unsigned long long start, end;
  // extern __shared__ int smem[];

  // start = clock();
  // no bank conflict
  // __syncthreads();
  // end = clock();
  // if (idx==0) printf(" elapsed time-no bank conflict: %llu \n", end-start); // 150
  // printf(" elapsed time-no bank conflict: %llu \n", end-start); // 150
  
  // __syncthreads();
  // printf("\n");
  // start = clock();
  // // no bank conflict
  // smem[idx] = 4; 
  // __syncthreads();
  // end = clock();
  // // if (idx==0) printf(" elapsed time-no bank conflict: %llu \n", end-start);  // 184
  // printf(" elapsed time-no bank conflict: %llu \n", end-start);  // 184

  // __syncthreads();
  // printf("\n");
  // start = clock();
  // // 2-way bank conflict
  // smem[idx*2] = 4;
  // __syncthreads();
  // end = clock();
  // // if (idx==0) printf(" elapsed time- 2-way: %llu \n", end-start); // 94
  // printf(" elapsed time- 2-way: %llu \n", end-start); // 94

  // __syncthreads();
  // printf("\n");
  // start = clock();
  // // 8-way bank conflict
  // smem[idx*8] = 4;
  // __syncthreads();
  // end = clock();
  // // if (idx==0) printf(" elapsed time- 8-way: %llu \n", end-start);  // 93
  // printf(" elapsed time- 8-way: %llu \n", end-start);  // 93

  // __syncthreads();
  // printf("\n");
  // start = clock();
  // // 32-way bank conflict
  // smem[idx*32] = 4;
  // __syncthreads();
  // end = clock();
  // // if (idx==0) printf(" elapsed time- 32-way: %llu \n", end-start); // 244
  // printff(" elapsed time- 32-way: %llu \n", end-start); // 244


}

int main()
{
  printf("code starts\n"); 
  int iters = 20000;
  // float  elapsedTime;
  // cudaEvent_t start, end; 
  // cudaEventCreate(&start); 
  // cudaEventCreate(&end); 

  // cudaEventRecord(start,0); 

  clock_t start = clock(); 
  for (int i = 0; i<iters; ++i)
    test_bank_conflict<<<1,1024,32*1024*sizeof(int)>>>();
  hipDeviceSynchronize();
  clock_t end = clock(); 
  printf(" elapsed time: %4.6f (ms)\n",(double)((double)(end - start)*1000.0f / CLOCKS_PER_SEC)); 
  
  // cudaEventRecord(end,0); 
  // cudaEventSynchronize(end); 
  // cudaEventElapsedTime(&elapsedTime, start, end);  // in millisecond 
  // // printf("Kernel execution time using events : %f (ms)\n",elapsedTime/iters);
  // printf("Kernel execution time using events : %f (ms)\n",elapsedTime);
  // cudaEventDestroy(start); 
  // cudaEventDestroy(end); 
  

  printf("end of the code\n"); 
  return 0;
}