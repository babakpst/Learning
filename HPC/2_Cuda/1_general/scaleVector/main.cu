
// scale a vector on CPU and GPU
// Babak
// 12/01/2023


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>

__global__ void scale(float* d_in, float *d_out, int size, int scaler)
{
  int tid = blockIdx.x*blockDim.x + threadIdx.x;
  if (tid < size)
    d_out[tid] = scaler * d_in[tid];
}

__global__ void scale2(float* d_in, float *d_out, int size, int scaler)
{
  extern __shared__  float arr[];

  int tid = blockIdx.x*blockDim.x + threadIdx.x;
  if (tid < size)
  {
    arr[tid] = scaler * d_in[tid];
    d_out[tid] = arr[tid];
  }
}

int main(int argc, char* argv[])
{

  printf(" scale a vector \n");

  int size = 1<<12;
  int sizeByte = size * sizeof(float);
  int scaler = 3.0f;

  // host memory allocation
  float *h_in, *h_out, *h_ref;
  
  h_in  = (float*)malloc(sizeByte);
  h_out = (float*)malloc(sizeByte);
  h_ref = (float*)malloc(sizeByte);
  
  // device memory allocation 
  float *d_in, *d_out;
  hipMalloc((void **)&d_in, sizeByte);
  hipMalloc((void **)&d_out, sizeByte);
  
  // initilize input vector
  for (int i = 0; i< size; ++i)
    h_in[i] = 1.1f*i;

  // scale on CPU
  for (int i = 0; i< size; ++i)
  { 
    h_ref[i] = h_in[i]*scaler;
  }
    
  dim3 block(128);
  dim3 grid(size/128+(size%128!=0));
  printf("block: %d, grid: %d \n", block.x, grid.x);

  hipEvent_t cstart, cHtoD, cfunc, cDtoH;
  hipEventCreate(&cstart);
  hipEventCreate(&cHtoD);
  hipEventCreate(&cDtoH);
  hipEventCreate(&cfunc);
  
  // double  accum=0.0;
  // for (int i = 0; i<20;++i)
  // {
    // clock_t start = clock();
    hipEventRecord(cstart,0);
    hipMemcpy(d_in, h_in, sizeByte, hipMemcpyHostToDevice);
    // clock_t HtoD=clock();
    hipEventRecord(cHtoD,0);
    hipEventSynchronize(cHtoD);
    // accum+= (double)( HtoD-start);
    // }
    
    // printf(" HtoD average: %4.6f (ms)\n",(double)( accum*1000.0f / CLOCKS_PER_SEC/20));
    
    // scale<<<grid, block>>>(d_in, d_out, size, 3);
    scale2<<<grid, block, sizeByte>>>(d_in, d_out, size, 3);
    
    hipDeviceSynchronize();
    hipEventRecord(cfunc,0);
    hipEventSynchronize(cfunc);
    // clock_t func=clock();
    
    hipMemcpy(h_out, d_out, sizeByte, hipMemcpyDeviceToHost);
    // clock_t DtoH=clock();
    hipEventRecord(cDtoH,0);
    hipEventSynchronize(cDtoH);

    float time1, time2, time3;
    hipEventElapsedTime(&time1, cstart, cHtoD);
    hipEventElapsedTime(&time1, cHtoD, cfunc);
    hipEventElapsedTime(&time1, cfunc, cDtoH);
  
  // printf(" HtoD: %4.6f (ms)\n",(double)((double)(HtoD - start)*1000.0f / CLOCKS_PER_SEC));
  // printf(" func: %4.6f (ms)\n",(double)((double)(func- HtoD)*1000.0f / CLOCKS_PER_SEC));
  // printf(" DtoH: %4.6f (ms)\n",(double)((double)(DtoH-func)*1000.0f / CLOCKS_PER_SEC));

  printf(" HtoD: %4.6f (ms)\n",time1);
  printf(" func: %4.6f (ms)\n",time2);
  printf(" DtoH: %4.6f (ms)\n",time3);


  // check the output
  for (int i = 0; i< size; ++i)
    // if (abs(h_out[i] - h_ref[i] )>0.001) printf(" error at %d: %f %f\n", i, h_in[i], h_ref[i]);
    if ((h_out[i] != h_ref[i] )) printf(" error at %d: %f %f\n", i, h_in[i], h_ref[i]);

  printf(" end successfully \n");

  hipFree(d_in);
  hipFree(d_out);
  free(h_in);
  free(h_out);
  free(h_ref);

  hipDeviceReset();
  return 0;
}