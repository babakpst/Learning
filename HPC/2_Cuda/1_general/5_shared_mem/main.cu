

#include <hip/hip_runtime.h>
#include <cstdio>


__global__ void AKernel(double *d_darr, int *d_iarr, int intSize, int doubleSize) 
{ 
  extern __shared__ int TwoArrays[]; 
  int *iarr = &TwoArrays[0];
  double *darr = (double*) &TwoArrays[intSize];

  int idx = threadIdx.x;
  iarr[idx] = d_iarr[idx] * 2;
  darr[idx] = d_darr[idx] * 2;

  d_iarr[idx] = iarr[idx];
  d_darr[idx] = darr[idx];
} 


int main()
{
  int intSize = 64;
  int doubleSize = 64;
  int size = intSize*sizeof(int) + doubleSize*sizeof(double);

  int *h_iarr = new int[intSize];
  double *h_darr = new double[doubleSize];

  for (int i = 0; i<intSize; ++i)
    h_iarr[i] = i;

  for (int i = 0; i<doubleSize; ++i)
    h_darr[i] = i*0.1;

  printf(" new values \n");
  printf(" integer: \n");
  for (int i = 0; i<intSize; ++i)
    printf( "%d ", h_iarr[i]);
  printf("\n");
  
  printf(" double: \n");
  for (int i = 0; i<doubleSize; ++i)
    printf( "%f ", h_darr[i]);
  printf("\n");

  int *d_iarr;
  hipMalloc((void**)&d_iarr, intSize*sizeof(int));
  
  double *d_darr;
  hipMalloc((void**)&d_darr, doubleSize*sizeof(double));

  hipMemcpy(d_iarr, h_iarr, intSize*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_darr, h_darr, doubleSize*sizeof(double), hipMemcpyHostToDevice);

  dim3 blocks(64,1,1);
  dim3 grids(1,1,1);

  AKernel<<<grids,blocks,size>>>(d_darr, d_iarr, intSize, doubleSize);
  hipDeviceSynchronize();

  hipMemcpy(h_iarr, d_iarr, intSize*sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(h_darr, d_darr, doubleSize*sizeof(double), hipMemcpyDeviceToHost);

  hipDeviceReset();
  
  printf(" new values \n");
  printf(" integer: \n");
  for (int i = 0; i<intSize; ++i)
    printf( "%d ", h_iarr[i]);
  printf("\n");
  
  printf(" double: \n");
  for (int i = 0; i<doubleSize; ++i)
    printf( "%f ", h_darr[i]);
  printf("\n");

  printf(" end of the code\n");
  return 0; 
}