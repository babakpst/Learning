#include "hip/hip_runtime.h"
#include <cstdint>
#include <iostream>
#include "helpers.cuh"
#include "encryption.cuh"

void encrypt_cpu(uint64_t * data, uint64_t num_entries, 
                 uint64_t num_iters, bool parallel=true) {

    #pragma omp parallel for if (parallel)
    for (uint64_t entry = 0; entry < num_entries; entry++)
        data[entry] = permute64(entry, num_iters);
}

__global__ 
void decrypt_gpu(uint64_t * data, uint64_t num_entries, 
                 uint64_t num_iters) {

    const uint64_t thrdID = blockIdx.x*blockDim.x+threadIdx.x;
    const uint64_t stride = blockDim.x*gridDim.x;

    for (uint64_t entry = thrdID; entry < num_entries; entry += stride)
        data[entry] = unpermute64(data[entry], num_iters);
}

bool check_result_cpu(uint64_t * data, uint64_t num_entries,
                      bool parallel=true) {

    uint64_t counter = 0;

    #pragma omp parallel for reduction(+: counter) if (parallel)
    for (uint64_t entry = 0; entry < num_entries; entry++)
        counter += data[entry] == entry;

    return counter == num_entries;
}


//=======================================
int main (int argc, char * argv[]) {


    std::cout << "starts ...\n";
    const char * encrypted_file = "fi";

    Timer timer;

    const uint64_t num_entries = 1UL << 26;
    const uint64_t num_iters = 1UL << 10;
    const bool openmp = false;


    std::cout << "allocation ...\n";
    uint64_t * data_cpu, * data_gpu;
    hipHostMalloc(&data_cpu, sizeof(uint64_t)*num_entries);
    hipMalloc    (&data_gpu, sizeof(uint64_t)*num_entries);
    check_last_error();

    std::cout << "encryption ...\n";
    if (!encrypted_file_exists(encrypted_file)) {
        encrypt_cpu(data_cpu, num_entries, num_iters, openmp);
        write_encrypted_to_file(encrypted_file, data_cpu, sizeof(uint64_t)*num_entries);
    } else {
        read_encrypted_from_file(encrypted_file, data_cpu, sizeof(uint64_t)*num_entries);
    }


    const uint64_t num_streams = 12; 

    hipStream_t streams[num_streams]; 

    for (uint64_t stream = 0; stream < num_streams; stream++) 
      hipStreamCreate(&streams[stream]); 

    const uint64_t chunk_size = sdiv(num_entries, num_streams); 

    std::cout << "calculations ...\n";

    timer.start();
    for (uint64_t stream = 0; stream < num_streams; stream++)
    { 
    
      const uint64_t lower = chunk_size*stream; 
      const uint64_t upper = min(lower+chunk_size, num_entries); 
      const uint64_t width = upper-lower; 
       
      hipMemcpyAsync(data_gpu+lower, data_cpu+lower, sizeof(uint64_t)*width, hipMemcpyHostToDevice, streams[stream]); 
       
      decrypt_gpu<<<80*32, 64, 0, streams[stream]>>>(data_gpu+lower, width, num_iters); 
       
      hipMemcpyAsync(data_cpu+lower, data_gpu+lower, sizeof(uint64_t)*width, hipMemcpyDeviceToHost, streams[stream]); 
    } 
 
    // Destroy streams. 
    for (uint64_t stream = 0; stream < num_streams; stream++) 
      hipStreamDestroy(streams[stream]); 


    timer.stop("total time on GPU");
    check_last_error();

    const bool success = check_result_cpu(data_cpu, num_entries, openmp);
    std::cout << "STATUS: test " 
              << ( success ? "passed" : "failed")
              << std::endl;

    hipHostFree(data_cpu);
    hipFree    (data_gpu);
    check_last_error();
}






























