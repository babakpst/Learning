#include "hip/hip_runtime.h"

#include <cstdint>
#include <iostream>
#include "helpers.cuh"
#include "encryption.cuh"

// Host function.
void encrypt_cpu(uint64_t * data, uint64_t num_entries, 
                 uint64_t num_iters, bool parallel=true) {

    // Use OpenMP to use all available CPU cores.
    #pragma omp parallel for if (parallel)
    for (uint64_t entry = 0; entry < num_entries; entry++)
        // Permute each data entry the number of iterations and then write result to data.
        data[entry] = permute64(entry, num_iters);
}

// Device function.
__global__ 
void decrypt_gpu(uint64_t * data, uint64_t num_entries, 
                 uint64_t num_iters) {

    const uint64_t thrdID = blockIdx.x*blockDim.x+threadIdx.x;
    const uint64_t stride = blockDim.x*gridDim.x;

    //printf(" checkpoint i0\n");

    // Utilize grid-stride loop for arbitrary data sizes.
    for (uint64_t entry = thrdID; entry < num_entries; entry += stride)
        // Unpermute each data entry the number of iterations then write result to data.
        data[entry] = unpermute64(data[entry], num_iters);
    //printf(" checkpoint i1\n");

}

// Host function.
bool check_result_cpu(uint64_t * data, uint64_t num_entries,
                      bool parallel=true) {

    uint64_t counter = 0;

    #pragma omp parallel for reduction(+: counter) if (parallel)
    for (uint64_t entry = 0; entry < num_entries; entry++)
    
        // Because we created initial data values by ranging from 0 to N-1,
        // and because encrypting and decrypting is symmetrical,
        // then each data entry should be equal to `entry`.
        counter += data[entry] == entry;

    // True if all values have been correctly decrypted.
    return counter == num_entries;
}

int main (int argc, char * argv[]) {
    // This file will be used to cache encryption results
    // so we don't have to wait on the CPU every time.
    //const char * encrypted_file = "/dli/task/encrypted";
    const char * encrypted_file = "hello";
    //"/home/babak/Codes/Learning/HPC/2_Cuda/6_Accelerating_CUDA_C++_Applications_with_Concurrent_Streams/3_Application/hello";

    // Timer instance to be used for sections of the application.
    Timer timer;
    
    // Timer instance to be used for total time on the GPU(s).
    Timer overall;

    const uint64_t num_entries = 1UL << 26;
    const uint64_t num_iters = 1UL << 10;
    
    // Use all available CPUs in parallel for host calculations.
    //const bool openmp = true;
    const bool openmp = false;

    // This timer start and then stop pattern will be used throughout the application.
    timer.start();
    uint64_t * data_cpu, * data_gpu;
    // hipHostMalloc will be discussed at length later in the course.
    hipHostMalloc(&data_cpu, sizeof(uint64_t)*num_entries);
    hipMalloc    (&data_gpu, sizeof(uint64_t)*num_entries);
    timer.stop("allocate memory");
    check_last_error();

    timer.start();
    // If encryption cache file does not exist...
    if (!encrypted_file_exists(encrypted_file)) {
        // ...encrypt data in parallel on CPU...
        std::cout << " encrypting... \n ";
        encrypt_cpu(data_cpu, num_entries, num_iters, openmp);
        // ...and make encryption cache file for later.
        write_encrypted_to_file(encrypted_file, data_cpu, sizeof(uint64_t)*num_entries);
    } else {
        std::cout << " reading... \n ";
        // Use encryption cache file if it exists.
        read_encrypted_from_file(encrypted_file, data_cpu, sizeof(uint64_t)*num_entries);
    }
    timer.stop("encrypt data on CPU");

    // Begin timing for total time on GPU(s).
    overall.start();
    timer.start();
    // Data copy from CPU to GPU.
    hipMemcpy(data_gpu, data_cpu, 
               sizeof(uint64_t)*num_entries, hipMemcpyHostToDevice);
    timer.stop("copy data from CPU to GPU");
    check_last_error();

    // non-default stream
    hipStream_t str;
    hipStreamCreate(&str);

    timer.start();
    // Decrypt data on GPU(s).
    decrypt_gpu<<<80*32, 64, 0, str>>>(data_gpu, num_entries, num_iters);
    timer.stop("decrypt data on GPU");
    //std::cout << " checkpoint 0\n";
    check_last_error();
    //std::cout << " checkpoint 1\n";    

    hipStreamDestroy(str);

    
    timer.start();
    // Copy data from GPU to CPU.
    hipMemcpy(data_cpu, data_gpu, 
               sizeof(uint64_t)*num_entries, hipMemcpyDeviceToHost);
    timer.stop("copy data from GPU to CPU");
    // Stop timer for total time on GPU(s).
    overall.stop("total time on GPU");
    check_last_error();

    timer.start();
    // Check results on CPU.
    const bool success = check_result_cpu(data_cpu, num_entries, openmp);
    std::cout << "STATUS: test " 
              << ( success ? "passed" : "failed")
              << std::endl;
    timer.stop("checking result on CPU");

    timer.start();
    // Free memory.
    hipHostFree(data_cpu);
    hipFree    (data_gpu);
    timer.stop("free memory");
    check_last_error();
}

