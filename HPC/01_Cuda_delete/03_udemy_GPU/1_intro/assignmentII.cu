#include "hip/hip_runtime.h"

// Babak Poursartip
// 09/16/2020

// udemy CUDA
// Error: hipError_t
// We cannot use hipError_t to check the launch part of the function.

#include "common.h"
#include <cstdio>
#include <time.h>

// =================================
// cuda error check macro
#define gpuErrchk(ans)                                                         \
  { gpuAssert(ans, __FILE__, __LINE__); }

inline void gpuAssert(hipError_t code, const char *file, int line,
                      bool abort = true) {
  if (code != hipSuccess) {
    fprintf(stderr, "GPUassert: %s. File: %s, line: %d. \n",
            hipGetErrorString(code), file, line);
    if (abort) {
      // printf(" Exists from the gpuErrorCheck func.\n");
      exit(code);
    }
  }
}

// =================================
__global__ void sum_array_gpu(int *a, int *b, int *c, int *out,
                              const int size) {
  // 1d grid, 1d block thread
  int gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid < size) {
    out[gid] = a[gid] + b[gid] + c[gid];
  }
}

// =================================
void sum_array_cpu(int *a, int *b, int *c, int *out, const int size) {
  for (int i = 0; i < size; ++i) {
    out[i] = a[i] + b[i] + c[i];
  }
}

// =================================
int main() {
  printf(" starts ... \n");

  int size = 1 << 20;
  const int NO_BYTES = size * sizeof(int);
  //  int block_size = 64;
  int block_size;
  printf(" Enter block size: ");
  scanf("%d", &block_size);

  printf(" size: %d, block_size: %d \n", size, block_size);

  int *h_a, *h_b, *h_c; // input arrays
  int *h_gpu_output, *h_cpu_output;

  // allocate arrays on the host
  h_a = (int *)malloc(NO_BYTES);
  h_b = (int *)malloc(NO_BYTES);
  h_c = (int *)malloc(NO_BYTES);

  h_gpu_output = (int *)malloc(NO_BYTES);
  h_cpu_output = (int *)malloc(NO_BYTES);

  // initialize arrays on the host
  time_t t;
  srand((unsigned)time(&t));

  for (int i = 0; i < size; ++i) {
    h_a[i] = (int)(rand() & 0xFF);
    h_b[i] = (int)(rand() & 0xFF);
    h_c[i] = (int)(rand() & 0xFF);
  }

  // for (int i = 0; i < size; ++i) {
  // h_b[i] = (int)(rand() & 0xFF);
  // }

  memset(h_gpu_output, 0, NO_BYTES);
  memset(h_cpu_output, 0, NO_BYTES);

  // cpu vector summation
  clock_t cpu_start, cpu_end;
  cpu_start = clock();
  sum_array_cpu(h_a, h_b, h_c, h_cpu_output, size);
  cpu_end = clock();

  // device arrays;
  int *d_a, *d_b, *d_c, *d_output;

  gpuErrchk(hipMalloc((void **)&d_a, NO_BYTES));
  gpuErrchk(hipMalloc((void **)&d_b, NO_BYTES));
  gpuErrchk(hipMalloc((void **)&d_c, NO_BYTES));
  gpuErrchk(hipMalloc((void **)&d_output, NO_BYTES));

  clock_t htod_start, htod_end;
  htod_start = clock();
  gpuErrchk(hipMemcpy(d_a, h_a, NO_BYTES, hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(d_b, h_b, NO_BYTES, hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(d_c, h_c, NO_BYTES, hipMemcpyHostToDevice));
  htod_end = clock();

  dim3 block(block_size);
  // +1 to guarantee that we have more threads than array size
  dim3 grid(size / block.x + 1);

  clock_t gpu_start, gpu_end;
  gpu_start = clock();
  sum_array_gpu<<<grid, block>>>(d_a, d_b, d_c, d_output, size);
  gpuErrchk(hipDeviceSynchronize());
  gpu_end = clock();

  clock_t dtoh_start, dtoh_end;
  dtoh_start = clock();
  gpuErrchk(
      hipMemcpy(h_gpu_output, d_output, NO_BYTES, hipMemcpyDeviceToHost));
  dtoh_end = clock();

  // comparison the results of cpu and gpu
  compare_arrays(h_gpu_output, h_cpu_output, size);

  // printing execution time

  printf(" htod mem transfer time: %4.6f \n",
         (double)((double)(htod_end - htod_start) / CLOCKS_PER_SEC));

  printf(" dtoh mem transfer time: %4.6f \n",
         (double)((double)(dtoh_end - dtoh_start) / CLOCKS_PER_SEC));

  printf(" Sum array GPU execution time: %4.6f \n",
         (double)((double)(gpu_end - gpu_start) / CLOCKS_PER_SEC));

  printf(" Sum array GPU total execution time: %4.6f \n",
         (double)((double)(dtoh_end - htod_start) / CLOCKS_PER_SEC));

  printf(" Sum array CPU execution time: %4.6f \n",
         (double)((double)(cpu_end - cpu_start) / CLOCKS_PER_SEC));

  gpuErrchk(hipFree(d_a));
  gpuErrchk(hipFree(d_b));
  gpuErrchk(hipFree(d_c));
  gpuErrchk(hipFree(d_output));

  free(h_a);
  free(h_b);
  free(h_c);
  free(h_gpu_output);
  free(h_cpu_output);

  hipDeviceReset();
  printf(" finished. \n");
  return 0;
}
