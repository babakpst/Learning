#include "hip/hip_runtime.h"

// Babak Poursartip
// 09/15/2020

// udemy CUDA
// sum array

#include "common.h"
#include <cstdio>
#include <time.h>

// =================================
__global__ void sum_array_gpu(int *a, int *b, int *c, const int size) {
  // 1d grid, 1d block thread
  int gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid < size) {
    c[gid] = a[gid] + b[gid];
  }
}

// =================================
void sum_array_cpu(int *a, int *b, int *c, const int size) {
  for (int i = 0; i < size; ++i) {
    c[i] = a[i] + b[i];
  }
}

// =================================
int main() {
  printf(" starts ... \n");

  int size = 10000;
  int block_size = 128;
  const int NO_BYTES = size * sizeof(int);

  int *h_a, *h_b, *h_output, *h_cpu_out;

  // allocate arrays on the host
  h_a = (int *)malloc(NO_BYTES);
  h_b = (int *)malloc(NO_BYTES);
  h_output = (int *)malloc(NO_BYTES);
  h_cpu_out = (int *)malloc(NO_BYTES);

  // initialize arrays on the host
  time_t t;
  srand((unsigned)time(&t));

  for (int i = 0; i < size; ++i) {
    h_a[i] = (int)(rand() & 0xFF);
  }

  for (int i = 0; i < size; ++i) {
    h_b[i] = (int)(rand() & 0xFF);
  }

  memset(h_output, 0, NO_BYTES);
  memset(h_cpu_out, 0, NO_BYTES);

  sum_array_cpu(h_a, h_b, h_cpu_out, size);

  // device arrays;
  int *d_a, *d_b, *d_output;

  hipMalloc((void **)&d_a, NO_BYTES);
  hipMalloc((void **)&d_b, NO_BYTES);
  hipMalloc((void **)&d_output, NO_BYTES);

  hipMemcpy(d_a, h_a, NO_BYTES, hipMemcpyHostToDevice);
  hipMemcpy(d_b, h_b, NO_BYTES, hipMemcpyHostToDevice);

  dim3 block(block_size);
  // +1 to guarantee that we have more threads than array size
  dim3 grid(size / block.x + 1);

  sum_array_gpu<<<grid, block>>>(d_a, d_b, d_output, size);
  hipDeviceSynchronize();

  hipMemcpy(h_output, d_output, NO_BYTES, hipMemcpyDeviceToHost);

  // comparison the results of cpu and gpu
  compare_arrays(h_output, h_cpu_out, size);

  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_output);

  free(h_a);
  free(h_b);
  free(h_output);

  hipDeviceReset();
  printf(" finished. \n");
  return 0;
}
