
// Babak Poursartip
// 09/14/2020

// Udemy Cuda
// dimensions, blockIdx, threadIdx, blockDim, gridDim
// blockDim.x,~.y,~.z: no. of threads in each direction of the block
// gridDim.x,~.y,~.z: no. of block in each direction of the grid

/*

    <<============== gridDim.x =  3  ========>>

                                   <blocDim.x=4>
/\   |--|--|--|--|  |--|--|--|--|  |--|--|--|--|      /\
||   |  |  |  |  |  |  |  |  |  |  |  |  |  |  |      ||
||   |--|--|--|--|  |--|--|--|--|  |--|--|--|--|  blockDim.y=2
||   |  |  |  |  |  |  |  |  |  |  |  |  |  |  |      ||
||   |--|--|--|--|  |--|--|--|--|  |--|--|--|--|      \/

gridDim.y = 2

||   |--|--|--|--|  |--|--|--|--|  |--|--|--|--|
||   |  |  |  |  |  |  |  |  |  |  |  |  |  |  |
||   |--|--|--|--|  |--|--|--|--|  |--|--|--|--|
||   |  |  |  |  |  |  |  |  |  |  |  |  |  |  |
\/   |--|--|--|--|  |--|--|--|--|  |--|--|--|--|

*/


#include <hip/hip_runtime.h>
#include <cstdio>
//#include <iostream>

#include <cuda_device_runtime_api.h>
#include <hip/hip_runtime_api.h>

__global__ void print_threadIds() {
  printf("blockDim.x=%d, blockDim.y=%d <> "
         "gridDim.x=%d, girdDim.y=%d <> "
         "blockIdx.x=%d, blockIdx.y=%d, blockIdx.z=%d <>"
         "threadIdx.x=%d, threadIdx.y=%d, threadIdx.z=%d"
         "\n",
         blockDim.x, blockDim.y, gridDim.x, gridDim.y, blockIdx.x, blockIdx.y,
         blockIdx.z, threadIdx.x, threadIdx.y, threadIdx.z);
}

// ================================================
int main() {
  printf(" starts ... ");

  int nx, ny, nz;
  nx = 4;
  ny = 4;
  nz = 4;

  dim3 block(2, 2, 2);
  dim3 grid(nx / block.x, ny / block.y, nz / block.z);

  print_threadIds<<<grid, block>>>();
  hipDeviceSynchronize();

  hipDeviceReset();
  return 0;
}