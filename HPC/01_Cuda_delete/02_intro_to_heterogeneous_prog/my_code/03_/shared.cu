#include "hip/hip_runtime.h"

//=================================================
__global__ void local_memory(float in){

float f; // f in local memory and private to each thread.
f=in;    // "in" is in local memory and private to each thread.

}

//=================================================
__global__ void global_memory(float *array){

array[threadIdx.x] = 2.0f* (float) threadIdx.x;

}

//=================================================
__global__ void  shared_memory(float *array){

// local var, private to each thread
int i, index=threadIdx.x;
float average, sum = 0.0f;

// __shared__ variables are visible to all threads in the threads block 
// and have the lifetime of the thread block.
__shared__ float sh_arr[128];

// copy datda from array in global memory to sh_arr in shared memory.
sh_arr[128] = array[index];

__syncthreads();  // ensures all the writes to shared memory have completed.

// find the average of all previous elements
for(i=0; i < index; ++i) sum+= sh_arr[i] ; // it is faster to do the computation on the shared memory (sh_arr) rather than the global memory (array).
average = sum/(index+1.0f);

//if array[index] is greater than the average of array[0...index-1], replace it with the average.
// since array{} is in global memory, this change will be seen by the host!!! ( and potentially by other thread blocks!!!!)

if (array[index]>average) array[index]=average;

// the following piece of code has no effect bcs it modifies shared memory but the resulting modified data is never copied back to the global memory and vanishes when the thread block completes.
sh_arr[index] = 3.14;

}

//=================================================
int main(){

// for local memory
local_memory<<<1,128>>>(2.0);

// for global memory
float h_arr[128];
float *d_arr;

// allocate global memory on the device, 
hipMalloc((void **) &d_arr, 128*sizeof(float));

// copy data from the host memory to the device memory
hipMemcpy((void*)d_arr, (void*)h_arr, 128*sizeof(float), cudaMemcpyHostToDivice);

// launch the kernel
global_memory<<<1,128>>>(d_arr);

// copy back from the device memory to the host memory
hipMemcpy((void*)h_arr, (void*)d_arr, 128*sizeof(float), hipMemcpyDeviceToHost);


// shared memory
shared_memory<<<1,128>>>(d_arr);

return 0;
}