// Babak Poursartip
// 30 Oct 2019


#include <hip/hip_runtime.h>
#include <sstream>

__global__ void mykernel(void) {
	printf("Hello, World!\n");
}

int main(void){
	mykernel<<<1,1>>>();
	hipDeviceSynchronize();
	return 0;
}
