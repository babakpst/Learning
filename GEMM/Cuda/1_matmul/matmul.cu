#include <iostream>
#include <hip/hip_runtime.h>
#include <cudablaslt.h>

#define CHECK_CUDA_ERROR(call) { \ 
  hipError_t err = call; \
  if (err != hipSuccess) { \
    std::cerr << "CUDA error in " << __FILE__ << ":" << __LINE__ << ": " << hipGetErrorString(err) << std::endl; \
    exit(EXIT_FAILURE); \
  }\


  #define CHECK_CUBLAS_ERROR(call) { \ 
    hipblasStatus_t err = call; \
    if (err != HIPBLAS_STATUS_SUCCESS) { \
      std::cerr << "cuBLAS error in " << __FILE__ << ":" << __LINE__ << ": " << err << std::endl; \
      exit(EXIT_FAILURE); \
    }\
  }

 
int main() {
  std::cout << " test hipblasLtMatmul SGEMM NN" << std::endl;
  // operation: C_MxN = alpha * A_MxK * B_KxN + beta * C_MxN

  // define sizes
  const int M = 1024, N = 1024, K = 1024;
  const int lda = M, ldb = K, ldc = M;

  // allocate host memory/matrices
  float *h_A, *h_B, *h_C;
  h_A = (float *)malloc(M * K * sizeof(float));
  h_B = (float *)malloc(K * N * sizeof(float));
  h_C = (float *)malloc(M * N * sizeof(float));

  // initialize matrices
  for (int i = 0; i < M * K; i++) {
    h_A[i] = 1.0f;
  }

  for (int i = 0; i < K * N; i++) {
    h_B[i] = 1.0f;
  }

  for (int i = 0; i < M * N; i++) {
    h_C[i] = 1.0f;
  }

  // allocate device memory
  CHECK_CUDA_ERROR(hipMalloc(&d_A, M * K * sizeof(float)));
  CHECK_CUDA_ERROR(hipMalloc(&d_B, K * N * sizeof(float)));
  CHECK_CUDA_ERROR(hipMalloc(&d_C, M * N * sizeof(float)));

  // copy data to device
  CHECK_CUDA_ERROR(hipMemcpy(d_A, h_A, M * K * sizeof(float), hipMemcpyHostToDevice));
  CHECK_CUDA_ERROR(hipMemcpy(d_B, h_B, K * N * sizeof(float), hipMemcpyHostToDevice));
  CHECK_CUDA_ERROR(hipMemcpy(d_C, h_C, M * N * sizeof(float), hipMemcpyHostToDevice));

  // GEMM coefficients
  float alpha = 1.5f, beta = 2.0f;

  // create cublasLt handle
  hipblasLtHandle_t handle;
  CHECK_CUBLAS_ERROR(hipblasLtCreate(&handle));

  // create cublasLt matrix descriptors



}  





