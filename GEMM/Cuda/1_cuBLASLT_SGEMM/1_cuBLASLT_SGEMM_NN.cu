
/*
Sample cuBLAS/GEMM code for general matrix multiplication
Babak Poursartip

*/

#include <iostream>
#include <hip/hip_runtime.h>
#include <cudablaslt.h>

#define CHECK_CUDA_ERROR(call) { \ 
  hipError_t err = call; \
  if (err != hipSuccess) { \
    std::cerr << "CUDA error in " << __FILE__ << ":" << __LINE__ << ": " << hipGetErrorString(err) << std::endl; \
    exit(EXIT_FAILURE); \
  }\


  #define CHECK_CUBLAS_ERROR(call) { \ 
    hipblasStatus_t err = call; \
    if (err != HIPBLAS_STATUS_SUCCESS) { \
      std::cerr << "cuBLAS error in " << __FILE__ << ":" << __LINE__ << ": " << err << std::endl; \
      exit(EXIT_FAILURE); \
    }\
  }

 
int main() {
  std::cout << " test hipblasLtMatmul SGEMM NN" << std::endl;
  // operation: C_MxN = alpha * A_MxK * B_KxN + beta * C_MxN

  // define sizes
  const int M = 1024, N = 1024, K = 1024;
  const int lda = M, ldb = K, ldc = M;

  // allocate host memory/matrices
  float *h_A, *h_B, *h_C;
  h_A = (float *)malloc(M * K * sizeof(float));
  h_B = (float *)malloc(K * N * sizeof(float));
  h_C = (float *)malloc(M * N * sizeof(float));

  // initialize matrices
  for (int i = 0; i < M * K; i++) {
    h_A[i] = 1.0f;
  }

  for (int i = 0; i < K * N; i++) {
    h_B[i] = 2.0f;
  }

  for (int i = 0; i < M * N; i++) {
    h_C[i] = 3.0f;
  }

  // allocate device memory
  CHECK_CUDA_ERROR(hipMalloc(&d_A, M * K * sizeof(float)));
  CHECK_CUDA_ERROR(hipMalloc(&d_B, K * N * sizeof(float)));
  CHECK_CUDA_ERROR(hipMalloc(&d_C, M * N * sizeof(float)));

  // copy data to device
  CHECK_CUDA_ERROR(hipMemcpy(d_A, h_A, M * K * sizeof(float), hipMemcpyHostToDevice));
  CHECK_CUDA_ERROR(hipMemcpy(d_B, h_B, K * N * sizeof(float), hipMemcpyHostToDevice));
  CHECK_CUDA_ERROR(hipMemcpy(d_C, h_C, M * N * sizeof(float), hipMemcpyHostToDevice));

  // GEMM coefficients
  const float alpha = 1.5f, beta = 2.5f;
  

  // set up matrix descriptors
  cuBLASLtMatrixLayout_t Adesc, Bdesc, Cdesc;
  // hipblasLtMatrixLayoutCreate: This function creates a matrix layout descriptor by allocating the memory needed to hold its opaque structure.
  CHECK_CUBLAS_ERROR(hipblasLtMatrixLayoutCreate(&Adesc, HIP_R_32F, M, K, lda));
  CHECK_CUBLAS_ERROR(hipblasLtMatrixLayoutCreate(&Bdesc, HIP_R_32F, K, N, ldb));
  CHECK_CUBLAS_ERROR(hipblasLtMatrixLayoutCreate(&Cdesc, HIP_R_32F, M, N, ldc));


  // create cublasLt handle
  hipblasLtHandle_t handle;
  CHECK_CUBLAS_ERROR(hipblasLtCreate(&handle));

  // create operation descriptor
  hipblasLtMatmulDesc_t operationDesc;
  CHECK_CUBLAS_ERROR(hipblasLtMatmulDescCreate(&operationDesc, HIPBLAS_OP_N, HIPBLAS_OP_N, HIP_R_32F));

  // Perform GEMM operation

  CHECK_CUBLAS_ERROR(hipblasLtMatmul(handle, operationDesc, &alpha, d_A, Adesc, d_B, Bdesc, &beta, d_C, Cdesc, d_C, Cdesc, nullptr, nullptr, 0, 0));

  // Copy result back to host
  CHECK_CUDA_ERROR(hipMemcpy(h_C, d_C, M * N * sizeof(float), hipMemcpyDeviceToHost));

  // Free resources
  CHECK_CUBLAS_ERROR(hipblasLtMatrixLayoutDestroy(Adesc));
  CHECK_CUBLAS_ERROR(hipblasLtMatrixLayoutDestroy(Bdesc));
  CHECK_CUBLAS_ERROR(hipblasLtMatrixLayoutDestroy(Cdesc));
  CHECK_CUBLAS_ERROR(hipblasLtMatmulDescDestroy(operationDesc));
  CHECK_CUBLAS_ERROR(hipblasLtDestroy(handle));
  CHECK_CUDA_ERROR(hipFree(d_A));
  CHECK_CUDA_ERROR(hipFree(d_B));
  CHECK_CUDA_ERROR(hipFree(d_C));

  std::cout << "Result matrix C:" << std::endl;
    for (int i = 0; i < M; ++i) {
        for (int j = 0; j < N; ++j) {
            std::cout << C[i * N + j] << " ";
        }
        std::cout << std::endl;
    }

    return 0;



}  





